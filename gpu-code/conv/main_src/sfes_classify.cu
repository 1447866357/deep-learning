///
///  \file conv3.cu
///

#include <iostream>
#include <fstream>
#include <sstream>
#include <cmath>
#include <omp.h>
#include "mpi.h"
#include "train_classification.hpp"
#include "convnet.hpp"

using namespace std;


int Param::_minibatch_size = 0;

void managerNode(TrainClassification<float> *model){

	cout << "Loading data...\n";
	model->createWBiasForManager();
	cout << "Initialize weight and bias...\n";
	model->createPixelAndLabel();
	cout << "Loading data is done.\n";
	model->createMPIDist();
	cout << "done12\n";
	model->initWeightAndBcastByRandom();
	cout << "done13\n";
	model->sendAndRecvForManager();
	cout << "CPU number: " << omp_get_num_procs() << endl;  
}

void detectionNode(TrainClassification<float> *model){

	cout << "Initialize layers...\n";

	model->createLayerForWorker();
	cout << "Initialize layers is done.\n";
	model->createWBiasForWorker();
	cout << "done2\n";
	model->createPixelAndLabel();
	cout << "done3\n";
	model->createYDEDYForWorker();
	cout << "done4\n";
	model->createMPIDist();
	cout << "done5\n";
	model->initWeightAndBcastByRandom();
	cout << "done6\n";
	model->train();

}

int main(int argc, char** argv){

	int pid; 
	int num_process;
	int prov;
	MPI_Init_thread(&argc,&argv,MPI_THREAD_MULTIPLE, &prov);
	if (prov < MPI_THREAD_MULTIPLE)
	{   
		printf("Error: the MPI library doesn't provide the required thread level\n");
		MPI_Abort(MPI_COMM_WORLD, 0); 
	}   
	MPI_Comm_rank(MPI_COMM_WORLD,&pid);
	MPI_Comm_size(MPI_COMM_WORLD,&num_process);

	if(num_process <= 1){
		printf("Error: process number must bigger than 1\n");
		MPI_Abort(MPI_COMM_WORLD, 0); 
	}

	//检测有几个gpu
	int num_gpu;
	hipGetDeviceCount(&num_gpu);
	hipSetDevice(pid % num_gpu);


	TrainClassification<float> *voc_model = new TrainClassification<float>(0, pid, true, false);
//	TrainModel<float> *voc_model = new TrainModel<float>(0, pid);

	voc_model->parseNetJson("script/sfes.json");
	voc_model->parseImgBinary(num_process, "../../data/sfes_train_and_label.bin", \
			"../../data/sfes_test_and_label.bin");

	if(pid == 0){ 
		managerNode(voc_model);
	}   
	else{
		detectionNode(voc_model);
	}
	 	
	delete voc_model;
	MPI_Finalize();


	return 0;
}

















