///
///  \file conv3.cu
///

#include <iostream>
#include <fstream>
#include <sstream>
#include <cmath>
#include <omp.h>
#include "mpi.h"
#include "train_classification.hpp"
#include "convnet.hpp"

using namespace std;


int Param::_minibatch_size = 0;

void managerNode(TrainClassification<float> *model){

	cout << "Loading data...\n";
	model->createWBiasForManager();
	cout << "Initialize weight and bias...\n";
	model->createPixelAndLabel();
	cout << "Loading data is done.\n";
	model->createMPIDist();
	cout << "done12\n";
	model->initWeightAndBcast();
	cout << "done13\n";
	model->sendAndRecvForManager();
	cout << "CPU number: " << omp_get_num_procs() << endl;  
}

void detectionNode(TrainClassification<float> *model){

	cout << "Initialize layers...\n";

	model->createLayerForWorker();
	cout << "Initialize layers is done.\n";
	model->createWBiasForWorker();
	cout << "done2\n";
	model->createPixelAndLabel();
	cout << "done3\n";
	model->createYDEDYForWorker();
	cout << "done4\n";
	model->createMPIDist();
	cout << "done5\n";
	model->initWeightAndBcast();
	cout << "done6\n";
	model->train();

}

int main(int argc, char** argv){

	int pid; 
	int num_process;
	int prov;
	MPI_Init_thread(&argc,&argv,MPI_THREAD_MULTIPLE, &prov);
	if (prov < MPI_THREAD_MULTIPLE)
	{   
		printf("Error: the MPI library doesn't provide the required thread level\n");
		MPI_Abort(MPI_COMM_WORLD, 0); 
	}   
	MPI_Comm_rank(MPI_COMM_WORLD,&pid);
	MPI_Comm_size(MPI_COMM_WORLD,&num_process);

	if(num_process <= 1){
		printf("Error: process number must bigger than 1\n");
		MPI_Abort(MPI_COMM_WORLD, 0); 
	}

	//检测有几个gpu
	int num_gpu;
	hipGetDeviceCount(&num_gpu);
	hipSetDevice(pid % num_gpu);


	TrainClassification<float> *DIC_model = new TrainClassification<float>(0, pid);

	DIC_model->parseNetJson("script/DIC_seg_64.json");
	DIC_model->parseImgBinary(num_process, "../data/DIC_seg_train_320.bin", \
			"../data/DIC_seg_valid_320.bin");

	if(pid == 0){ 
		managerNode(DIC_model);
	}   
	else{
		detectionNode(DIC_model);
	}
	 	
	delete DIC_model;
	MPI_Finalize();


	return 0;
}

















