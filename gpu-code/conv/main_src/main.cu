#include "hip/hip_runtime.h"
/*
 * filename:main.cu
 */

#include <iostream>
#include <fstream>
#include <sstream>
#include <cmath>
#include <omp.h>
#include "mpi.h"
#include "nvmatrix.cuh"
#include "convnet.cuh"
#include "pooling_layer.cuh"
#include "inner_product_layer.cuh"
#include "utils.cuh"
#include "logistic.cuh"
#include "load_layer.hpp"
#include "layer_kernel.cuh"

using namespace std;

#define THREAD_END 100000
enum swapInfo{SWAP_CNN1_W_PUSH, SWAP_CNN1_BIAS_PUSH, \
	SWAP_CNN2_W_PUSH, SWAP_CNN2_BIAS_PUSH,	\
	SWAP_CNN3_W_PUSH, SWAP_CNN3_BIAS_PUSH,	\
	SWAP_INNER1_W_PUSH, SWAP_INNER1_BIAS_PUSH, \
	SWAP_SOFTMAX_W_PUSH, SWAP_SOFTMAX_BIAS_PUSH, \
	SWAP_CNN1_W_FETCH, SWAP_CNN1_BIAS_FETCH, \
	SWAP_CNN2_W_FETCH, SWAP_CNN2_BIAS_FETCH, \
	SWAP_CNN3_W_FETCH, SWAP_CNN3_BIAS_FETCH, \
	SWAP_INNER1_W_FETCH, SWAP_INNER1_BIAS_FETCH, \
	SWAP_SOFTMAX_W_FETCH, SWAP_SOFTMAX_BIAS_FETCH};

int num_process;
int rank;

void managerNode(pars* layer_pars){

	cout << "\n===========overall==============" \
		<< "\ntrain: " << layer_pars[0].num_train \
		<< "\nvalid: " << layer_pars[0].num_valid \
		<< "\nbatchSize: " << layer_pars[0].minibatch_size \
		<< "\nn_fetch: " << layer_pars[0].n_fetch \
		<< "\nn_push: " << layer_pars[0].n_push;

	cout << "\n===========cnn1==============" \
		<< "\nin_size: " << layer_pars[0].in_size \
		<< "\nin_channel: " << layer_pars[0].in_channel \
		<< "\nfilter_size: " << layer_pars[0].filter_size \
		<< "\nfilter_channel: " << layer_pars[0].filter_channel \
		<< "\nstride: " << layer_pars[0].stride \
		<< "\nw_lr: " << layer_pars[0].w_lr \
		<< "\nb_lr: " << layer_pars[0].b_lr \
		<< "\nmomentum: " << layer_pars[0].momentum \
		<< "\nweight_decay: " << layer_pars[0].weight_decay \
		<< "\nlr_scale: " << layer_pars[0].lr_down_scale;

	
	cout << "\n===========pool1==============" \
		<< "\nin_size: " << layer_pars[1].in_size \
		<< "\nin_channel: " << layer_pars[1].in_channel \
		<< "\nstride: " << layer_pars[1].stride \
		<< "\npool_size: " << layer_pars[1].pool_size;
	
	cout << "\n===========cnn2==============" \
		<< "\nin_size: " << layer_pars[2].in_size \
		<< "\nin_channel: " << layer_pars[2].in_channel \
		<< "\nfilter_size: " << layer_pars[2].filter_size \
		<< "\nfilter_channel: " << layer_pars[2].filter_channel \
		<< "\nstride: " << layer_pars[2].stride \
		<< "\nw_lr: " << layer_pars[2].w_lr \
		<< "\nb_lr: " << layer_pars[2].b_lr \
		<< "\nmomentum: " << layer_pars[2].momentum \
		<< "\nweight_decay: " << layer_pars[2].weight_decay \
		<< "\nlr_scale: " << layer_pars[2].lr_down_scale;

	cout << "\n===========pool2==============" \
		<< "\nin_size: " << layer_pars[3].in_size \
		<< "\nin_channel: " << layer_pars[3].in_channel \
		<< "\nstride: " << layer_pars[3].stride \
		<< "\npool_size: " << layer_pars[3].pool_size;

	cout << "\n===========cnn3==============" \
		<< "\nin_size: " << layer_pars[4].in_size \
		<< "\nin_channel: " << layer_pars[4].in_channel \
		<< "\nfilter_size: " << layer_pars[4].filter_size \
		<< "\nfilter_channel: " << layer_pars[4].filter_channel \
		<< "\nstride: " << layer_pars[4].stride \
		<< "\nw_lr: " << layer_pars[4].w_lr \
		<< "\nb_lr: " << layer_pars[4].b_lr \
		<< "\nmomentum: " << layer_pars[4].momentum \
		<< "\nweight_decay: " << layer_pars[4].weight_decay \
		<< "\nlr_scale: " << layer_pars[4].lr_down_scale;

	cout << "\n===========pool3==============" \
		<< "\nin_size: " << layer_pars[5].in_size \
		<< "\nin_channel: " << layer_pars[5].in_channel \
		<< "\nstride: " << layer_pars[5].stride \
		<< "\npool_size: " << layer_pars[5].pool_size;


	cout << "\n===========inner_product1==============" \
		<< "\nnum_in: " << layer_pars[6].num_in \
		<< "\nnum_out: " << layer_pars[6].num_out \
		<< "\nw_lr: " << layer_pars[6].w_lr \
		<< "\nb_lr: " << layer_pars[6].b_lr \
		<< "\nmomentum: " << layer_pars[6].momentum \
		<< "\nweight_decay: " << layer_pars[6].weight_decay \
		<< "\nlr_scale: " << layer_pars[6].lr_down_scale;

	cout << "\n===========softmax==============" \
		<< "\nnum_in: " << layer_pars[7].num_in \
		<< "\nnum_out: " << layer_pars[7].num_out \
		<< "\nw_lr: " << layer_pars[7].w_lr \
		<< "\nb_lr: " << layer_pars[7].b_lr \
		<< "\nmomentum: " << layer_pars[7].momentum \
		<< "\nweight_decay: " << layer_pars[7].weight_decay \
		<< "\nlr_scale: " << layer_pars[7].lr_down_scale << endl;


	int cnn1_in_len = layer_pars[0].in_size * layer_pars[0].in_size * layer_pars[0].in_channel;
	int cnn1_w_len = layer_pars[0].filter_channel * layer_pars[0].filter_size \
			* layer_pars[0].filter_size * layer_pars[0].in_channel;
	int cnn1_b_len = layer_pars[0].filter_channel;

	int cnn2_w_len = layer_pars[2].filter_channel * layer_pars[2].filter_size \
			* layer_pars[2].filter_size * layer_pars[2].in_channel;
	int cnn2_b_len = layer_pars[2].filter_channel;

	int cnn3_w_len = layer_pars[4].filter_channel * layer_pars[4].filter_size \
			* layer_pars[4].filter_size * layer_pars[4].in_channel;
	int cnn3_b_len = layer_pars[4].filter_channel;

	int inner1_w_len = layer_pars[6].num_in * layer_pars[6].num_out;
	int inner1_b_len = layer_pars[6].num_out;

	int softmax_w_len = layer_pars[7].num_in * layer_pars[7].num_out;
	int softmax_b_len = layer_pars[7].num_out;

	int train_data_len_part = layer_pars[0].num_train * cnn1_in_len / (num_process - 1);
	int train_label_len_part = layer_pars[0].num_train / (num_process - 1);
	int valid_data_len_part = layer_pars[0].num_valid * cnn1_in_len / (num_process - 1);
	int valid_label_len_part = layer_pars[0].num_valid / (num_process - 1);

cout << "done8\n";
	NVMatrix* train_data = new NVMatrix(layer_pars[0].num_train, cnn1_in_len);
	NVMatrix* valid_data = new NVMatrix(layer_pars[0].num_valid, cnn1_in_len);
	NVMatrix* train_label = new NVMatrix(layer_pars[0].num_train, 1);
	NVMatrix* valid_label = new NVMatrix(layer_pars[0].num_valid, 1);

/*
    readData(train_data, "../data/input/mnist_train.bin", true);
    readData(valid_data, "../data/input/mnist_valid.bin", true);
    readData(train_label, "../data/input/mnist_label_train.bin", false);
    readData(valid_label, "../data/input/mnist_label_valid.bin", false);
*/

cout << "done7\n";

	ImgInfo<float> *cifar10_info = new ImgInfo<float>;
	LoadCifar10<float> cifar10(cifar10_info);
    for(int i = 1; i < 6; i++){
        string s;
        stringstream ss;
        ss << i;
        ss >> s;    
		string filename = "../data/cifar-10-batches-bin/data_batch_"+s+".bin";
        cifar10.loadBinary(filename, cifar10_info->train_pixel_ptr, \
				cifar10_info->train_label_ptr);    
    }   
    cifar10.loadBinary("../data/cifar-10-batches-bin/test_batch.bin", \
            cifar10_info->test_pixel_ptr, cifar10_info->test_label_ptr);

	train_data->copyFromHost(cifar10_info->train_pixel, layer_pars[0].num_train * cnn1_in_len);
	train_label->copyFromHost(cifar10_info->train_label, layer_pars[0].num_train);
	valid_data->copyFromHost(cifar10_info->test_pixel, layer_pars[0].num_valid * cnn1_in_len);
	valid_label->copyFromHost(cifar10_info->test_label, layer_pars[0].num_valid);


cout << "done6\n";
	NVMatrix* cnn1_w = new NVMatrix(layer_pars[0].filter_size * \
			layer_pars[0].filter_size * layer_pars[0].in_channel, \
			layer_pars[0].filter_channel);
	NVMatrix* cnn1_bias = new NVMatrix(1, layer_pars[0].filter_channel);

	NVMatrix* cnn2_w = new NVMatrix(layer_pars[2].filter_size * \
			layer_pars[2].filter_size * layer_pars[2].in_channel, \
			layer_pars[2].filter_channel);
	NVMatrix* cnn2_bias = new NVMatrix(1, layer_pars[2].filter_channel);

	NVMatrix* cnn3_w = new NVMatrix(layer_pars[4].filter_size * \
			layer_pars[4].filter_size * layer_pars[4].in_channel, \
			layer_pars[4].filter_channel);
	NVMatrix* cnn3_bias = new NVMatrix(1, layer_pars[4].filter_channel);

	NVMatrix* inner1_w = new NVMatrix(inner1_w_len / layer_pars[6].num_out, layer_pars[6].num_out);
	NVMatrix* inner1_bias = new NVMatrix(1, layer_pars[6].num_out);


	NVMatrix* softmax_w = new NVMatrix(softmax_w_len / layer_pars[7].num_out, layer_pars[7].num_out);
	NVMatrix* softmax_bias = new NVMatrix(1, layer_pars[7].num_out);

cout << "done5\n";
	gaussRand(cnn1_w, 0.0001);
//	initW(cnn1_w);
	gaussRand(cnn2_w, 0.001);
//	initW(cnn2_w);
	gaussRand(cnn3_w, 0.01);
//	initW(cnn3_w);
	hipMemset(cnn1_bias->getDevData(), 0, sizeof(float) * cnn1_b_len);
	hipMemset(cnn2_bias->getDevData(), 0, sizeof(float) * cnn2_b_len);
	hipMemset(cnn3_bias->getDevData(), 0, sizeof(float) * cnn3_b_len);

	gaussRand(inner1_w, 0.1);
//	initW(inner1_w);
	hipMemset(inner1_bias->getDevData(), 0, sizeof(float) * inner1_b_len);
	gaussRand(softmax_w, 0.1);
//	initW(softmax_w);
	hipMemset(softmax_bias->getDevData(), 0, sizeof(float) * softmax_b_len);

	//	readPars(hHidVis, "hHidVis_t1.bin");
	//	readPars(hHidBiases, "hHidBiases_t1.bin");
	//	readPars(hsoftmax_w, "hsoftmax_w_t1.bin");
	//	readPars(hsoftmax_bias, "hsoftmax_bias_t1.bin");

	MPI_Bcast(cnn1_w->getDevData(), cnn1_w_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(cnn1_bias->getDevData(), cnn1_b_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(cnn2_w->getDevData(), cnn2_w_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(cnn2_bias->getDevData(), cnn2_b_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(cnn3_w->getDevData(), cnn3_w_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(cnn3_bias->getDevData(), cnn3_b_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(inner1_w->getDevData(), inner1_w_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(inner1_bias->getDevData(), inner1_b_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(softmax_w->getDevData(), softmax_w_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(softmax_bias->getDevData(), softmax_b_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	
	for(int i = 1; i < num_process; i++){
		MPI_Send(train_data->getDevData()+(i-1)*train_data_len_part, train_data_len_part, \
				MPI_FLOAT, i, i, MPI_COMM_WORLD);
		MPI_Send(train_label->getDevData()+(i-1)*train_label_len_part, \
				train_label_len_part, MPI_FLOAT, i, i, MPI_COMM_WORLD);
		MPI_Send(valid_data->getDevData()+(i-1)*valid_data_len_part, valid_data_len_part, \
				MPI_FLOAT, i, i, MPI_COMM_WORLD);
		MPI_Send(valid_label->getDevData()+(i-1)*valid_label_len_part, \
				valid_label_len_part, MPI_FLOAT, i, i, MPI_COMM_WORLD);

	}
	delete cifar10_info;
	delete train_data;
	delete train_label;
	delete valid_data;
	delete valid_label;

	//pro进程，每个进程进行的数据交换次数，0123是push，4567是fetch
	//4个数据地址，8个线程来分别实现两种操作
	const int trans_ops = 20;
	const int num_pars_type = 10;
	float* my_pars[num_pars_type] = {cnn1_w->getDevData(), cnn1_bias->getDevData(), \
			cnn2_w->getDevData(), cnn2_bias->getDevData(), \
			cnn3_w->getDevData(), cnn3_bias->getDevData(), \
			inner1_w->getDevData(), inner1_bias->getDevData(), \
			softmax_w->getDevData(), softmax_bias->getDevData()};
	int pars_len[num_pars_type] = {cnn1_w_len, cnn1_b_len, cnn2_w_len, cnn2_b_len, \
				cnn3_w_len, cnn3_b_len, inner1_w_len, inner1_b_len, \
				softmax_w_len, softmax_b_len};

	#pragma omp parallel num_threads(trans_ops * (num_process - 1)) 
	{

		MPI_Status status;
		int par_state = 0;

		int tid = omp_get_thread_num();
		int pid = tid / trans_ops + 1;
		int swap_id = tid % trans_ops;
		int pars_addr = tid % num_pars_type;

		while(par_state != THREAD_END){
			MPI_Recv(&par_state, 1, MPI_INT, pid, \
					swap_id, MPI_COMM_WORLD, &status);

			if(swap_id < num_pars_type){
				MPI_Recv(my_pars[pars_addr], pars_len[pars_addr], MPI_FLOAT, pid, \
						swap_id+ par_state, MPI_COMM_WORLD, &status);
			}else{
				MPI_Send(my_pars[pars_addr], pars_len[pars_addr], MPI_FLOAT, pid, \
						swap_id + par_state, MPI_COMM_WORLD);
			}   
		}
	}

	delete cnn1_w;
	delete cnn1_bias;
	delete cnn2_w;
	delete cnn2_bias;
	delete cnn3_w;
	delete cnn3_bias;
	delete inner1_w;
	delete inner1_bias;
	delete softmax_w;
	delete softmax_bias;
}


void workerNode(pars* layer_pars){
	
	layer_pars[0].num_train /= (num_process - 1);
	int cnn1_in_len = layer_pars[0].in_size * layer_pars[0].in_size * layer_pars[0].in_channel;

	int cnn1_w_len = layer_pars[0].filter_channel * layer_pars[0].filter_size \
			* layer_pars[0].filter_size * layer_pars[0].in_channel;
	int cnn1_b_len = layer_pars[0].filter_channel;

	int cnn2_w_len = layer_pars[2].filter_channel * layer_pars[2].filter_size \
			* layer_pars[2].filter_size * layer_pars[2].in_channel;
	int cnn2_b_len = layer_pars[2].filter_channel;

	int cnn3_w_len = layer_pars[4].filter_channel * layer_pars[4].filter_size \
			* layer_pars[4].filter_size * layer_pars[4].in_channel;
	int cnn3_b_len = layer_pars[4].filter_channel;

	int inner1_w_len = layer_pars[6].num_in * layer_pars[6].num_out;
	int inner1_b_len = layer_pars[6].num_out;

	int softmax_w_len = layer_pars[7].num_in * layer_pars[7].num_out;
	int softmax_b_len = layer_pars[7].num_out;

	int mini_data_len = layer_pars->minibatch_size * cnn1_in_len;
	int mini_label_len = layer_pars->minibatch_size;

	int train_data_len_part = layer_pars->num_train * cnn1_in_len;
	int train_label_len_part = layer_pars->num_train;
	int valid_data_len_part = layer_pars->num_valid * cnn1_in_len;
	int valid_label_len_part = layer_pars->num_valid;

cout << "done4\n";
	ConvNet cnn1(layer_pars);
	cnn1.initCuda();

	PoolingLayer pool1(layer_pars + 1);
	pool1.initCuda();

	ConvNet cnn2(layer_pars + 2);
	cnn2.initCuda();

	PoolingLayer pool2(layer_pars + 3);
	pool2.initCuda();

	ConvNet cnn3(layer_pars + 4);
	cnn3.initCuda();

	PoolingLayer pool3(layer_pars + 5);
	pool3.initCuda();

	InnerProductLayer inner1(layer_pars + 6);
	inner1.initCuda();

	Logistic softmax1(layer_pars + 7);
	softmax1.initCuda();


	NVMatrix* cnn1_w = cnn1.getW();
	NVMatrix* cnn1_bias = cnn1.getBias();
	NVMatrix* cnn2_w = cnn2.getW();
	NVMatrix* cnn2_bias = cnn2.getBias();
	NVMatrix* cnn3_w = cnn3.getW();
	NVMatrix* cnn3_bias = cnn3.getBias();
	NVMatrix* inner1_w = inner1.getW();
	NVMatrix* inner1_bias = inner1.getBias();
	NVMatrix* softmax_w = softmax1.getW();
	NVMatrix* softmax_bias = softmax1.getBias();

	MPI_Bcast(cnn1_w->getDevData(), cnn1_w_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(cnn1_bias->getDevData(), cnn1_b_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(cnn2_w->getDevData(), cnn2_w_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(cnn2_bias->getDevData(), cnn2_b_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(cnn3_w->getDevData(), cnn3_w_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(cnn3_bias->getDevData(), cnn3_b_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(inner1_w->getDevData(), inner1_w_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(inner1_bias->getDevData(), inner1_b_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(softmax_w->getDevData(), softmax_w_len, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(softmax_bias->getDevData(), softmax_b_len, MPI_FLOAT, 0, MPI_COMM_WORLD);

	NVMatrix* train_data = new NVMatrix(layer_pars->num_train, cnn1_in_len);
	NVMatrix* train_label = new NVMatrix(layer_pars->num_train, 1);
	NVMatrix* valid_data = new NVMatrix(layer_pars->num_valid, cnn1_in_len);
	NVMatrix* valid_label = new NVMatrix(layer_pars->num_valid, 1);

	NVMatrix* mini_data = new NVMatrix(layer_pars->minibatch_size, cnn1_in_len);
	NVMatrix* mini_label = new NVMatrix(layer_pars->minibatch_size, 1);

	MPI_Status status;
	MPI_Recv(train_data->getDevData(), train_data_len_part, \
			MPI_FLOAT, 0, rank, MPI_COMM_WORLD, &status);
	MPI_Recv(train_label->getDevData(), train_label_len_part, \
			MPI_FLOAT, 0, rank, MPI_COMM_WORLD, &status);
	MPI_Recv(valid_data->getDevData(), valid_data_len_part, \
			MPI_FLOAT, 0, rank, MPI_COMM_WORLD, &status);
	MPI_Recv(valid_label->getDevData(), valid_label_len_part, \
			MPI_FLOAT, 0, rank, MPI_COMM_WORLD, &status);

cout << "done9\n";
	int passMsg = 0;

	NVMatrix* cnn1_y;
	NVMatrix* cnn1_dE_dy;
	NVMatrix* pool1_y;
	NVMatrix* pool1_dE_dy;
	NVMatrix* cnn2_y;
	NVMatrix* cnn2_dE_dy;
	NVMatrix* pool2_y;
	NVMatrix* pool2_dE_dy;
	NVMatrix* cnn3_y;
	NVMatrix* cnn3_dE_dy;
	NVMatrix* pool3_y;
	NVMatrix* pool3_dE_dy;
	NVMatrix* inner1_y;
	NVMatrix* inner1_dE_dy;

	const int num_pars_type = 10;
	float* my_pars[num_pars_type] = {cnn1_w->getDevData(), cnn1_bias->getDevData(), \
			cnn2_w->getDevData(), cnn2_bias->getDevData(), \
			cnn3_w->getDevData(), cnn3_bias->getDevData(), \
			inner1_w->getDevData(), inner1_bias->getDevData(), \
			softmax_w->getDevData(), softmax_bias->getDevData()};
	int pars_len[num_pars_type] = {cnn1_w_len, cnn1_b_len, cnn2_w_len, cnn2_b_len, \
				cnn3_w_len, cnn3_b_len, inner1_w_len, inner1_b_len, \
			     softmax_w_len, softmax_b_len};

	clock_t t;
	t = clock();
	clock_t t1;
	t1 = clock();
	for(int epoch_idx = 0; epoch_idx < layer_pars->num_epoch; epoch_idx++){
		int error = 0;
/*
if(epoch_idx > 1){
	cnn1_w->showValue("cnn1w");
	cnn1_bias->showValue("cnn1b");
	cnn2_w->showValue("cnn2w");
	cnn2_bias->showValue("cnn2b");
	inner1_w->showValue("innerw");
	cnn3_w->showValue("cnn3w");
	cnn3_bias->showValue("cnn3b");
	inner1_w->showValue("inner1w");
	inner1_bias->showValue("inner1b");
	softmax_w->showValue("softmaxw");
	softmax_bias->showValue("softmaxb");
}
*/
		for(int batch_idx = 0; batch_idx < layer_pars->num_minibatch; batch_idx++){

			mini_data->changePtrFromStart(train_data->getDevData(), \
					mini_data_len * batch_idx);
			mini_label->changePtrFromStart(train_label->getDevData(), \
					mini_label_len * batch_idx);
//cout << "\n===================";
//		printTime(t, "start");
			cnn1.computeOutputs(mini_data);
			cnn1_y = cnn1.getY();
//		printTime(t, "cnn1 out");
			pool1.computeOutputs(cnn1_y);
			pool1_y = pool1.getY();
//		printTime(t, "pool out");
			cnn2.computeOutputs(pool1_y);			
			cnn2_y = cnn2.getY();
//		printTime(t, "cnn2 out");
			pool2.computeOutputs(cnn2_y);
			pool2_y = pool2.getY();
//		printTime(t, "pool2 out");
			cnn3.computeOutputs(pool2_y);
			cnn3_y = cnn3.getY();
//		printTime(t, "cnn3 out");
			pool3.computeOutputs(cnn3_y);
			pool3_y = pool3.getY();
//		printTime(t, "pool3 out");
			inner1.computeOutputs(pool3_y);
			inner1_y = inner1.getY();
//		printTime(t, "inner1 out");
			softmax1.computeOutputs(inner1_y);
			softmax1.computeError(mini_label, error);
//		printTime(t, "softmax out");

			softmax1.computeDerivsOfPars(inner1_y, mini_label);
//		printTime(t, "softmax pars");
			inner1_dE_dy = inner1.getDEDY();
			softmax1.computeDerivsOfInput(inner1_dE_dy);
//		printTime(t, "softmax input");
			inner1.computeDerivsOfPars(pool3_y);
//		printTime(t, "inner pars");
			pool3_dE_dy = pool3.getDEDY();
			inner1.computeDerivsOfInput(pool3_dE_dy);
//		printTime(t, "inner input");
			cnn3_dE_dy = cnn3.getDEDY();
			pool3.computeDerivsOfInput(cnn3_dE_dy);
//		printTime(t, "pool3 input");
			cnn3.computeDerivsOfPars(pool2_y);
//		printTime(t, "cnn3 pars");
			pool2_dE_dy = pool2.getDEDY();
			cnn3.computeDerivsOfInput(pool2_dE_dy);
//		printTime(t, "cnn3 input");
			cnn2_dE_dy = cnn2.getDEDY();
			pool2.computeDerivsOfInput(cnn2_dE_dy);
//		printTime(t, "pool2 input");
			cnn2.computeDerivsOfPars(pool1_y);
//		printTime(t, "cnn2 pars");
			pool1_dE_dy = pool1.getDEDY();
			cnn2.computeDerivsOfInput(pool1_dE_dy);
//		printTime(t, "cnn2 input");
			cnn1_dE_dy = cnn1.getDEDY();
			pool1.computeDerivsOfInput(cnn1_dE_dy);
//		printTime(t, "pool1 input");
			cnn1.computeDerivsOfPars(mini_data);
//		printTime(t, "cnn1 pars");

			cnn1.updatePars();
			cnn2.updatePars();
			cnn3.updatePars();
			inner1.updatePars();
			softmax1.updatePars();

			if((batch_idx + 1) % layer_pars->n_push == 0){
				if(epoch_idx == layer_pars->num_epoch - 1){
					if((batch_idx + layer_pars->n_push) >= layer_pars->num_minibatch \
							|| batch_idx == layer_pars->num_minibatch - 1)
						passMsg = THREAD_END;
					else
						passMsg = batch_idx;
				}
				else
					passMsg = batch_idx;
				#pragma omp parallel num_threads(num_pars_type)
				{
					int tid = omp_get_thread_num();
					int pars_addr = tid % num_pars_type;
					int swap_id = tid % num_pars_type;

					MPI_Send(&passMsg, 1, MPI_INT, 0, \
						swap_id, MPI_COMM_WORLD);
					MPI_Send(my_pars[pars_addr], pars_len[pars_addr], \
						MPI_FLOAT, 0, swap_id + passMsg, MPI_COMM_WORLD);
					
				}
			}
			if((batch_idx + 1) % layer_pars->n_fetch == 0){
				if(epoch_idx == layer_pars->num_epoch - 1){
					if((batch_idx + layer_pars->n_fetch) >= layer_pars->num_minibatch \
							|| batch_idx == layer_pars->num_minibatch - 1)
						passMsg = THREAD_END;
					else
						passMsg = batch_idx;
				}else
					passMsg = batch_idx;
			
				#pragma omp parallel num_threads(num_pars_type)
				{
					int tid = omp_get_thread_num();
					int pars_addr = tid % num_pars_type;
					int swap_id = tid % num_pars_type + num_pars_type;

					MPI_Send(&passMsg, 1, MPI_INT, 0, \
						swap_id, MPI_COMM_WORLD);
					MPI_Recv(my_pars[pars_addr], pars_len[pars_addr], \
						MPI_FLOAT, 0, swap_id + passMsg, MPI_COMM_WORLD, &status);
					
				}
			}

			if(batch_idx == layer_pars->num_minibatch - 1){ 
				int errorValid = 0;
				float loglihoodValid = 0;
				for(int validIdx = 0; validIdx < layer_pars->num_validbatch; validIdx++){

					mini_data->changePtrFromStart(valid_data->getDevData(), \
							mini_data_len * validIdx);
					mini_label->changePtrFromStart(valid_label->getDevData(), \
							mini_label_len * validIdx);
					cnn1.computeOutputs(mini_data);
					cnn1_y = cnn1.getY();
					pool1.computeOutputs(cnn1_y);
					pool1_y = pool1.getY();
					cnn2.computeOutputs(pool1_y);			
					cnn2_y = cnn2.getY();
					pool2.computeOutputs(cnn2_y);
					pool2_y = pool2.getY();
					cnn3.computeOutputs(pool2_y);			
					cnn3_y = cnn3.getY();
					pool3.computeOutputs(cnn3_y);
					pool3_y = pool3.getY();
					inner1.computeOutputs(pool3_y);
					inner1_y = inner1.getY();
					softmax1.computeOutputs(inner1_y);
					loglihoodValid += softmax1.computeError(mini_label, errorValid);

				}
				int totalValid = errorValid;
				if(num_process > 2){
					if(rank == 1){
						for(int i = 2; i < num_process; i++){
							MPI_Recv(&errorValid, 1, MPI_INT, i, i, \
									MPI_COMM_WORLD, &status);   
							totalValid += errorValid;
						}       
					}else{  
						MPI_Send(&errorValid, 1, MPI_INT, 1, rank, MPI_COMM_WORLD);
					}       
				}       
				if(rank == 1)
					cout << "epoch_idx: " << epoch_idx << ", error: " \
						<<  (float)totalValid/layer_pars->num_valid \
						<< ",likelihood: "<< loglihoodValid<< endl;
			}
		}
		
		if(rank == 1){
			t1 = clock() - t1;
			cout << " " << ((float)t1/CLOCKS_PER_SEC) << " seconds.\n";
			t1 = clock();
		}
		
		if((epoch_idx + 1) % 5 == 0){
			cout << "-----cnn1-----\n";
			cnn1.transfarLowerPars();
			cout << "-----cnn2-----\n";
			cnn2.transfarLowerPars();
			cout << "-----cnn3-----\n";
			cnn3.transfarLowerPars();
			cout << "-----inner1-----\n";
			inner1.transfarLowerPars();
			cout << "-----softmax1-----\n";
			softmax1.transfarLowerPars();
		}
		if((epoch_idx + 1)% 100 == 0){
        	string s;
        	stringstream ss;
        	ss << epoch_idx;
        	ss >> s;    
			savePars(cnn1_w, "../snapshot/w_snap/cnn1_w_" + s + "_t1.bin");
			cout << s << endl;
			savePars(cnn1_bias, "../snapshot/w_snap/cnn1_bias_" + s + "_t1.bin");
			savePars(cnn2_w, "../snapshot/w_snap/cnn2_w_" + s + "_t1.bin");
			savePars(cnn2_bias, "../snapshot/w_snap/cnn2_bias_" + s + "_t1.bin");
			savePars(cnn3_w, "../snapshot/w_snap/cnn3_w_" + s + "_t1.bin");
			savePars(cnn3_bias, "../snapshot/w_snap/cnn3_bias_" + s + "_t1.bin");
			savePars(inner1_w, "../snapshot/w_snap/inner1_w_" + s + "_t1.bin");
			savePars(inner1_bias, "../snapshot/w_snap/inner1_bias_" + s + "_t1.bin");
			savePars(softmax_w, "../snapshot/w_snap/softmax1_w_" + s + "_t1.bin");
			savePars(softmax_bias, "../snapshot/w_snap/softmax1_bias_" + s + "_t1.bin");
		}


	}
	if(rank == 1){
		t = clock() - t;
		cout << " " << ((float)t/CLOCKS_PER_SEC) / layer_pars->num_epoch << " seconds.\n";
		t = clock();
	}

	delete mini_data;
	delete mini_label;
	delete train_data;
	delete train_label;
	delete valid_data;
	delete valid_label;
}

int main(int argc, char** argv){

	int prov;
	MPI_Init_thread(&argc,&argv,MPI_THREAD_MULTIPLE, &prov);
	if (prov < MPI_THREAD_MULTIPLE)
	{   
		printf("Error: the MPI library doesn't provide the required thread level\n");
		MPI_Abort(MPI_COMM_WORLD, 0); 
	}   
	MPI_Comm_rank(MPI_COMM_WORLD,&rank);
	MPI_Comm_size(MPI_COMM_WORLD,&num_process);

	if(num_process <= 1){
		printf("Error: process number must bigger than 1\n");
		MPI_Abort(MPI_COMM_WORLD, 0); 
	}
	//检测有几个gpu
	int numGpus;
	hipGetDeviceCount(&numGpus);
	hipSetDevice(rank%numGpus);

/*
	// Ensure that RDMA ENABLED CUDA is set correctly
    int direct = getenv("MPICH_RDMA_ENABLED_CUDA")==NULL?0:atoi(getenv ("MPICH_RDMA_ENABLED_CUDA"));
    if(direct != 1){
        printf ("MPICH_RDMA_ENABLED_CUDA not enabled!\n");
        exit (EXIT_FAILURE);
    }
*/

	const int num_layer = 8;

	pars* layer_pars = new pars[num_layer];

	layer_pars[0].w_lr = 0.1;
	layer_pars[0].b_lr = 0.1;
	layer_pars[2].w_lr = 0.1;
	layer_pars[2].b_lr = 0.2;
	layer_pars[4].w_lr = 0.01;
	layer_pars[4].b_lr = 0.01;

	layer_pars[6].w_lr = 0.001;
	layer_pars[6].b_lr = 0.005;
	layer_pars[7].w_lr = 0.0005;
	layer_pars[7].b_lr = 0.0005;

	layer_pars[0].lr_down_scale = 0.95;
	layer_pars[2].lr_down_scale = 0.95;
	layer_pars[4].lr_down_scale = 0.95;
	layer_pars[6].lr_down_scale = 0.95;
	layer_pars[7].lr_down_scale = 0.95;

	layer_pars[0].weight_decay = 0;
	layer_pars[2].weight_decay = 0;
	layer_pars[4].weight_decay = 0;
	layer_pars[6].weight_decay = 0;
	layer_pars[7].weight_decay = 0;

//	layer_pars[0].w_lr = 5;
//	layer_pars[0].b_lr = 10;
	layer_pars[0].momentum = 0.9;
	layer_pars[0].in_size = 32; 
	layer_pars[0].in_channel = 3;
	layer_pars[0].filter_size = 5;
	layer_pars[0].filter_channel = 32; 
	layer_pars[0].stride = 1;
	layer_pars[0].pad = 2;
	layer_pars[0].padded_in_size = layer_pars[0].in_size + 2 * layer_pars[0].pad;
	layer_pars[0].out_size = (layer_pars[0].padded_in_size - layer_pars[0].filter_size) / layer_pars[0].stride + 1;
	layer_pars[0].num_train = 50000;
	layer_pars[0].num_valid = 10000;
	layer_pars[0].minibatch_size = 100;
	layer_pars[0].num_minibatch = layer_pars[0].num_train / (layer_pars[0].minibatch_size * (num_process - 1));
	layer_pars[0].num_validbatch = layer_pars[0].num_valid / (layer_pars[0].minibatch_size * (num_process - 1));
	layer_pars[0].num_epoch = 1; 
	layer_pars[0].n_push = 49;
	layer_pars[0].n_fetch = 50;
//	layer_pars[0].lr_down_scale = 0.95;

	layer_pars[1].in_size = layer_pars[0].out_size; 
	layer_pars[1].in_channel = layer_pars[0].filter_channel;
	layer_pars[1].filter_channel = layer_pars[0].filter_channel;
	layer_pars[1].pool_size = 3;
	layer_pars[1].stride = 2;
	layer_pars[1].out_size = ceil(((layer_pars[0].out_size - layer_pars[1].pool_size) * 1.0f) \
					 / layer_pars[1].stride) + 1;
	layer_pars[1].minibatch_size = layer_pars[0].minibatch_size;

//	layer_pars[2].w_lr = 1;
//	layer_pars[2].b_lr = 2;
	layer_pars[2].momentum = 0.9;
	layer_pars[2].in_size = layer_pars[1].out_size; 
	layer_pars[2].in_channel = layer_pars[1].filter_channel;
	layer_pars[2].filter_size = 5;
	layer_pars[2].filter_channel = 32; 
	layer_pars[2].stride = 1;
	layer_pars[2].pad = 2;
	layer_pars[2].padded_in_size = layer_pars[2].in_size + 2 * layer_pars[2].pad;
	layer_pars[2].out_size = (layer_pars[2].padded_in_size - layer_pars[2].filter_size) / layer_pars[2].stride + 1;
	layer_pars[2].minibatch_size = layer_pars[0].minibatch_size;
//	layer_pars[2].lr_down_scale = 0.95;


	layer_pars[3].in_size = layer_pars[2].out_size; 
	layer_pars[3].in_channel = layer_pars[2].filter_channel;
	layer_pars[3].filter_channel = layer_pars[2].filter_channel;
	layer_pars[3].pool_size = 3;
	layer_pars[3].stride = 2;
	layer_pars[3].out_size = ceil(((layer_pars[2].out_size - layer_pars[3].pool_size) * 1.0f)\
					 / layer_pars[3].stride) + 1;
	layer_pars[3].minibatch_size = layer_pars[2].minibatch_size;

//	layer_pars[4].w_lr = 1;
//	layer_pars[4].b_lr = 2;
	layer_pars[4].momentum = 0.9;
	layer_pars[4].in_size = layer_pars[3].out_size; 
	layer_pars[4].in_channel = layer_pars[3].filter_channel;
	layer_pars[4].filter_size = 5;
	layer_pars[4].filter_channel = 64; 
	layer_pars[4].stride = 1;
	layer_pars[4].pad = 2;
	layer_pars[4].padded_in_size = layer_pars[4].in_size + 2 * layer_pars[4].pad;
	layer_pars[4].out_size = (layer_pars[4].padded_in_size - layer_pars[4].filter_size) / layer_pars[4].stride + 1;
	layer_pars[4].minibatch_size = layer_pars[0].minibatch_size;
	layer_pars[4].lr_down_scale = 0.95;

	layer_pars[5].in_size = layer_pars[4].out_size; 
	layer_pars[5].in_channel = layer_pars[4].filter_channel;
	layer_pars[5].filter_channel = layer_pars[4].filter_channel;
	layer_pars[5].pool_size = 3;
	layer_pars[5].stride = 2;
	layer_pars[5].out_size = ceil(((layer_pars[4].out_size - layer_pars[5].pool_size) * 1.0f) \
					 / layer_pars[5].stride) + 1;
	layer_pars[5].minibatch_size = layer_pars[0].minibatch_size;

//	layer_pars[6].w_lr = 1;
//	layer_pars[6].b_lr = 2;
	layer_pars[6].momentum = 0.9;
	layer_pars[6].num_in = layer_pars[5].out_size * layer_pars[5].out_size * layer_pars[5].filter_channel;
	layer_pars[6].num_out = 64;
	layer_pars[6].minibatch_size = layer_pars[0].minibatch_size;
//	layer_pars[6].lr_down_scale = 0.95;

//	layer_pars[7].w_lr = 1;
//	layer_pars[7].b_lr = 2;
	layer_pars[7].momentum = 0.9;
	layer_pars[7].num_in = layer_pars[6].num_out;
	layer_pars[7].num_out = 10;
	layer_pars[7].minibatch_size = layer_pars[0].minibatch_size;
//	layer_pars[7].lr_down_scale = 0.95;



	if(rank == 0){ 
		managerNode(layer_pars);
	}   
	else{
		workerNode(layer_pars);
	} 	

	delete[] layer_pars;
	MPI_Finalize();
	return 0;
}



















