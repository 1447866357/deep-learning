#include "hip/hip_runtime.h"
/*
 * filename: pooling_layer.cu
 */

#include "pooling_layer.cuh"
#include "layer_kernel.cuh"

using namespace std;

PoolingLayer::PoolingLayer(pars* network){
	this->_in_size                  = network->in_size;
	this->_in_channel               = network->in_channel;
	this->_pool_size				= network->pool_size;
	this->_out_size					= this->_in_size / this->_pool_size;

	//w_hk的learning rate
	this->_w_lr                     = network->w_lr;
	//out bias learning rate
	this->_b_lr                     = network->b_lr;
	//上一次更新的参数控制增长趋势
	this->_momentum                 = network->momentum;
	this->_weight_decay             = network->weight_decay;

	this->_minibatch_size           = network->minibatch_size;
	this->_lr_down_scale            = network->lr_down_scale;

	hipblasCreate(&handle);
}

PoolingLayer::~PoolingLayer() {

	//	delete _w;
	//	delete _w_inc;
	//	delete _bias;
	//	delete _bias_inc;

	delete  _y;
	delete  _dE_dy;
	//	delete _dE_db;
	//	delete _dE_dw;
	hipblasDestroy(handle);
}

void PoolingLayer::initCuda() {

	//	this->_w            = new NVMatrix(_num_in, _num_out);
	//                  NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	//	this->_bias         = new NVMatrix(1, _num_out);
	//                  NVMatrix::ALLOC_ON_UNIFIED_MEMORY);

	this->_y               = new NVMatrix(_minibatch_size, \
					_out_size * _out_size * _in_channel);

	this->_dE_dy           = new NVMatrix(_y);
	//	this->_dE_db           = new NVMatrix(_bias);
	//	this->_dE_dw          = new NVMatrix(_w);

	//	this->_w_inc         = new NVMatrix(_w);
	//	this->_bias_inc        = new NVMatrix(1, _num_out);
	//	this->_w_inc->zeros();
	//	this->_bias_inc->zeros();

	hipError_t status = hipMalloc((void**) &_max_pos, \
			_minibatch_size * _in_channel * _out_size * _out_size * sizeof(int));
	if (status != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error\n");
		exit(EXIT_FAILURE);
	}

}

void PoolingLayer::computeOutputs(NVMatrix* x){
	dim3 blocks = dim3(_minibatch_size, _in_channel);
	dim3 threads = dim3(ceil(_out_size / 16.0) * 16,  ceil(_out_size / 16.0) * 16);
	//24*24,pooling到12*12
	max_pooling<<<blocks, threads, sizeof(float) * _in_size * _in_size>>>(x->getDevData(), \
			_y->getDevData(), _max_pos, _in_size, _out_size, _pool_size);  
	hipDeviceSynchronize();

}

void PoolingLayer::computeDerivsOfInput(NVMatrix* dE_dx){

	dim3 blocks = dim3(_minibatch_size, _in_channel);
	dim3 threads = dim3(ceil(_out_size / 16.0) * 16,  ceil(_out_size / 16.0) * 16);
	//dE_dy_h, 16*16*24*24
	dE_dx->zeros();
	compute_dE_dy_max<<<blocks, threads>>>(_dE_dy->getDevData(), \
			dE_dx->getDevData(), _max_pos, _in_size, \
			_out_size, _pool_size);
	hipDeviceSynchronize();

}

/*
   void ConvNet::computeAvgOutputs(){
	   //16*16
	   dim3 blocks = dim3(_minibatch_size, _in_channels);
	   dim3 threads = dim3(_out_size, _out_size);
	   //24*24,pooling到12*12
	   avg_pooling<<<blocks, threads>>>(_y_h->getDevData(), _y_i->getDevData());
	   hipDeviceSynchronize();
   }*/




