#include "hip/hip_runtime.h"
/*
 * filename: convnet.cu
 */
//#include <cutil_inline.h>
#include <time.h>

#include "convnet.cuh"
#include "layer_kernel.cuh"

using namespace std;


ConvNet::ConvNet(pars* netWork){

	this->_filter_channel           = netWork->filter_channel;

	this->_w_lr             		= netWork->w_lr;
	//hidden bias的learning rate
	this->_b_lr            			= netWork->b_lr;
	//上一次更新的参数控制增长趋势
	this->_momentum                 = netWork->momentum;
	//hidden原值的参数
	this->_weight_decay             = netWork->weight_decay;
	//out原值的参数
	this->_minibatch_size         	= netWork->minibatch_size;
	this->_in_size				 	= netWork->in_size;
	this->_pad						= netWork->pad;
	this->_padded_in_size			= _in_size + _pad * 2;
	this->_filter_size			 	= netWork->filter_size;
	this->_stride              		= netWork->stride;
	this->_out_size		 			= (_padded_in_size - _filter_size) / _stride + 1;
	this->_in_channel			 	= netWork->in_channel;
	this->_lr_down_scale			= netWork->lr_down_scale;
	this->_filt_pixs				= _filter_size * _filter_size;
	this->_conv_pixs				= _out_size * _out_size;
	hipblasCreate(&handle);
}
ConvNet::~ConvNet() {

	delete _w;
	delete _w_inc;
	delete _bias;
	delete _bias_inc;

	delete _y; 
	delete _dE_dy;
	delete _dE_dw;
	delete _dE_db;

	delete _dE_dx_sigmoid;
	
	delete unrolled_x1;
	delete unranged_y;
	delete unrolled_x2;
	delete ranged_dE_dx;
	delete dE_db_tmp;
	delete unrolled_conv;
	delete ranged_w;
	delete unranged_in;

	hipblasDestroy(handle);
}

void ConvNet::initCuda() {
	//hipSetDevice(cutGetAvgGflopsDeviceId());
	//NVMatrix::initDeviceProps();

	this->_w            = new NVMatrix(_filt_pixs * _in_channel, _filter_channel);
	//					NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	this->_bias         = new NVMatrix(1, _filter_channel);
	//					NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	this->_y               = new NVMatrix(_minibatch_size, \
					_filter_channel * _conv_pixs);
	this->_dE_dy           = new NVMatrix(_y);
	//dE_dx_sigmoid是对sigmoid函数的输入求导
	this->_dE_dx_sigmoid           = new NVMatrix(_y);
	this->_dE_dw          = new NVMatrix(_w);
	this->_dE_db           = new NVMatrix(_bias);

	this->_w_inc		 = new NVMatrix(_w);
	this->_bias_inc		 = new NVMatrix(_bias);

	//中间变量
	unrolled_x1 = new NVMatrix(_minibatch_size * _conv_pixs, \
			_filt_pixs * _in_channel);
	unranged_y = new NVMatrix(_minibatch_size * _conv_pixs, _filter_channel);

	unrolled_x2 = new NVMatrix(_filt_pixs * _in_channel, \
			_minibatch_size * _conv_pixs);
	ranged_dE_dx = new NVMatrix(_minibatch_size * _conv_pixs, _filter_channel);
	dE_db_tmp = new NVMatrix(_minibatch_size, _filter_channel);

	unrolled_conv = new NVMatrix(_minibatch_size * _padded_in_size * _padded_in_size, \
			_filter_size * _filter_size * _filter_channel);
	ranged_w = new NVMatrix(_filter_channel * _filt_pixs, _in_channel);
	unranged_in = new NVMatrix(_minibatch_size * _padded_in_size * _padded_in_size, _in_channel);


	this->_w_inc->zeros();
	this->_bias_inc->zeros();


}

void ConvNet::computeOutputs(NVMatrix* _x){

	//100*3*28*28 * 5*5, then add to 100*28*28 * 5*5


	int num_kernel = _minibatch_size * _conv_pixs * _filt_pixs *_in_channel;
	int num_block = num_kernel / 1024 + 1;
//	_x->reValue(32);
//	_w->reValue(1.0f);
	//_bias->reValue(2.0f);
	hipMemset(unrolled_x1->getDevData(), 0, sizeof(float) * num_kernel);
	im2col_filt<<<num_block, 1024>>>(_x->getDevData(), \
			unrolled_x1->getDevData(), num_kernel, \
			_in_size, _padded_in_size, \
			_in_channel, _filter_size, _out_size, _stride);

	unrolled_x1->rightMult(_w, 1, unranged_y, handle);

	unranged_y->addRowVector(_bias);

	num_kernel = _minibatch_size * _conv_pixs * _filter_channel;
	num_block = num_kernel / 1024 + 1;
	reshape_y<<<num_block, 1024>>>(unranged_y->getDevData(), _y->getDevData(), \
			num_kernel, _out_size, _filter_channel);
//	unrolled_x1->showValue("data");
	//_w->showValue("whk");
	//_y->showValue("yh");
}


void ConvNet::computeDerivsOfPars(NVMatrix* x){
	//assert(_minibatch_size % 16 == 0);

	//dE_dx_sigmoid, 16*16*24*24
	_y->subtractFromScalar(1, _dE_dx_sigmoid);

	_dE_dx_sigmoid->eltWiseMult(_y);

	_dE_dx_sigmoid->eltWiseMult(_dE_dy);

//_dE_dy->showValue("dedy");
//_dE_dx_sigmoid->showValue("dedxsigmoid");
	int num_kernel = _minibatch_size * _conv_pixs * _filt_pixs * _in_channel;
	int num_block = num_kernel / 1024 + 1;

//	x->reValue(32);
	//另外一种排列方式，因为需要排列的是24*24的块

	hipMemset(unrolled_x2->getDevData(), 0, sizeof(float) * num_kernel);
	im2col_conv<<<num_block, 1024>>>(x->getDevData(), \
			unrolled_x2->getDevData(), num_kernel, \
			_minibatch_size, _in_size, _padded_in_size, _in_channel, _filter_size, \
			_out_size, _stride);	
	//	}
	//_x->showValue("data1");
	//_dE_dx_sigmoid->reValue(12544);
//unrolled_x2->showValue("data");

	num_kernel = _minibatch_size * _conv_pixs * _filter_channel;
	num_block = num_kernel / 1024 + 1;
	reshape_dE_dx_sigmoid<<<num_block, 1024>>>(ranged_dE_dx->getDevData(), \
			_dE_dx_sigmoid->getDevData(), num_kernel, _out_size, _filter_channel);

	unrolled_x2->rightMult(ranged_dE_dx, 1, _dE_dw, handle);

//ranged_dE_dx->showValue("dedxdh");	
//_dE_dx_sigmoid->showValue("dedxsigmoid");
//_dE_dw->showValue("dedwhk");

	dim3 blocks = dim3(_minibatch_size, _filter_channel);
	dim3 threads = dim3(_out_size, _out_size);
	compute_dE_db<<<blocks, threads, sizeof(float)>>>(_dE_dx_sigmoid->getDevData(), \
			dE_db_tmp->getDevData(), _out_size);
	hipDeviceSynchronize();
	dE_db_tmp->sumRow(_dE_db);
}

void ConvNet::computeDerivsOfInput(NVMatrix* dE_dx){

	int num_kernel = _minibatch_size * _padded_in_size * _padded_in_size * _filt_pixs * _filter_channel;
	int num_block = 4096;
//	int num_block = num_kernel / 1024 + 1;

	hipMemset(unrolled_conv->getDevData(), 0, sizeof(float) * num_kernel);
	im2col_img<<<num_block, 1024>>>(_dE_dx_sigmoid->getDevData(), unrolled_conv->getDevData(), \
			num_kernel, _padded_in_size, _filter_channel, \
			_in_channel, _filter_size, _out_size, _stride);
	hipDeviceSynchronize();

//_w->reValue(50);
	num_kernel = _filter_channel * _filt_pixs * _in_channel;
	num_block = num_kernel / 1024 + 1;
	reshape_w<<<num_block, 1024>>>(ranged_w->getDevData(), \
			_w->getDevData(), num_kernel, _filter_size, \
			_filter_channel, _in_channel);
	hipDeviceSynchronize();

	unrolled_conv->rightMult(ranged_w, 1, unranged_in, handle);
	num_kernel = _minibatch_size * _padded_in_size * _padded_in_size * _in_channel;
	num_block = 4096;

//unranged_in->reValue(32*12);

	reshape_In<<<num_block, 1024>>>(dE_dx->getDevData(), unranged_in->getDevData(), \
			num_kernel, _in_size, _padded_in_size, _in_channel);
	hipDeviceSynchronize();

//t = clock() - t;
//cout << "3: " << ((float)t/CLOCKS_PER_SEC) << " seconds.\n";
//t = clock();
//	_w->showValue("whk");
//	ranged_w->showValue("rangWhk");
//	unrolled_conv->showValue("unrolledconv");
//	unranged_in->showValue("unrangIN");
//	dE_dx->showValue("dx");

}






























