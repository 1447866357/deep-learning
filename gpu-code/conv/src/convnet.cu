#include "hip/hip_runtime.h"
/*
 * filename: convnet.cu
 */
//#include <cutil_inline.h>
#include <time.h>

#include "convnet.cuh"
#include "convnet_kernel.cuh"

using namespace std;
	

ConvNet::ConvNet(Matrix* hHidVis, Matrix* hAvgOut, Matrix* hHidBiases, \
		Matrix*	hOutBiases,	pars* netWork){

	this->_numFilters            = hHidVis->getNumRows();
	this->_numOut                = hAvgOut->getNumCols();

	this->_hHidVis               = hHidVis;
	this->_hAvgOut               = hAvgOut;
	this->_hHidBiases            = hHidBiases;
	this->_hOutBiases            = hOutBiases;
	//w_ij的learning rate
	this->_epsHidVis             = netWork->epsHidVis;
	//w_hk的learning rate
	this->_epsAvgOut             = netWork->epsAvgOut;
	//hidden bias的learning rate
	this->_epsHidBias            = netWork->epsHidBias;
	//out bias learning rate
	this->_epsOutBias            = netWork->epsOutBias;
	//上一次更新的参数控制增长趋势
	this->_mom                   = netWork->mom;
	//hidden原值的参数
	this->_wcHidVis              = netWork->wcHidVis;
	//out原值的参数
	this->_wcAvgOut              = netWork->wcAvgOut;
	this->_minibatchSize         = netWork->minibatchSize;
	this->_inSize				 = netWork->inSize;
	this->_filterSize			 = netWork->filterSize;
	this->_convResultSize		 = _inSize - _filterSize + 1;
	this->_poolResultSize		 = this->_convResultSize / AVG_POOL_X;
	this->_inChannel			 = netWork->inChannel;
	hipblasCreate(&handle);
}
ConvNet::~ConvNet() {
	/*
		delete _hHidVis;
		delete _hHidVisInc;
		delete _hHidBiases;
		delete _hHidBiasInc;
		delete _hAvgOut;
		delete _hAvgOutInc;
		delete _hOutBiases;
		delete _hOutBiasInc;

		delete _hidVis;
		delete _hidVisInc;
		delete _hidBiases;
		delete _hidBiasInc;
		delete _avgOut;
		delete _avgOutInc;
		delete _outBiases;
		delete _outBiasInc;

		delete _y_h; 
		delete  _y_i; 
		delete  _y_j; 
		delete  _dE_dy_j;
		delete _dE_db_j;
		delete _dE_dw_ij;
		delete _dE_dy_i;
		delete _dE_dy_h;
		delete _dE_dx_h;
		delete _dE_dw_hk;
		delete _dE_db_h;
	*/ 
	hipblasDestroy(handle);
}

void ConvNet::initCuda() {
	//hipSetDevice(cutGetAvgGflopsDeviceId());
	//NVMatrix::initDeviceProps();

	//hidVis大小是16*5*5,bias是5*5
	this->_hidVis            = new NVMatrix(_hHidVis, true, \
											NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	this->_avgOut            = new NVMatrix(_hAvgOut, true, \
											NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	this->_hidBiases         = new NVMatrix(_hHidBiases, true, \
											NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	this->_outBiases         = new NVMatrix(_hOutBiases, true, \
											NVMatrix::ALLOC_ON_UNIFIED_MEMORY);

	this->_y_h               = new NVMatrix(_minibatchSize, \
			_numFilters * _convResultSize * _convResultSize);
	this->_y_i               = new NVMatrix(_minibatchSize, \
			_numFilters * _poolResultSize * _poolResultSize);
	this->_y_j               = new NVMatrix(_minibatchSize, _numOut);

	this->_dE_dy_j           = new NVMatrix(_y_j);
	this->_dE_db_j           = new NVMatrix(_outBiases);
	this->_dE_dw_ij          = new NVMatrix(_avgOut);
	this->_dE_dy_i           = new NVMatrix(_y_i);

	this->_dE_dy_h           = new NVMatrix(_y_h);
	this->_dE_dx_h           = new NVMatrix(_y_h);
	this->_dE_dw_hk          = new NVMatrix(_hidVis);
	this->_dE_db_h           = new NVMatrix(_hidBiases);

	this->_avgOutInc		 = new NVMatrix(_avgOut);
	this->_outBiasInc		 = new NVMatrix(1, _numOut);
	this->_hidVisInc		 = new NVMatrix(_numFilters, _filterSize * _filterSize);
	this->_hidBiasInc		 = new NVMatrix(_numFilters, 1);
	this->_avgOutInc->zeros();
	this->_outBiasInc->zeros();
	this->_hidVisInc->zeros();
	this->_hidBiasInc->zeros();
	
	hipError_t status = hipMalloc((void**) &_maxPoolPos, \
			_minibatchSize * _numFilters * _poolResultSize * _poolResultSize * sizeof(int));
    if (status != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error\n");
        exit(EXIT_FAILURE);
	}


}

void ConvNet::computeConvOutputs(NVMatrix* miniData){
	//16*16
	dim3 blocks = dim3(_minibatchSize, _numFilters);
	//28*5，此处需要改变，低效
	dim3 threads = dim3(_convResultSize, _convResultSize);
	int filConvtimes = _filterSize / _convResultSize;
	int imgConvtimes = _inSize / _convResultSize;
	convolution_forward<<<blocks, threads>>>(miniData->getDevData(), \
			_hidVis->getDevData(), _hidBiases->getDevData(), _y_h->getDevData(), \
			filConvtimes, imgConvtimes);
	hipDeviceSynchronize();
	//	cutilCheckMsg("Kernel execution failed");
}

void ConvNet::computeAvgOutputs(){
	//16*16
	dim3 blocks = dim3(_minibatchSize, _numFilters);
	dim3 threads = dim3(_poolResultSize, _poolResultSize);
	//24*24,pooling到12*12
	avg_pooling<<<blocks, threads>>>(_y_h->getDevData(), _y_i->getDevData());	
	hipDeviceSynchronize();
}

void ConvNet::computeMaxOutputs(){
	//16*16
	dim3 blocks = dim3(_minibatchSize, _numFilters);
	dim3 threads = dim3(_poolResultSize, _poolResultSize);
	//24*24,pooling到12*12
	max_pooling<<<blocks, threads>>>(_y_h->getDevData(), _y_i->getDevData(), \
			_maxPoolPos);	

	hipDeviceSynchronize();
}

void ConvNet::computeClassOutputs(){
	_y_i->rightMult(_avgOut, 1, _y_j, handle);

	_y_j->addRowVector(_outBiases);
	//防止_y_j溢出，保证E^x，控制在e^-15~e^15
	_y_j->apply(NVMatrix::SOFTMAX);

}

double ConvNet::computeError(const NVMatrix* miniLabels, int& numError){
	Matrix* hlabels = new Matrix(miniLabels->getNumRows(), miniLabels->getNumCols());
	miniLabels->copyToHost(hlabels);
	Matrix* y_j_CPU = new Matrix(_y_j->getNumRows(), _y_j->getNumCols());
	_y_j->copyToHost(y_j_CPU);
	Matrix* correctProbs = new Matrix(_y_j->getNumRows(), 1); 
	
	NVMatrix* maxPosOfOutGpu = new NVMatrix(_y_j->getNumRows(), 1);
	_y_j->maxPosInRow(maxPosOfOutGpu);
	Matrix* maxPosCpu = new Matrix(_y_j->getNumRows(), 1);
	maxPosOfOutGpu->copyToHost(maxPosCpu);

	for (int c = 0; c < _y_j->getNumRows(); c++) {
		int trueLabel = hlabels->getCell(c, 0);
		int predictLabel = maxPosCpu->getCell(c, 0);
		correctProbs->getCell(c, 0) = y_j_CPU->getCell(c, trueLabel);

		if(predictLabel != trueLabel)
			numError++;
	}   
	correctProbs->apply(Matrix::LOG);
	double result = -correctProbs->sum();
	hipDeviceSynchronize();

	delete hlabels;
	delete y_j_CPU;
	delete correctProbs;
	delete maxPosOfOutGpu;
	delete maxPosCpu;
	return result;
}

void ConvNet::computeDerivs(NVMatrix* miniData, NVMatrix* miniLabels){
	assert(_minibatchSize % 16 == 0);
	assert(miniLabels->getNumRows() == miniData->getNumRows());

	const int numThreads = DIVUP(_numOut, ADD_BLOCK_SIZE) * ADD_BLOCK_SIZE;
	compute_dE_dy_j<<<_minibatchSize, numThreads>>>(_y_j->getDevData(), \
			miniLabels->getDevData(), _dE_dy_j->getDevData(), _numOut);
	hipDeviceSynchronize();

	NVMatrix* y_i_T = new NVMatrix(_y_i->getNumCols(), _y_i->getNumRows());
	_y_i->getTranspose(y_i_T);
	//16*12*12 * 16, 16 * 10
	y_i_T->rightMult(_dE_dy_j, 1, _dE_dw_ij, handle);
/*
t = clock() - t;
cout << "dEdwij: " << (float)t/CLOCKS_PER_SEC << " seconds. \n";
t = clock();
*/
	//16 * 10
	_dE_dy_j->sumRow(_dE_db_j);

/*
t = clock() - t;
cout << "dEdbj: " << (float)t/CLOCKS_PER_SEC << " seconds. \n";
t = clock();
*/

	//dE_dy_i, 16*16*12*12
	NVMatrix* avgOut_T = new NVMatrix(_avgOut->getNumCols(), _avgOut->getNumRows());
	_avgOut->getTranspose(avgOut_T);
	_dE_dy_j->rightMult(avgOut_T, 1, _dE_dy_i, handle);


	//每次还原一个点，因为四个点只需还原一个，因此只用12*12的线程做
	dim3 blocks = dim3(_minibatchSize, _numFilters);
	dim3 threads = dim3(ceil(_poolResultSize / 16.0) * 16,  ceil(_poolResultSize / 16.0) * 16);
	//dE_dy_h, 16*16*24*24
/*
t = clock() - t;
cout << "dEdyi: " << (float)t/CLOCKS_PER_SEC << " seconds. \n";
t = clock();
*/

	_dE_dy_h->zeros();
	compute_dE_dy_h_max<<<blocks, threads>>>(_dE_dy_i->getDevData(), \
			_dE_dy_h->getDevData(), _maxPoolPos);
	hipDeviceSynchronize();
/*
t = clock() - t;
cout << "dEdyi: " << (float)t/CLOCKS_PER_SEC << " seconds. \n";
t = clock();
*/

	//dE_dx_h, 16*16*24*24
	_y_h->subtractFromScalar(1, _dE_dx_h);

	_dE_dx_h->eltWiseMult(_y_h);

	_dE_dx_h->eltWiseMult(_dE_dy_h);

//clock_t t = clock();
	NVMatrix* dE_dw_hk_tmp = new NVMatrix(_minibatchSize, \
			_numFilters * _filterSize *_filterSize);
	blocks = dim3(_minibatchSize, _numFilters);
	threads = dim3(_filterSize, _filterSize);
	int filConvtimes = _convResultSize / _filterSize;
	int imgConvtimes = _inSize / _filterSize;
	convolution_backward<<<blocks, threads>>>(miniData->getDevData(), \
			_dE_dx_h->getDevData(), dE_dw_hk_tmp->getDevData(), \
			filConvtimes, imgConvtimes);
	hipDeviceSynchronize();

	/*
t = clock() - t;
cout << "dEdwhktmp: " << (float)t/CLOCKS_PER_SEC << " seconds. \n";
t = clock();
*/

	//按每一列作为一个线程，故两者乘积要比16*24*24大
	dE_dw_hk_tmp->sumRow(_dE_dw_hk);
	
	NVMatrix* dE_db_h_tmp = new NVMatrix(_minibatchSize, _numFilters);
	blocks = dim3(_minibatchSize, _numFilters);
	threads = dim3(_convResultSize, _convResultSize);
	compute_dE_db_h<<<blocks, threads, sizeof(float)>>>(_dE_dx_h->getDevData(), \
			dE_db_h_tmp->getDevData());
	hipDeviceSynchronize();
/*
t = clock() - t;
cout << "dEdbhtmp: " << (float)t/CLOCKS_PER_SEC << " seconds. \n";
t = clock();
*/
	dE_db_h_tmp->sumRow(_dE_db_h);


	delete y_i_T;
	delete avgOut_T;
	delete dE_dw_hk_tmp;
	delete dE_db_h_tmp;

}

void ConvNet::updatePars(){
	
	_avgOutInc->addSum(_avgOut, _dE_dw_ij, _mom, -_wcAvgOut, \
			-_epsAvgOut / _minibatchSize);

	_avgOut->add(_avgOutInc, 1, 1);

	_outBiasInc->add(_dE_db_j, _mom, -_epsHidBias / _minibatchSize);
	_outBiases->add(_outBiasInc, 1, 1);

	_hidVisInc->addSum(_hidVis, _dE_dw_hk, _mom, -_wcHidVis, \
			-_epsHidVis / _minibatchSize);
	_hidVis->add(_hidVisInc, 1, 1);

	_hidBiasInc->add(_dE_db_h, _mom, -_epsHidBias / _minibatchSize);
	_hidBiases->add(_hidBiasInc, 1, 1);

}

void ConvNet::computeLogistic(NVMatrix* miniData, NVMatrix* miniLabels, bool isTrain){

//	clock_t t = clock();
	miniData->rightMult(_avgOut, 1, _y_j, handle);
	
//	t = clock() - t;                                                                                        
//		cout << "rightmulti1: " << (float)t/CLOCKS_PER_SEC << " seconds. \n";
//		t = clock();

	_y_j->addRowVector(_outBiases);
//	t = clock() - t;                                                                
//	cout << "yj: " << (float)t/CLOCKS_PER_SEC << " seconds. \n";
//	t = clock();

	_y_j->apply(NVMatrix::SOFTMAX);

	const int numThreads = DIVUP(_numOut, ADD_BLOCK_SIZE) * ADD_BLOCK_SIZE;
	compute_dE_dy_j<<<_minibatchSize, numThreads>>>(_y_j->getDevData(), \
			miniLabels->getDevData(), _dE_dy_j->getDevData(), _numOut);
//	t = clock() - t;                                                                
//	cout << "yj: " << (float)t/CLOCKS_PER_SEC << " seconds. \n";
//	t = clock();

	NVMatrix* y_i_T = new NVMatrix(miniData->getNumCols(), miniData->getNumRows());
	miniData->getTranspose(y_i_T);

	//16*12*12 * 16, 16 * 10
	y_i_T->rightMult(_dE_dy_j, 1, _dE_dw_ij, handle);

	//16 * 10
	_dE_dy_j->sumRow(_dE_db_j);

	if(isTrain){
		_avgOutInc->addSum(_avgOut, _dE_dw_ij, _mom, -_wcAvgOut, \
				-_epsAvgOut / _minibatchSize);

		_avgOut->add(_avgOutInc, 1, 1);

		_outBiasInc->add(_dE_db_j, _mom, -_epsHidBias / _minibatchSize);
		_outBiases->add(_outBiasInc, 1, 1);
	}
	delete y_i_T;
}
































