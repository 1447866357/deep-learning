#include "hip/hip_runtime.h"
/*
 * filename:testMultiGpu.cu
 */

#include <iostream>
#include <fstream>
#include <time.h>
#include <cmath>
#include "mpi.h"
#include "matrix.h"
#include "nvmatrix.cuh"
#include "convnet.cuh"
#include "convnet_kernel.cuh"

using namespace std;

void initW(float* a, int length){
	srand((unsigned)time(NULL));
	float bound = sqrt(1.0 / length);
	for(int i = 0; i < length; i++){
		int k = rand() % 200;
		if(k < 100)
			a[i] = (k/100.0)*(-bound);
		else
			a[i] = ((k - 100)/100.0)*bound; 
	}
}

void readPars(Matrix* par, string filename){
	ifstream fin1(filename.c_str(), ios::binary);
	int dataLen = par->getNumRows() * par->getNumCols();
	fin1.read((char*)(par->getData()), sizeof(float) * dataLen);
	fin1.close();
}

void savePars(Matrix* par, string filename){
	ofstream fout(filename.c_str(), ios::binary);
	int dataLen = par->getNumRows() * par->getNumCols();
	fout.write((char*)(par->getData()), sizeof(float) * dataLen);
	fout.close();
}

void readData(NVMatrix* nvData, string filename, bool isData, int addZerosInFront = 0){
	int length = nvData->getNumRows() * nvData->getNumCols();
	ifstream fin(filename.c_str(), ios::binary);
	float* data = new float[length];
	char* readData = new char[length];
	fin.read(readData + addZerosInFront, length - addZerosInFront);
	for(int i = 0; i < length; i++){
		if(i < addZerosInFront)
			readData[i] = 0;
		unsigned char tmp = readData[i];
		if(isData){
			data[i] = (int)tmp / 255.0;
		}
		else
			data[i] = (int)tmp;
	}
	nvData->copyFromHost(data, length);
	fin.close();
}

int main(int argc, char** argv){

	int rank;
	int numProcess;

	MPI_Init(&argc,&argv);
	MPI_Comm_rank(MPI_COMM_WORLD,&rank);
	MPI_Comm_size(MPI_COMM_WORLD,&numProcess);

	float epsHidVis = 0.001;
	float epsHidBias = 0.001;
	float epsAvgOut = 0.13;
	float epsOutBias = 0.13;
	float mom = 0;
	float wcHidVis = 0;
	float wcAvgOut = 0;

	int inSize = 28;
	int filterSize = 5;
	int numFilters = 16;
	int numOut = 10;
	int trainNum = 50000;
	int validNum = 10000;
	int minibatchSize = 1000;
	int numMinibatches = trainNum / (minibatchSize * numProcess);
	int numValidBatches = validNum / (minibatchSize * numProcess);
	int numEpoches = 1000; 
	int inChannel = 1;


	float* trainDataPtr;
	float* trainLabelPtr;
	float* validDataPtr;
	float* validLabelPtr;

	int inSqrt = inSize * inSize;

	int hidVisLen = numFilters * filterSize * filterSize;
	//	int hidBiasLen = numFilters;
	int avgOutLen = inSqrt * numOut;
	int outBiasLen = numOut;

	int miniDataLen = minibatchSize * inSqrt;
	int miniLabelLen = minibatchSize;

	hipSetDevice(rank%2);

	NVMatrix* nvTrainData;
	NVMatrix* nvValidData;
	NVMatrix* nvTrainLabel;
	NVMatrix* nvValidLabel;

	NVMatrix* miniTrainData;
	NVMatrix* miniTrainLabel;
	NVMatrix* miniValidData;
	NVMatrix* miniValidLabel;

	NVMatrix* avgOut;
	NVMatrix* outBiases;
	if(rank == 0){
		cout << "=========================\n" \
			<< "train: " << trainNum \
			<< "\nvalid: " << validNum \
			<< "\nfiltersize: " << filterSize \
			<< "\nnumFilters: " << numFilters \
			<< "\nepsHidVis: " << epsHidVis \
			<< "\nepsHidBias: " << epsHidBias \
			<< "\nepsAvgOut: " << epsAvgOut \
			<< "\nepsOutBias: " << epsOutBias \
			<< "\nmom: " << mom \
			<< "\nwcHidVis: " << wcHidVis \
			<< "\nwcAvgOut: " << wcAvgOut \
			<< "\n========================" << endl;

		/*
		 *注意此处，一个线程作为server时，要在文件前面补上一个minibatch的空数据
		 */

		nvTrainData = new NVMatrix(trainNum + minibatchSize, inSqrt, \
				NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
		nvValidData = new NVMatrix(validNum + minibatchSize, inSqrt, \
				NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
		nvTrainLabel = new NVMatrix(trainNum + minibatchSize, 1, \
				NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
		nvValidLabel = new NVMatrix(validNum + minibatchSize, 1, \
				NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
		avgOut = new NVMatrix(inSqrt, numOut, \
				NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
		outBiases = new NVMatrix(1, numOut, NVMatrix::ALLOC_ON_UNIFIED_MEMORY);

		//0号进程来读取输入数据
		readData(nvTrainData, "../data/input/mnist_train.bin", true, miniDataLen);
		readData(nvValidData, "../data/input/mnist_valid.bin", true, miniDataLen);
		readData(nvTrainLabel, "../data/input/mnist_label_train.bin", false, \
								minibatchSize);
		readData(nvValidLabel, "../data/input/mnist_label_valid.bin", false, \
								minibatchSize);

		//0号进程移动数据指针
		trainDataPtr = nvTrainData->getDevData();
		trainLabelPtr = nvTrainLabel->getDevData();
		validDataPtr = nvValidData->getDevData();
		validLabelPtr = nvValidLabel->getDevData();
	}
//	else{
		
		miniTrainData = new NVMatrix(minibatchSize, inSqrt, \
				NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
		miniTrainLabel = new NVMatrix(minibatchSize, 1, \
				NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
		miniValidData = new NVMatrix(minibatchSize, inSqrt, \
				NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
		miniValidLabel = new NVMatrix(minibatchSize, 1, \
				NVMatrix::ALLOC_ON_UNIFIED_MEMORY);	
//	}
	//参数全部都需要

	Matrix* hHidVis = new Matrix(numFilters, filterSize * filterSize);
	Matrix* hHidBiases = new Matrix(numFilters, 1);
	Matrix* hAvgout = new Matrix(inSqrt, numOut);
	Matrix* hOutBiases = new Matrix(1, numOut);

	//0号进程初始化参数，进行分发
	if(rank == 0){
		initW(hHidVis->getData(), hidVisLen);
		memset(hHidBiases->getData(), 0, sizeof(float) * numFilters);
		memset(hAvgout->getData(), 0, sizeof(float) * avgOutLen);
		memset(hOutBiases->getData(), 0, sizeof(float) * numOut);
		//	readPars(hHidVis, "hHidVis_t1.bin");
		//	readPars(hHidBiases, "hHidBiases_t1.bin");
		//	readPars(hAvgout, "hAvgout_t1.bin");
		//	readPars(hOutBiases, "hOutBiases_t1.bin");
	}
	//先只处理一层的logistic

	MPI_Bcast(hAvgout->getData(), avgOutLen, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(hOutBiases->getData(), outBiasLen, MPI_FLOAT, 0, MPI_COMM_WORLD);

	ConvNet layer1(hHidVis, hAvgout, hHidBiases, hOutBiases, epsHidVis, epsAvgOut, \
			epsHidBias, epsOutBias, mom, wcHidVis, wcAvgOut, minibatchSize, \
			inSize, filterSize, inChannel, numFilters);
	if(rank != 0){
		layer1.initCuda();
	}
	double loglihood = 0;

	int nPush = 1;
	int nFetch = 1;

	clock_t t;
	if(rank == 0)
		t = clock();
	for(int epochIdx = 0; epochIdx < numEpoches; epochIdx++){
		if(rank == 0){
			nvTrainData->setPtr(trainDataPtr);
			nvTrainLabel->setPtr(trainLabelPtr);
			nvValidData->setPtr(validDataPtr);
			nvValidLabel->setPtr(validLabelPtr);
		}
		for(int batchIdx = 0; batchIdx < numMinibatches; batchIdx++){
			//读取数据
			//使用send和receive
			
		//	MPI_Scatter(nvTrainData->getDevData(), miniDataLen, MPI_FLOAT, \
					miniTrainData->getDevData(), miniDataLen, MPI_FLOAT, \
					0, MPI_COMM_WORLD);
		//	MPI_Scatter(nvTrainLabel->getDevData(), miniLabelLen, MPI_FLOAT, \
					miniTrainLabel->getDevData(), miniLabelLen, MPI_FLOAT, \
					0, MPI_COMM_WORLD);
//			cout << "rank: " << rank << " done3\n";
			if(rank == 1){
				float* tmp = miniTrainData->getDevData();
				for(int i = 0; i < 100; i++){
					cout << tmp[i] << "  ";
				}			
				cout << endl;
			}

			int error = 0;
			if(rank != 0){
				//Forward pass

				layer1.computeLogistic(miniTrainData, miniTrainLabel, true);

				loglihood = layer1.computeError(miniTrainLabel, error);
			}
		/*	if(rank == 0){
				nvTrainData->changePtr((numProcess-1) * miniDataLen);
				nvTrainLabel->changePtr((numProcess-1) * miniLabelLen);
			}
			//点对点的send，然后再recv
			avgOut = layer1.getAvgOut();
			outBiases = layer1.getOutBias();
			if((batchIdx + 1) % nPush == 0){
				if(rank != 0){
					MPI_Send((rank-1)*avgOutLen/(numProcess-1) + avgOut->getDevData(), \
							avgOutLen/(numProcess-1), MPI_FLOAT, 0, 0, MPI_COMM_WORLD);
					MPI_Send((rank-1)*outBiasLen/(numProcess-1) + outBiases->getDevData(), \
							outBiasLen/(numProcess-1), MPI_FLOAT, 0, 0, MPI_COMM_WORLD);
				}else{
					for(int i = 1; i < numProcess; i++){
					MPI_Recv((rank-1)*avgOutLen/(numProcess-1) + avgOut->getDevData(), \
							avgOutLen/(numProcess-1), MPI_FLOAT, i, 0, MPI_COMM_WORLD, \
							MPI_STATUS_IGNORE);
					MPI_Recv((rank-1)*outBiasLen/(numProcess-1) + outBiases->getDevData(), \
							outBiasLen/(numProcess-1), MPI_FLOAT, i, 0, MPI_COMM_WORLD, \
							MPI_STATUS_IGNORE);
					}
				}
			}
			if((batchIdx + 1) % nFetch == 0){
				if(rank == 0){
					for(int i = 1; i < numProcess; i++){
					MPI_Send(avgOut->getDevData(), avgOutLen, MPI_FLOAT, i, \
							0, MPI_COMM_WORLD);
					MPI_Send(outBiases->getDevData(), outBiasLen, MPI_FLOAT, i, \
							0, MPI_COMM_WORLD);
					}
				}else{
					MPI_Recv(avgOut->getDevData(), avgOutLen, MPI_FLOAT, 0, 0, \
							MPI_COMM_WORLD, MPI_STATUS_IGNORE);
					MPI_Recv(outBiases->getDevData(), outBiasLen, MPI_FLOAT, 0, 0, \
							MPI_COMM_WORLD, MPI_STATUS_IGNORE);
				}
			}
			if(rank == 0){
				cout << "batchIdx: " << batchIdx << ",error: " \
					<< (float)error/minibatchSize \
					<< ",likelihood: "<< loglihood<< endl;
			}

			if(batchIdx == numMinibatches - 1){
				int errorValid = 0;
				float loglihoodValid = 0;
				for(int validIdx = 0; validIdx < numValidBatches; validIdx++){
					MPI_Scatter(nvValidData->getDevData(), miniDataLen, MPI_FLOAT, \
							miniValidData->getDevData(), miniDataLen, MPI_FLOAT, \
							0, MPI_COMM_WORLD);
					MPI_Scatter(nvValidLabel->getDevData(), miniLabelLen, MPI_FLOAT, \
							miniValidLabel->getDevData(), miniLabelLen, MPI_FLOAT, \
							0, MPI_COMM_WORLD);
					if(rank != 0){
						layer1.computeLogistic(miniValidData, miniValidLabel, false);
						loglihoodValid += layer1.computeError(miniValidLabel, errorValid);
					}
					else{
						nvValidData->changePtr((numProcess - 1) * miniDataLen);
						nvValidLabel->changePtr((numProcess - 1)* miniLabelLen);
					}
				}
				int totalValid;
				MPI_Reduce(&errorValid, &totalValid, 1, MPI_INT, MPI_SUM, \
						0, MPI_COMM_WORLD);
				if(rank == 0){
					t = clock() - t;
					cout << " " << (float)t/CLOCKS_PER_SEC << " seconds. \n";
					t = clock();
					cout << "epoch: " << epochIdx 
						<< ",error rate: " << (float)totalValid/validNum  \
						<< ",likelihood: "<< loglihoodValid << endl;
				}
			}*/
		}
	}
	//				savePars(hHidVis, "../data/pars/hHidVis_t1.bin");
	//  			savePars(hHidBiases, "../data/pars/hHidBiases_t1.bin");
	//				savePars(hAvgout, "../data/pars/hAvgout_t1.bin");
	//				savePars(hOutBiases, "../data/pars/hOutBiases_t1.bin");

	if(rank == 0){
		delete nvTrainData;
		delete nvTrainLabel;
		delete nvValidData;
		delete nvValidLabel;
	}
		delete miniTrainData;
		delete miniTrainLabel;
		delete miniValidData;
		delete miniValidLabel;
	

	MPI_Finalize();
	return 0;
}
