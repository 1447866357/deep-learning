#include "hip/hip_runtime.h"
///
/// \file recommendation_layer.cu
/// @brief

#include "recommendation_layer.hpp"

using namespace std;

template <typename Dtype>
RecommendationLayer<Dtype>::RecommendationLayer(FullConnectParam* fcp){

	this->_fcp           = fcp;
	if(_fcp->getLayerType() == RECOMMENDCOMPATIBLE)
		_is_compatible = true;
	else
		_is_compatible = false;
}

template <typename Dtype>
RecommendationLayer<Dtype>::~RecommendationLayer() {
	delete[] y_CPU;
	delete[] x_CPU;
	delete[] dE_dx_CPU;
	if(_is_compatible){
		delete[] w_CPU;
		delete[] dE_dw_CPU;
	}
	delete[] h_labels;
}

template <typename Dtype>
void RecommendationLayer<Dtype>::initCuda() {

	y_CPU             = new Dtype[_fcp->getMinibatchSize()/2];
	x_CPU 				= new Dtype[_fcp->getMinibatchSize()*_fcp->getNumIn()];
	dE_dx_CPU 				= new Dtype[_fcp->getMinibatchSize()*_fcp->getNumIn()];
	if(_is_compatible){
		w_CPU				= new Dtype[_fcp->getNumIn()*_fcp->getNumOut()];
		dE_dw_CPU				= new Dtype[_fcp->getNumIn()*_fcp->getNumOut()];
		dE_dw				= new Matrix<Dtype>(_fcp->getNumIn(), _fcp->getNumOut());
		gaussRand(w_CPU, _fcp->getNumIn()*_fcp->getNumOut(), 0.1);
	}

	h_labels = new int[_fcp->getMinibatchSize()];

}

template <typename Dtype>
double RecommendationLayer<Dtype>::computeError(Matrix<Dtype>* x, \
		Matrix<int>* labels){ 
//	x->reValue(_fcp->getNumIn(), true);
//	x->showValue("data");


	x->copyToHost(x_CPU, x->getNumEles());
	labels->copyToHost(h_labels, labels->getNumEles());
	
	memset(y_CPU, 0, (_fcp->getMinibatchSize()/2)*sizeof(int));
	
	double result = 0;
	for(int i=0; i < _fcp->getMinibatchSize()/2; i++){
		int pos_or_neg = 1;
		if(h_labels[2*i] < 0)
			pos_or_neg = -1;

		if(!_is_compatible){
			for(int j=0; j < x->getNumCols(); j++){
				y_CPU[i] += pow(x_CPU[i*2*x->getNumCols() + j] \
						- x_CPU[(i*2+1)*x->getNumCols() + j], 2); 
			}	
		}else{

			for(int k=0; k < _fcp->getNumOut(); k++){
				Dtype ele = 0;
				for(int j=0; j < _fcp->getNumIn(); j++){
					ele += (x_CPU[i*2*x->getNumCols() + j] \
							- x_CPU[(i*2+1)*x->getNumCols() + j])\
						*w_CPU[j*_fcp->getNumOut()+k]; 
				}
				y_CPU[i] += pow(ele, 2);
			}
		}
	//	cout << h_labels[i] << ",   y_cpu: "<< y_CPU[i] << endl;

		/***用log来算
		if(y_CPU[i] != 0)
			result -= log(y_CPU[i]);
		***/
		//负样本因为要减去结果值
		result += pos_or_neg*y_CPU[i];

		cout << h_labels[1+i*2] << "\t" << h_labels[2*i] << "\t";
		cout << y_CPU[i] << "\n";
	}
//cout << result << endl;	
	return result;

}

template <typename Dtype>
void RecommendationLayer<Dtype>::computeDerivsOfInput(Matrix<Dtype>* dE_dx){
	
	if(_is_compatible)
		memset(dE_dw_CPU, 0, _fcp->getNumIn()*_fcp->getNumOut()*sizeof(Dtype));
	for(int i=0; i < _fcp->getMinibatchSize()/2; i++){
		int pos_or_neg = 1;
		if(h_labels[2*i] < 0)
			pos_or_neg = -1;

		if(!_is_compatible){
			for(int j=0; j < dE_dx->getNumCols(); j++){
				dE_dx_CPU[i*2*dE_dx->getNumCols()+j] \
						= (x_CPU[i*2*dE_dx->getNumCols() + j] \
							- x_CPU[(i*2+1)*dE_dx->getNumCols() + j]) \
						*2*x_CPU[i*2*dE_dx->getNumCols() + j];
				dE_dx_CPU[(i*2+1)*dE_dx->getNumCols()+j] \
						= (x_CPU[(i*2+1)*dE_dx->getNumCols() + j] \
							- x_CPU[i*2*dE_dx->getNumCols() + j]) \
						*2*x_CPU[(i*2+1)*dE_dx->getNumCols() + j];

				/***用log算的时候的求导
				if(y_CPU[i] < 0.00001){
					dE_dx_CPU[i*2*dE_dx->getNumCols()+j] = 0;
					dE_dx_CPU[(i*2+1)*dE_dx->getNumCols()+j] = 0;
				}else{
					dE_dx_CPU[i*2*dE_dx->getNumCols()+j] \
						= (x_CPU[(i*2+1)*dE_dx->getNumCols() + j] \
							- x_CPU[i*2*dE_dx->getNumCols() + j]) / pow(y_CPU[i],2);
					dE_dx_CPU[(i*2+1)*dE_dx->getNumCols()+j] \
						= (x_CPU[i*2*dE_dx->getNumCols() + j] \
							- x_CPU[(i*2+1)*dE_dx->getNumCols() + j]) / pow(y_CPU[i],2);
				}***/
			}
		}else{

			for(int j=0; j < _fcp->getNumIn(); j++){
				Dtype tmp = 0;
				for(int k=0; k < _fcp->getNumOut(); k++){
					tmp += pow(w_CPU[j*_fcp->getNumOut()+k], 2);
				}
				if(y_CPU[i] == 0.0f){
					dE_dx_CPU[i*2*dE_dx->getNumCols()+j] = 0;
					dE_dx_CPU[(i*2+1)*dE_dx->getNumCols()+j] = 0;
				}else{
					dE_dx_CPU[i*2*dE_dx->getNumCols()+j] \
						= pos_or_neg*(x_CPU[i*2*dE_dx->getNumCols() + j] \
							- x_CPU[(i*2+1)*dE_dx->getNumCols() + j])*tmp;
					dE_dx_CPU[(i*2+1)*dE_dx->getNumCols()+j] \
						= pos_or_neg*(x_CPU[(i*2+1)*dE_dx->getNumCols() + j] \
							- x_CPU[i*2*dE_dx->getNumCols()+j])*tmp;
				}
			}
		}

		if(_is_compatible){
			
			for(int k=0; k < _fcp->getNumOut(); k++){
				for(int j=0; j < _fcp->getNumIn(); j++){
					if(y_CPU[i] == 0.0f){
						dE_dw_CPU[j*_fcp->getNumOut()+k] += 0;
					}else{
						dE_dw_CPU[j*_fcp->getNumOut()+k] += pos_or_neg \
							*pow(x_CPU[(i*2+1)*dE_dx->getNumCols() + j] \
							- x_CPU[i*2*dE_dx->getNumCols() + j], 2) \
							*w_CPU[j*_fcp->getNumOut()+k];
					}
				}
			}
		}
	}
//	cout << "-------------dedw_cpu-----------\n";
	if(_is_compatible){
		for(int k=0; k < _fcp->getNumOut(); k++){
			for(int j=0; j < _fcp->getNumIn(); j++){
//			cout << w_CPU[j*_fcp->getNumOut()+k] << ":";
				w_CPU[j*_fcp->getNumOut()+k] -= 0.00005*dE_dw_CPU[j*_fcp->getNumOut()+k]/_fcp->getMinibatchSize();
//			cout << w_CPU[j*_fcp->getNumOut()+k] << "\t";
			}
//		cout << endl;
		}
	}

	dE_dx->copyFromHost(dE_dx_CPU, dE_dx->getNumEles());


//dE_dx->showValue("Recommendation_dedx");

}


