#include "hip/hip_runtime.h"
/*
 * filename:testMultiGpu.cu
 */

#include <iostream>
#include <fstream>
#include <time.h>
#include <cmath>
#include "mpi.h"
#include "matrix.h"
#include "nvmatrix.cuh"
#include "convnet.cuh"
#include "convnet_kernel.cuh"

using namespace std;

void initW(float* a, int length){
	srand((unsigned)time(NULL));
	float bound = sqrt(1.0 / length);
	for(int i = 0; i < length; i++){
		int k = rand() % 200;
		if(k < 100)
			a[i] = (k/100.0)*(-bound);
		else
			a[i] = ((k - 100)/100.0)*bound; 
	}
}

void readPars(Matrix* par, string filename){
	ifstream fin1(filename.c_str(), ios::binary);
	int dataLen = par->getNumRows() * par->getNumCols();
	fin1.read((char*)(par->getData()), sizeof(float) * dataLen);
	fin1.close();
}

void savePars(Matrix* par, string filename){
	ofstream fout(filename.c_str(), ios::binary);
	int dataLen = par->getNumRows() * par->getNumCols();
	fout.write((char*)(par->getData()), sizeof(float) * dataLen);
	fout.close();
}

void readData(NVMatrix* nvData, string filename, bool isData){
	int length = nvData->getNumRows() * nvData->getNumCols();
	ifstream fin(filename.c_str(), ios::binary);
	float* data = new float[length];
	char* readData = new char[length];
	fin.read(readData, length);
	for(int i = 0; i < length; i++){
		unsigned char tmp = readData[i];
		if(isData){
			data[i] = (int)tmp / 255.0;
		}
		else
			data[i] = (int)tmp;
	}
	nvData->copyFromHost(data, length);
	fin.close();
}

int main(int argc, char** argv){

	int rank;
	int numProcess;

	MPI_Init(&argc,&argv);
	MPI_Comm_rank(MPI_COMM_WORLD,&rank);
	MPI_Comm_size(MPI_COMM_WORLD,&numProcess);
	cout<<"numprocess:"<<numProcess<<endl;

	float epsHidVis = 0.001;
	float epsHidBias = 0.001;
	float epsAvgOut = 0.13;
	float epsOutBias = 0.13;
	float mom = 0;
	float wcHidVis = 0;
	float wcAvgOut = 0;

	int inSize = 28;
	int filterSize = 5;
	int numFilters = 16;
	int numOut = 10;
	int trainNum = 50000;
	int validNum = 10000;
	int minibatchSize = 1000;
	int numMinibatches = trainNum / (minibatchSize * numProcess);
	int numValidBatches = validNum / (minibatchSize * numProcess);
	int numEpoches = 1000; 
	int inChannel = 1;


	float* trainDataPtr;
	float* trainLabelPtr;
	float* validDataPtr;
	float* validLabelPtr;

	int inSqrt = inSize * inSize;
	
	hipSetDevice(rank%2);
	NVMatrix* nvTrainData = new NVMatrix(trainNum, inSize * inSize, \
										NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvValidData = new NVMatrix(validNum, inSize * inSize, \
										NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvTrainLabel = new NVMatrix(trainNum, 1, \
										NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvValidLabel = new NVMatrix(validNum, 1, \
										NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	
	NVMatrix* miniTrainData = new NVMatrix(minibatchSize, inSqrt, \
										NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* miniTrainLabel = new NVMatrix(minibatchSize, 1, \
										NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* miniValidData = new NVMatrix(minibatchSize, inSqrt, \
										NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* miniValidLabel = new NVMatrix(minibatchSize, 1, \
										NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	if(rank == 0){
		cout << "=========================\n" \
			<< "train: " << trainNum \
			<< "\nvalid: " << validNum \
			<< "\nfiltersize: " << filterSize \
			<< "\nnumFilters: " << numFilters \
			<< "\nepsHidVis: " << epsHidVis \
			<< "\nepsHidBias: " << epsHidBias \
			<< "\nepsAvgOut: " << epsAvgOut \
			<< "\nepsOutBias: " << epsOutBias \
			<< "\nmom: " << mom \
			<< "\nwcHidVis: " << wcHidVis \
			<< "\nwcAvgOut: " << wcAvgOut \
			<< "\n========================" << endl;

		//0号进程来读取输入数据
		readData(nvTrainData, "../data/input/mnist_train.bin", true);
		readData(nvValidData, "../data/input/mnist_valid.bin", true);
		readData(nvTrainLabel, "../data/input/mnist_label_train.bin", false);
		readData(nvValidLabel, "../data/input/mnist_label_valid.bin", false);

		//0号进程移动数据指针
		trainDataPtr = nvTrainData->getDevData();
		trainLabelPtr = nvTrainLabel->getDevData();
		validDataPtr = nvValidData->getDevData();
		validLabelPtr = nvValidLabel->getDevData();
	}
	//参数全部都需要
	int hidVisLen = numFilters * filterSize * filterSize;
//	int hidBiasLen = numFilters;
	int avgOutLen = inSqrt * numOut;
	int outBiasLen = numOut;
	Matrix* hHidVis = new Matrix(numFilters, filterSize * filterSize);
	Matrix* hHidBiases = new Matrix(numFilters, 1);
	Matrix* hAvgout = new Matrix(inSqrt, numOut);
	Matrix* hOutBiases = new Matrix(1, numOut);

	int miniDataLen = minibatchSize * inSqrt;
	int miniLabelLen = minibatchSize;
	//0号进程初始化参数，进行分发
	NVMatrix* avgOut;
	NVMatrix* outBiases;
	avgOut = new NVMatrix(inSqrt, numOut, \
									NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	outBiases = new NVMatrix(1, numOut, NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	if(rank == 0){
		//avgOut = new NVMatrix(inSqrt, numOut, \
									NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
		//outBiases = new NVMatrix(1, numOut, NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
		initW(hHidVis->getData(), hidVisLen);
		memset(hHidBiases->getData(), 0, sizeof(float) * numFilters);
		memset(hAvgout->getData(), 0, sizeof(float) * avgOutLen);
		memset(hOutBiases->getData(), 0, sizeof(float) * numOut);
		//	readPars(hHidVis, "hHidVis_t1.bin");
		//	readPars(hHidBiases, "hHidBiases_t1.bin");
		//	readPars(hAvgout, "hAvgout_t1.bin");
		//	readPars(hOutBiases, "hOutBiases_t1.bin");
	}
	//先只处理一层的logistic

	MPI_Bcast(hAvgout->getData(), avgOutLen, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(hOutBiases->getData(), outBiasLen, MPI_FLOAT, 0, MPI_COMM_WORLD);

	ConvNet layer1(hHidVis, hAvgout, hHidBiases, hOutBiases, epsHidVis, epsAvgOut, \
			epsHidBias, epsOutBias, mom, wcHidVis, wcAvgOut, minibatchSize, \
			inSize, filterSize, inChannel, numFilters);
	layer1.initCuda();
	double loglihood = 0;

	int nPush = 1;
	int nFetch = 1;

	clock_t t;
	if(rank == 0)
		t = clock();
	for(int epochIdx = 0; epochIdx < numEpoches; epochIdx++){
		if(rank == 0){
			nvTrainData->setPtr(trainDataPtr);
			nvTrainLabel->setPtr(trainLabelPtr);
			nvValidData->setPtr(validDataPtr);
			nvValidLabel->setPtr(validLabelPtr);
		}

		for(int batchIdx = 0; batchIdx < numMinibatches; batchIdx++){
			//读取数据
			MPI_Scatter(nvTrainData->getDevData(), miniDataLen, MPI_FLOAT, \
					miniTrainData->getDevData(), miniDataLen, MPI_FLOAT, \
					0, MPI_COMM_WORLD);
			MPI_Scatter(nvTrainLabel->getDevData(), miniLabelLen, MPI_FLOAT, \
					miniTrainLabel->getDevData(), miniLabelLen, MPI_FLOAT, \
					0, MPI_COMM_WORLD);
			int error = 0;
			//Forward pass
			layer1.computeLogistic(miniTrainData, miniTrainLabel, true);

			loglihood = layer1.computeError(miniTrainLabel, error);
			
			if(rank == 0){
				nvTrainData->changePtr(numProcess * miniDataLen);
				nvTrainLabel->changePtr(numProcess * miniLabelLen);
			}

			//点对点的send，然后再recv
			NVMatrix* avgOutTmp = layer1.getAvgOut();
			NVMatrix* outBiasesTmp = layer1.getOutBias();
			if((batchIdx + 1) % nPush == 0){
				MPI_Gather(rank*avgOutLen/numProcess + avgOutTmp->getDevData(), \
						avgOutLen/numProcess, MPI_FLOAT, avgOut->getDevData(), \
						avgOutLen/numProcess, MPI_FLOAT, 0, MPI_COMM_WORLD);
				MPI_Gather(rank*outBiasLen/numProcess + outBiasesTmp->getDevData(), \
						outBiasLen/numProcess, MPI_FLOAT, outBiases->getDevData(), \
						outBiasLen/numProcess, MPI_FLOAT, 0, MPI_COMM_WORLD);
			}
			if((batchIdx + 1) % nFetch == 0){
				MPI_Bcast(avgOut->getDevData(), avgOutLen, MPI_FLOAT, \
						0, MPI_COMM_WORLD);
				MPI_Bcast(outBiases->getDevData(), outBiasLen, MPI_FLOAT, \
						0, MPI_COMM_WORLD);
				//数据复制回参数
				avgOutTmp->copyFromDevice(avgOut);
				outBiasesTmp->copyFromDevice(outBiases);
			}
		/*	if(rank == 0){
				cout << "batchIdx: " << batchIdx << ",error: " \
					<< (float)error/minibatchSize \
					<< ",likelihood: "<< loglihood<< endl;
			}
*/
			if(batchIdx == numMinibatches - 1){
				int errorValid = 0;
				float loglihoodValid = 0;
				for(int validIdx = 0; validIdx < numValidBatches; validIdx++){
					MPI_Scatter(nvValidData->getDevData(), miniDataLen, MPI_FLOAT, \
								miniValidData->getDevData(), miniDataLen, MPI_FLOAT, \
								0, MPI_COMM_WORLD);
					MPI_Scatter(nvValidLabel->getDevData(), miniLabelLen, MPI_FLOAT, \
							miniValidLabel->getDevData(), miniLabelLen, MPI_FLOAT, \
							0, MPI_COMM_WORLD);

					layer1.computeLogistic(miniValidData, miniValidLabel, false);
					loglihoodValid += layer1.computeError(miniValidLabel, errorValid);

					if(rank == 0){
						nvValidData->changePtr(numProcess * miniDataLen);
						nvValidLabel->changePtr(numProcess * miniLabelLen);
					}
				}
				int totalValid;
				MPI_Reduce(&errorValid, &totalValid, 1, MPI_INT, MPI_SUM, \
						0, MPI_COMM_WORLD);
				if(rank == 0){
					t = clock() - t;
					cout << " " << (float)t/CLOCKS_PER_SEC << " seconds. \n";
					t = clock();
					cout << "epoch: " << epochIdx 
						<< ",error rate: " << (float)totalValid/validNum  \
						<< ",likelihood: "<< loglihoodValid << endl;
				}
			}
		}
	}
/*	if(rank == 0){
		savePars(hHidVis, "../data/pars/hHidVis_t1.bin");
		savePars(hHidBiases, "../data/pars/hHidBiases_t1.bin");
		savePars(hAvgout, "../data/pars/hAvgout_t1.bin");
		savePars(hOutBiases, "../data/pars/hOutBiases_t1.bin");
	}
*/
	delete nvTrainData;
	delete nvTrainLabel;
	delete nvValidData;
	delete nvValidLabel;
	delete miniTrainData;
	delete miniTrainLabel;
	delete miniValidData;
	delete miniValidLabel;
	MPI_Finalize();
	return 0;
}
