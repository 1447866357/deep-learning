
//#include "svd.hpp"
using namespace std;

template <typename Dtype>
SVD<Dtype>::SVD(Matrix<Dtype> *A, const int m, const int n) \
        : _A(A), _height(m), _width(n), _alpha(0), \
        _sigma_u(0), _beta(0), _sigma_v(0), _scale_one(1), \
        _scale_minus_one(-1), _scale_zero(0){

    _householder_mat_p = new Matrix<Dtype>(n, n);
    _householder_mat_q = new Matrix<Dtype>(m, m);

    _householder_vec_u = new Matrix<Dtype>(m, 1);
    _householder_vec_v = new Matrix<Dtype>(n, 1);

    _w = new Matrix<Dtype>(m, 1);
    _z = new Matrix<Dtype>(n, 1);
    _x = new Matrix<Dtype>(n, 1);

    _k = new Matrix<Dtype>(m, 1);
    _l = new Matrix<Dtype>(n, 1);

    _h = new Matrix<Dtype>(m, m);
    _g = new Matrix<Dtype>(n, n);

    _cropped_A_for_u_v = new Matrix<Dtype>(m, 1);
    _cropped_A_for_z_w = new Matrix<Dtype>(m, n);

    hipblasCreate(&handle);
}

template <typename Dtype>
SVD<Dtype>::~SVD(){
    delete _householder_mat_p;
    delete _householder_mat_q;

    delete _householder_vec_u;
    delete _householder_vec_v;

    delete _w;
    delete _z;
    delete _x;
    delete _k;
    delete _l;
    delete _h;
    delete _g;

    delete _cropped_A_for_u_v;
    delete _cropped_A_for_z_w;
//    cublasDestory(&handle);
}

template <typename Dtype>
void SVD<Dtype>::computeHouseHolderVecU(const int vec_start_idx){
   	_vec_start_idx = vec_start_idx;
	_vec_u_len = _height - vec_start_idx;
    _vec_v_len = _width - vec_start_idx - 1;

    _A->cropMatToNew(_cropped_A_for_u_v, _vec_start_idx, \
			_vec_u_len, _vec_start_idx, 1);

    computeHouseHolderVecAndAlpha(_vec_u_len, \
         _householder_vec_u, _alpha, _sigma_u);

    cout << _vec_u_len << ":" << _alpha << ":" << _sigma_u << endl;
//    _cropped_A_for_u_v->showValue("cropped_a");
    _householder_vec_u->showValue("u");

}

template <typename Dtype>
void SVD<Dtype>::computeHouseHolderVecV(){
    if (_vec_v_len <= 0) {
        return;
    }
    _A->cropMatToNew(_cropped_A_for_u_v, _vec_start_idx, 1, \
          _vec_start_idx+1, _vec_v_len);
    computeHouseHolderVecAndAlpha(_vec_v_len, \
         _householder_vec_v, _beta, _sigma_v);
}

template <typename Dtype>
void SVD<Dtype>::computeHouseHolderVecAndAlpha(const int vec_len, \
		Matrix<Dtype> *householder_vector_gpu, Dtype &alpha_cpu, \
        Dtype &sigma_gpu){
    Dtype u_norm = _cropped_A_for_u_v->computeNorm(vec_len);
    Dtype y1_u = _cropped_A_for_u_v->getFirstPosValue();

//    cout << y1_u << ":"<< u_norm << endl;
    alpha_cpu = y1_u > 0 ? -u_norm : u_norm;
    sigma_gpu = (y1_u - alpha_cpu) / (-alpha_cpu);
    kComputeHouseholderVec<<<1, 1024>>>(_cropped_A_for_u_v->getDevData(), \
            householder_vector_gpu->getDevData(), \
			-alpha_cpu, 1/(y1_u - alpha_cpu), vec_len);
}

template <typename Dtype>
void SVD<Dtype>::computeH() {
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, _vec_u_len, \
				_vec_u_len, 1, &_scale_one, \
				_householder_vec_u->getDevData(), _vec_u_len, \
		        _householder_vec_u->getDevData(), 1, &_scale_zero, \
                _h->getDevData(), _vec_u_len);
    // Hi = I - sigma_u * u * u'
    _h->subedByUnitMat();
}

//更新A(i:m, i)
template <typename Dtype>
void SVD<Dtype>::eliminateAForV() {
	A->setValueAt(_vec_start_idx, _vec_start_idx, _alpha);
	for(int i = _vec_start_idx+1; i < _height; i++) {
		A->setValueAt(i, _vec_start_idx, 0);
	}
}

//更新A(i, i+1:n)
template <typename Dtype>
void SVD<Dtype>::eliminateAForU() {
	A->setValueAt(_vec_start_idx, _vec_start_idx+1, _beta);
	for(int i = _vec_start_idx+2; i < _width; i++) {
		A->setValueAt(_vec_start_idx, i, 0);
	}
}

//更新Q(1:m, i)
template <typename Dtype>
void SVD<Dtype>::updateQ() {
	
/*	
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, vec_len, vec_len,
                1, &_scale_one, _householder_vec_u->getDevData(), \
				vec_len, _householder_vec_u->getDevData(), 1, \
				&_scale_zero, _delta_q->getDevData(), vec_len);
*/	
}





