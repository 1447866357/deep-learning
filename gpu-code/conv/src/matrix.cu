#include "hip/hip_runtime.h"
///
/// \file matrix.cu
/// \brief 矩阵类源文件

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include "matrix.hpp"
#include "matrix_kernel.hpp"

using namespace std;

template <typename Dtype>
Matrix<Dtype>::Matrix(int num_row, int num_col){
	_init(num_row, num_col);
}

template <typename Dtype>
Matrix<Dtype>::Matrix(const Matrix<Dtype>* like, bool copy){
	_init(like->getNumRows(), like->getNumCols());
	if (copy) {
		copyFromDevice(like);
	}
}

template <typename Dtype>
Matrix<Dtype>::Matrix(const Matrix<Dtype>* like) {
	_init(like->getNumRows(), like->getNumCols());
}

template <typename Dtype>
Matrix<Dtype>::~Matrix(){
	if(this->_is_own_data && this->_amount > 0){
		hipFree(this->_data_value);
	}
}

template <typename Dtype>
void Matrix<Dtype>::_init(int num_row, int num_col) {
	this->_shape.push_back(num_row);
	this->_shape.push_back(num_col);
	this->_amount = num_row * num_col;
	this->_is_own_data = true;
	if (this->_amount > 0) {
		hipError_t status;
		status = hipMalloc((void**) &this->_data_value, \
				this->_amount * sizeof(Dtype));
		/*
		else if(a == ALLOC_ON_UNIFIED_MEMORY){
			status = hipMallocManaged(&this->_data_value, \
				this->_shape[0] * this->_shape[1] * sizeof(Dtype));
		}*/
		if (status != hipSuccess) {
			fprintf(stderr, "!!!! device memory allocation error\n");
			exit(EXIT_FAILURE);
		}
	} 
}


template <typename Dtype>
void Matrix<Dtype>::getTranspose(Matrix<Dtype>* target){
	
	const int width = this->_shape[1];
	const int height = this->_shape[0];
	const int num_blocks_x = DIVUP(width, ADD_BLOCK_SIZE);
	assert(num_blocks_x < NUM_BLOCKS_MAX);
	const int num_blocks_y = max(1, min(DIVUP(height, ADD_BLOCK_SIZE), \
				NUM_BLOCKS_MAX));
	dim3 grid_size(num_blocks_x, num_blocks_y, 1); 
	dim3 block_size(ADD_BLOCK_SIZE, ADD_BLOCK_SIZE, 1); 
	
	kTranspose<Dtype><<<grid_size, block_size>>>(this->_data_value, \
				target->getDevData(), width, height);
	hipDeviceSynchronize();
	cudaCheckError();
}

template <typename Dtype>
void Matrix<Dtype>::rightMult(Matrix<Dtype>* b, float scale_AB, \
		Matrix<Dtype> *target, hipblasHandle_t& handle) {

	clock_t t = clock();

	int m = this->_shape[0];
	int k = this->_shape[1];
	int n = b->getNumCols();
	float scale_tar = 0;
	assert(k == b->getNumRows());
	//列主
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &scale_AB, \
				b->getDevData(), n, this->_data_value, k, \
				&scale_tar, target->getDevData(), n);
}

template <typename Dtype>
void Matrix<Dtype>::addColVector(Matrix<Dtype>* vec){
	addColVector(vec, 1, this);
}

template <typename Dtype>
void Matrix<Dtype>::addColVector(Matrix<Dtype>* vec, float scaleVec, Matrix<Dtype>* target){

	Matrix<Dtype>* ori_trans = new Matrix(this->_shape[1], this->_shape[0]);
	this->getTranspose(ori_trans);
	ori_trans->addRowVector(vec);
	ori_trans->getTranspose(target);
	delete ori_trans;
}

template <typename Dtype>
void Matrix<Dtype>::addRowVector(Matrix<Dtype>* vec){
	addRowVector(vec, 1, this);	
}

template <typename Dtype>
void Matrix<Dtype>::addRowVector(Matrix<Dtype>* vec, float scaleVec, Matrix<Dtype>* target){
	assert(vec->getNumRows() == 1 || vec->getNumCols() == 1);
	assert(vec->getNumRows() == this->_shape[0] || vec->getNumCols() == this->_shape[1]);
	const int width = this->_shape[1];
	const int height = this->_shape[0];

	//表达成了矩阵的结构，就分开处理算了,block和thread的x维控制列数
	const int num_blocks_x = DIVUP(width, ADD_BLOCK_SIZE);
	assert(num_blocks_x < NUM_BLOCKS_MAX);
	const int num_blocks_y = max(1, min(DIVUP(height, ADD_BLOCK_SIZE), \
				NUM_BLOCKS_MAX));
	dim3 grid_size(num_blocks_x, num_blocks_y, 1); 
	dim3 block_size(ADD_BLOCK_SIZE, ADD_BLOCK_SIZE, 1); 

	kAddRowVector<Dtype><<<grid_size, block_size>>>(this->_data_value, vec->getDevData(), \
			target->getDevData(), width, height, scaleVec);
	hipDeviceSynchronize();
	cudaCheckError();
	
}

template <typename Dtype>
void Matrix<Dtype>::subtractFromScalar(float scalar, Matrix<Dtype>* target) { 

	const int width = this->_shape[1];
	const int height = this->_shape[0];
	const int num_blocks_x = DIVUP(width, ADD_BLOCK_SIZE);
	assert(num_blocks_x < NUM_BLOCKS_MAX);
	const int num_blocks_y = max(1, min(DIVUP(height, ADD_BLOCK_SIZE), \
				NUM_BLOCKS_MAX));
	dim3 grid_size(num_blocks_x, num_blocks_y, 1); 
	dim3 block_size(ADD_BLOCK_SIZE, ADD_BLOCK_SIZE, 1); 
	
	kSubtractFromScalar<Dtype><<<grid_size, block_size>>>(this->_data_value, scalar, \
			target->getDevData(), width, height);
	hipDeviceSynchronize();
	cudaCheckError();
}

template <typename Dtype>
void Matrix<Dtype>::subtractFromScalar(float scalar) {
	subtractFromScalar(scalar, this);
}

template <typename Dtype>
void Matrix<Dtype>::apply(Matrix<Dtype>::FUNCTIONS f, Matrix<Dtype> *target){
	
	const int width = this->_shape[1];
	const int height = this->_shape[0];
	const int num_blocks_x = DIVUP(width, ADD_BLOCK_SIZE);
	assert(num_blocks_x < NUM_BLOCKS_MAX);
	const int num_blocks_y = max(1, min(DIVUP(height, ADD_BLOCK_SIZE), \
				NUM_BLOCKS_MAX));
	dim3 grid_size(num_blocks_x, num_blocks_y, 1); 
	dim3 block_size(ADD_BLOCK_SIZE, ADD_BLOCK_SIZE, 1); 

	if(f == Matrix<Dtype>::SOFTMAX){
		//一个block只计算一行数据
		grid_size = dim3(1, height, 1);
		block_size = dim3(num_blocks_x * ADD_BLOCK_SIZE, 1, 1);
		kSoftmax<Dtype><<<grid_size, block_size, sizeof(Dtype) * width>>>(this->_data_value, \
				target->getDevData(), this->_shape[1], this->_shape[0]);
	}else if(f == Matrix<Dtype>::RECIPROCAL) {
		kReciprocal<Dtype><<<grid_size, block_size>>>(this->_data_value, target->getDevData(), \
				width, height);
	}else if(f == Matrix<Dtype>::LOG) {
		kLog<Dtype><<<grid_size, block_size>>>(this->_data_value, target->getDevData(), \
				width, height);
	}else if(f == Matrix<Dtype>::SIGMOID) {
		kSigmoid<Dtype><<<grid_size, block_size>>>(this->_data_value, target->getDevData(), \
				width, height);
	}
	hipDeviceSynchronize();
	cudaCheckError();
}

template <typename Dtype>
void Matrix<Dtype>::applyRelu(Matrix<Dtype> *target, Matrix<int>* record, \
		bool direction){
	const int width = this->_shape[1];
	const int height = this->_shape[0];
	const int num_blocks_x = DIVUP(width, ADD_BLOCK_SIZE);
	assert(num_blocks_x < NUM_BLOCKS_MAX);
	const int num_blocks_y = max(1, min(DIVUP(height, ADD_BLOCK_SIZE), \
				NUM_BLOCKS_MAX));
	dim3 grid_size(num_blocks_x, num_blocks_y, 1); 
	dim3 block_size(ADD_BLOCK_SIZE, ADD_BLOCK_SIZE, 1);

	if(direction)
		kRelu<Dtype><<<grid_size, block_size>>>(this->_data_value, \
				target->getDevData(), record->getDevData(), width, height);	
	else
		kReluBack<Dtype><<<grid_size, block_size>>>(this->_data_value, \
				target->getDevData(), record->getDevData(), width, height);	
	hipDeviceSynchronize();
	cudaCheckError();
}

template <typename Dtype>
void Matrix<Dtype>::applyDropout(Matrix<Dtype> *target, Matrix<int>* record, \
		Matrix<hiprandState>* rand_probs, bool is_set_up){

	const int width = this->_shape[1];
	const int height = this->_shape[0];
	const int num_blocks_x = DIVUP(width, ADD_BLOCK_SIZE);
	assert(num_blocks_x < NUM_BLOCKS_MAX);
	const int num_blocks_y = max(1, min(DIVUP(height, ADD_BLOCK_SIZE), \
				NUM_BLOCKS_MAX));
	dim3 grid_size(num_blocks_x, num_blocks_y, 1); 
	dim3 block_size(ADD_BLOCK_SIZE, ADD_BLOCK_SIZE, 1);

	if(is_set_up == false){
		kSetUpCurand<Dtype><<<grid_size, block_size>>>(rand_probs->getDevData(), \
				width, height);	
		hipDeviceSynchronize();
		cudaCheckError();
	
	}

	kDropout<Dtype><<<grid_size, block_size>>>(this->_data_value, \
			target->getDevData(), record->getDevData(), \
			rand_probs->getDevData(), width, height);	
	hipDeviceSynchronize();
	cudaCheckError();
}

template <typename Dtype>
void Matrix<Dtype>::apply(Matrix<Dtype>::FUNCTIONS f) {
	apply(f, this);
}

template <typename Dtype>
void Matrix<Dtype>::sumCol(Matrix<Dtype>* target){
	const int width = this->_shape[1];
	const int height = this->_shape[0];

	kDumbSumCols<Dtype><<<height, 1024, sizeof(Dtype) * width>>>(this->_data_value, \
			target->getDevData(), width, height);
	hipDeviceSynchronize();
	cudaCheckError();
}

template <typename Dtype>
void Matrix<Dtype>::sumRow(Matrix<Dtype>* target){
	Matrix<Dtype>* trans = new Matrix(this->_shape[1], this->_shape[0]);
	this->getTranspose(trans);
	trans->sumCol(target);
	delete trans;
}

//位置下标从0开始
template <typename Dtype>
void Matrix<Dtype>::maxPosInRow(Matrix<Dtype>* maxVec){
	const int width = this->_shape[1];
	const int height = this->_shape[0];
	const int num_blocks_x = DIVUP(width, ADD_BLOCK_SIZE);
	assert(num_blocks_x < NUM_BLOCKS_MAX);
	dim3 grid_size(1, height, 1); 
	dim3 block_size(num_blocks_x * ADD_BLOCK_SIZE, 1, 1); 

	kDumbMaxPosInRow<Dtype><<<grid_size, block_size, \
			sizeof(Dtype) * width>>>(this->_data_value, \
			maxVec->getDevData(), width, height);
	hipDeviceSynchronize();
	cudaCheckError();
}

template <typename Dtype>
void Matrix<Dtype>::eltWiseMult(Matrix<Dtype>* b, Matrix<Dtype>* target) {

	assert(b->getNumCols() == this->_shape[1]);

	const int width = this->_shape[1];
	const int height = this->_shape[0];
	const int num_blocks_x = DIVUP(width, ADD_BLOCK_SIZE);
	assert(num_blocks_x < NUM_BLOCKS_MAX);
	const int num_blocks_y = max(1, min(DIVUP(height, ADD_BLOCK_SIZE), \
				NUM_BLOCKS_MAX));
	dim3 grid_size(num_blocks_x, num_blocks_y, 1); 
	dim3 block_size(ADD_BLOCK_SIZE, ADD_BLOCK_SIZE, 1); 

	kMult<Dtype><<<grid_size, block_size>>>(this->_data_value, \
			b->getDevData(), target->getDevData(), width, height);
	hipDeviceSynchronize();
	cudaCheckError();
}

template <typename Dtype>
void Matrix<Dtype>::eltWiseMult(Matrix<Dtype>* b) {
	eltWiseMult(b, this);
}

template <typename Dtype>
void Matrix<Dtype>::addSum(Matrix<Dtype>* b, Matrix<Dtype>* c, float scaleThis, \
		float scaleB, float scaleC){
	this->add(b, scaleThis, scaleB);	
	this->add(c, 1, scaleC);	
}

template <typename Dtype>
void Matrix<Dtype>::add(Matrix<Dtype>* b, float scale_this, float scale_B){
	assert(this->isSameDims(b));
	const int width = this->_shape[1];
	const int height = this->_shape[0];
	const int num_blocks_x = DIVUP(width, ADD_BLOCK_SIZE);
	assert(num_blocks_x < NUM_BLOCKS_MAX);
	const int num_blocks_y = max(1, min(DIVUP(height, ADD_BLOCK_SIZE), \
				NUM_BLOCKS_MAX));
	dim3 grid_size(num_blocks_x, num_blocks_y, 1); 
	dim3 block_size(ADD_BLOCK_SIZE, ADD_BLOCK_SIZE, 1); 
	
	kAdd<Dtype><<<grid_size, block_size>>>(this->getDevData(), b->getDevData(), \
			this->getDevData(), scale_this, scale_B, width, height);
	hipDeviceSynchronize();
	cudaCheckError();
}


template <typename Dtype>
void Matrix<Dtype>::showValue(string name){

	Dtype* tmp_yh = new Dtype[this->_amount];
	this->copyToHost(tmp_yh, this->_amount);
	cout << "-------------"<< name << "--------------" << endl;
	cout << this->_shape[0] << ":" << this->_shape[1] << endl;
	for(int i = 0; i < this->_shape[0]; i++){
		for(int j = 0; j < this->_shape[1]; j++){
			cout << tmp_yh[i * this->_shape[1] + j] << "\t";
			if(j != 0 && j % (this->_shape[1]) == this->_shape[1]  - 1)
				cout << endl;
			if(this->_shape[1] == 1)
				cout << endl;
		}
	}
	delete[] tmp_yh;
}

template <typename Dtype>
void Matrix<Dtype>::reValue(float value){
	int length = this->getNumRows() * this->getNumCols();
	Dtype* tmp_yh = new Dtype[length];
	for(int i = 0; i < length; i++){
		tmp_yh[i] = value;
	}
	this->copyFromHost(tmp_yh, length);
	delete[] tmp_yh;
}

template <typename Dtype>
void Matrix<Dtype>::reValue(int value){
	int length = this->getNumRows() * this->getNumCols();
	Dtype* tmp_yh = new Dtype[length];
	for(int i = 0; i < length; i++){
		tmp_yh[i] = i % value;
	}
	this->copyFromHost(tmp_yh, length);
	delete[] tmp_yh;
}

template <typename Dtype>
Dtype Matrix<Dtype>::computeNorm(int len){
	Dtype norm_cpu;
	Matrix<Dtype>* norm_gpu = new Matrix<Dtype>(1, 1);
	kComputeNorm<<<1, 1024, sizeof(Dtype)*len>>>(this->_data_value, \
			norm_gpu->getDevData(), len);
	hipDeviceSynchronize();
	cudaCheckError();
	norm_gpu->copyToHost(&norm_cpu, 1);
	delete norm_gpu;
	return norm_cpu;
}

template <typename Dtype>
void Matrix<Dtype>::cropMatToNew(Matrix<Dtype> *tar, const int row_start, \
		const int cropped_height, const int col_start, const int cropped_width){
	kCropImg<<<1, 1024>>>(this->_data_value, tar->getDevData(), row_start, \
			cropped_height, col_start, cropped_width, this->_shape[1]);
	hipDeviceSynchronize();
	cudaCheckError();
}

template <typename Dtype>
Dtype Matrix<Dtype>::getPosValue(int pos){
	Dtype tmp;
	hipMemcpy(&tmp, this->_data_value + pos, sizeof(Dtype), hipMemcpyDeviceToHost);
	return tmp;
}

template <typename Dtype>
Dtype Matrix<Dtype>::getFirstPosValue(){
	return getPosValue(0);
}

template <typename Dtype>
void Matrix<Dtype>::subedByUnitMat(){

	const int width = this->_shape[1];
	const int height = this->_shape[0];
	const int num_blocks_x = DIVUP(width, ADD_BLOCK_SIZE);
	assert(num_blocks_x < NUM_BLOCKS_MAX);
	const int num_blocks_y = max(1, min(DIVUP(height, ADD_BLOCK_SIZE), \
				NUM_BLOCKS_MAX));
	dim3 grid_size(num_blocks_x, num_blocks_y, 1);
	dim3 block_size(ADD_BLOCK_SIZE, ADD_BLOCK_SIZE, 1);

	kSubedByUnitMat<Dtype><<<grid_size, block_size>>>(this->getDevData(), \
			this->getDevData(), width, height);
	hipDeviceSynchronize();
	cudaCheckError();
}

template <typename Dtype>
void Matrix<Dtype>::setValueAt(const int height_idx, \
		const int width_idx, const Dtype value){
	int pos = height_idx*this->_shape[1] + width_idx;
	hipMemcpy(this->_data_value + pos, &value, sizeof(Dtype), \
			hipMemcpyHostToDevice);
}

template <typename Dtype>
void Matrix<Dtype>::subPortion(Matrix<Dtype>* b, const int b_row, \
			const int b_col){

	const int width = b_col;
	const int height = b_row;
	const int num_blocks_x = DIVUP(width, ADD_BLOCK_SIZE);
	assert(num_blocks_x < NUM_BLOCKS_MAX);
	const int num_blocks_y = max(1, min(DIVUP(height, ADD_BLOCK_SIZE), \
				NUM_BLOCKS_MAX));
	dim3 grid_size(num_blocks_x, num_blocks_y, 1);
	dim3 block_size(ADD_BLOCK_SIZE, ADD_BLOCK_SIZE, 1);

	kSubPortion<Dtype><<<grid_size, block_size>>>(this->getDevData(), \
			b->getDevData()+b_col, this->getDevData(), this->_shape[1], \
			this->_shape[0], width, height);
	hipDeviceSynchronize();
	cudaCheckError();
}

template <typename Dtype>
void Matrix<Dtype>::readPars(string filename){
	ifstream fin1(filename.c_str(), ios::binary);
	int dataLen = this->getNumRows() * this->getNumCols();
	Dtype* tmp = new Dtype[dataLen];
	fin1.read((char*)(tmp), sizeof(Dtype) * dataLen);
	hipMemcpy(this->getDevData(), tmp, sizeof(Dtype)*dataLen, \
				hipMemcpyHostToDevice);
	fin1.close();
	delete tmp;
}

template <typename Dtype>
void Matrix<Dtype>::savePars(string filename){
	ofstream fout(filename.c_str(), ios::binary);
	int dataLen = this->getNumRows() * this->getNumCols();
	Dtype* tmp = new Dtype[dataLen];
	hipMemcpy(tmp, this->getDevData(), sizeof(Dtype)*dataLen, \
				hipMemcpyDeviceToHost);
	fout.write((char*)(tmp), sizeof(Dtype) * dataLen);
	fout.close();
	delete tmp;
}










