#include "hip/hip_runtime.h"
/*
 * filename:testMultiGpu.cu
 */

#include <iostream>
#include <fstream>
#include <time.h>
#include <cmath>
#include <pthread.h>
#include "mpi.h"
#include "utils.h"
#include "matrix.h"
#include "nvmatrix.cuh"
#include "convnet.cuh"
#include "convnet_kernel.cuh"

using namespace std;

enum swapInfo{SWAP_AVGOUT_PUSH, \
	SWAP_AVGOUT_FETCH, \
		SWAP_BIAS_PUSH, \
		SWAP_BIAS_FETCH};

typedef struct _ThreadControlMSG{
	int sendPid;
	int recvPid;
	//传递batchIdx
	int myState;
	enum swapInfo mySwap; 
	float* data;
	bool isMoveDataPos;
	bool isSender;
	int transLen;
} ThreadControlMSG, *pThreadControlMSG;

int rank;
int numProcess;
MPI_Comm newComm[10];

pthread_mutex_t mutexS;
#define THREAD_END 10000

void* watchState(void* msg){
	pThreadControlMSG myMsg = (pThreadControlMSG)msg;
	float* myData = myMsg->data;
	MPI_Request req;
	MPI_Status status;
	int numCompute = -1;
	while(myMsg->myState != THREAD_END){
		if(myMsg->isSender)
			MPI_Recv(&myMsg->myState, 1, MPI_INT, myMsg->recvPid, \
					myMsg->mySwap*100 + (numCompute+1)*(numProcess-1), \
					newComm[myMsg->recvPid], &status);
		else
			MPI_Recv(&myMsg->myState, 1, MPI_INT, myMsg->sendPid, \
					myMsg->mySwap*100 + (numCompute+1)*(numProcess-1), \
					newComm[myMsg->recvPid], &status);
		//			cout <<"send:" << myMsg->mySwap*100 + (numCompute+1) * (numProcess - 1) \
		<< endl;
		if(myMsg->isSender){
			//			pthread_mutex_lock(&mutexS);
			MPI_Send(myData, myMsg->transLen, MPI_FLOAT, myMsg->recvPid, \
					myMsg->mySwap*200 + (numCompute+1) * (numProcess-1), \
					newComm[myMsg->recvPid]);
			//			pthread_mutex_unlock(&mutexS);
		}
		else{
			//			pthread_mutex_lock(&mutexS);
			MPI_Recv(myData, myMsg->transLen, MPI_FLOAT, myMsg->sendPid, \
					myMsg->mySwap*200 + (numCompute+1)*(numProcess-1), \
					newComm[myMsg->sendPid], &status);
			//			pthread_mutex_unlock(&mutexS);
		}
		//		cout <<"send data:" << myMsg->mySwap*200 + (numCompute+1)*(numProcess-1) \
		<< endl;
		numCompute = myMsg->myState;
	}
	pthread_exit(0);
}


//默认创建线程的为发送方
void createAndRun(pthread_t* tid, pThreadControlMSG tMSG, float* data, \
		const int transLen, enum swapInfo mySwap, \
		bool isSender = true){

	for(int i = 0; i < numProcess - 1; i++){
		if(isSender){
			tMSG[i].sendPid = 0;
			tMSG[i].recvPid = i + 1;
			tMSG[i].isSender = true;
		}else{
			tMSG[i].sendPid = i + 1;
			tMSG[i].recvPid = 0;
			tMSG[i].isSender = false;
		}
		tMSG[i].data = data;
		tMSG[i].mySwap = mySwap;
		tMSG[i].transLen = transLen;
		int error = pthread_create(&tid[i], NULL, \
				watchState, (void*)&tMSG[i]);
		if(error){
			cout << "Error - pthread_create() return code: " << error << endl;
			exit(EXIT_FAILURE);
		}
	}
}

void managerNode(pars* logistic){
	
	

	int inSqrt = logistic->inSize * logistic->inSize;
	int hidVisLen = logistic->numFilters * logistic->filterSize \
					* logistic->filterSize;
	//	int hidBiasLen = numFilters;
	int avgOutLen = inSqrt * logistic->numOut;
	int outBiasLen = logistic->numOut;

	int proTrainDataLen = logistic->trainNum * inSqrt / (numProcess - 1);
	int proTrainLabelLen = logistic->trainNum / (numProcess - 1);
	int proValidDataLen = logistic->validNum * inSqrt / (numProcess - 1);
	int proValidLabelLen = logistic->validNum / (numProcess - 1);

	NVMatrix* nvTrainData = new NVMatrix(logistic->trainNum, inSqrt,  
			NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvValidData = new NVMatrix(logistic->validNum, inSqrt, \
			NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvTrainLabel = new NVMatrix(logistic->trainNum, 1, \
			NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvValidLabel = new NVMatrix(logistic->validNum, 1, \
			NVMatrix::ALLOC_ON_UNIFIED_MEMORY);

	NVMatrix* avgOut = new NVMatrix(inSqrt, logistic->numOut, \
			NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* outBiases = new NVMatrix(1, logistic->numOut, \
			NVMatrix::ALLOC_ON_UNIFIED_MEMORY);

	cout << "=========================\n" \
		<< "train: " << logistic->trainNum \
		<< "\nvalid: " << logistic->validNum \
		<< "\nfiltersize: " << logistic->filterSize \
		<< "\nnumFilters: " << logistic->numFilters \
		<< "\nepsHidVis: " << logistic->epsHidVis \
		<< "\nepsHidBias: " << logistic->epsHidBias \
		<< "\nepsAvgOut: " << logistic->epsAvgOut \
		<< "\nepsOutBias: " << logistic->epsOutBias \
		<< "\nmom: " << logistic->mom \
		<< "\nwcHidVis: " << logistic->wcHidVis \
		<< "\nwcAvgOut: " << logistic->wcAvgOut \
		<< "\n========================" << endl;
	//0号进程来读取输入数据
	readData(nvTrainData, "../data/input/mnist_train.bin", true);
	readData(nvValidData, "../data/input/mnist_valid.bin", true);
	readData(nvTrainLabel, "../data/input/mnist_label_train.bin", false);
	readData(nvValidLabel, "../data/input/mnist_label_valid.bin", false);

	Matrix* hHidVis = new Matrix(logistic->numFilters, logistic->filterSize \
			* logistic->filterSize);
	Matrix* hHidBiases = new Matrix(logistic->numFilters, 1);
	Matrix* hAvgout = new Matrix(inSqrt, logistic->numOut);
	Matrix* hOutBiases = new Matrix(1, logistic->numOut);

	//0号进程初始化参数，进行分发
	initW(hHidVis->getData(), hidVisLen);
	memset(hHidBiases->getData(), 0, sizeof(float) * logistic->numFilters);
	memset(hAvgout->getData(), 0, sizeof(float) * avgOutLen);
	memset(hOutBiases->getData(), 0, sizeof(float) * logistic->numOut);
	//	readPars(hHidVis, "hHidVis_t1.bin");
	//	readPars(hHidBiases, "hHidBiases_t1.bin");
	//	readPars(hAvgout, "hAvgout_t1.bin");
	//	readPars(hOutBiases, "hOutBiases_t1.bin");

	MPI_Bcast(hAvgout->getData(), avgOutLen, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(hOutBiases->getData(), outBiasLen, MPI_FLOAT, 0, MPI_COMM_WORLD);

	//直接将input数据scatter给剩下的线程。让它们自己移动，用循环处理
	for(int i = 1; i < numProcess; i++){
		MPI_Send(nvTrainData->getDevData()+(i-1)*proTrainDataLen, proTrainDataLen, \
				MPI_FLOAT, i, i, MPI_COMM_WORLD);
		MPI_Send(nvTrainLabel->getDevData()+(i-1)*proTrainLabelLen, \
				proTrainLabelLen, MPI_FLOAT, i, i, MPI_COMM_WORLD);
		MPI_Send(nvValidData->getDevData()+(i-1)*proValidDataLen, proValidDataLen, \
				MPI_FLOAT, i, i, MPI_COMM_WORLD);
		MPI_Send(nvValidLabel->getDevData()+(i-1)*proValidLabelLen, \
				proValidLabelLen, MPI_FLOAT, i, i, MPI_COMM_WORLD);
	}


	pthread_mutex_init(&mutexS, NULL);
	int openTimes = 4;
	pthread_t openThread[openTimes * (numProcess - 1)];
	pThreadControlMSG tMSG = new ThreadControlMSG[openTimes * (numProcess - 1)];


	//接收更新的参数
	createAndRun(openThread, tMSG, avgOut->getDevData(), \
			avgOutLen, SWAP_AVGOUT_PUSH, false);	
	createAndRun(openThread + (numProcess - 1), tMSG + (numProcess - 1), \
			outBiases->getDevData(), outBiasLen, SWAP_BIAS_PUSH, false);	
	//发送参数
	createAndRun(openThread + (numProcess - 1) * 2, tMSG + (numProcess - 1) * 2, \
			avgOut->getDevData(), avgOutLen, SWAP_AVGOUT_FETCH);	
	createAndRun(openThread + (numProcess - 1) * 3, tMSG + (numProcess - 1) * 3, \
			outBiases->getDevData(), outBiasLen, SWAP_BIAS_FETCH);	

	for(int i = 0; i < openTimes * (numProcess - 1); i++){
		pthread_join(openThread[i], NULL);
	}

	//				savePars(hHidVis, "../data/pars/hHidVis_t1.bin");
	//  			savePars(hHidBiases, "../data/pars/hHidBiases_t1.bin");
	//				savePars(hAvgout, "../data/pars/hAvgout_t1.bin");
	//				savePars(hOutBiases, "../data/pars/hOutBiases_t1.bin");	
	int errorValid = 0;
	int totalValid;
	MPI_Reduce(&errorValid, &totalValid, 1, MPI_INT, MPI_SUM, \
			1, MPI_COMM_WORLD);

	delete[] tMSG;
	delete nvTrainData;
	delete nvTrainLabel;
	delete nvValidData;
	delete nvValidLabel;
	delete avgOut;
	delete outBiases;

	delete hHidVis;
	delete hHidBiases;
	delete hAvgout;
	delete hOutBiases;

	pthread_mutex_destroy(&mutexS);
}

void workerNode(pars* logistic){
	int inSqrt = logistic->inSize * logistic->inSize;

	//int hidVisLen = logistic->numFilters * logistic->filterSize \
	* logistic->filterSize;
	//	int hidBiasLen = numFilters;
	int avgOutLen = inSqrt * logistic->numOut;
	int outBiasLen = logistic->numOut;

	int miniDataLen = logistic->minibatchSize * inSqrt;
	int miniLabelLen = logistic->minibatchSize;

	int proTrainDataLen = logistic->trainNum * inSqrt / (numProcess - 1);
	int proTrainLabelLen = logistic->trainNum / (numProcess - 1);
	int proValidDataLen = logistic->validNum * inSqrt / (numProcess - 1);
	int proValidLabelLen = logistic->validNum / (numProcess - 1);

	NVMatrix* avgOut;
	NVMatrix* outBiases;

	NVMatrix* nvTrainData = new NVMatrix(logistic->trainNum / (numProcess - 1), \
			inSqrt, NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvValidData = new NVMatrix(logistic->validNum / (numProcess - 1), \
			inSqrt, NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvTrainLabel = new NVMatrix(logistic->trainNum / (numProcess - 1), 1, \
			NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvValidLabel = new NVMatrix(logistic->validNum / (numProcess - 1), 1, \
			NVMatrix::ALLOC_ON_UNIFIED_MEMORY);

	NVMatrix* miniData = new NVMatrix(nvTrainData->getDevData(), \
			logistic->minibatchSize, inSqrt);
	NVMatrix* miniLabel = new NVMatrix(nvTrainLabel->getDevData(), \
			logistic->minibatchSize, 1);

	Matrix* hHidVis = new Matrix(logistic->numFilters, logistic->filterSize \
			* logistic->filterSize);
	Matrix* hHidBiases = new Matrix(logistic->numFilters, 1);
	Matrix* hAvgout = new Matrix(inSqrt, logistic->numOut);
	Matrix* hOutBiases = new Matrix(1, logistic->numOut);

	MPI_Bcast(hAvgout->getData(), avgOutLen, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(hOutBiases->getData(), outBiasLen, MPI_FLOAT, 0, MPI_COMM_WORLD);

	ConvNet layer1(hHidVis, hAvgout, hHidBiases, hOutBiases, logistic);
	layer1.initCuda();
	//	double loglihood = 0;

	MPI_Request reqs;
	MPI_Status status;

	MPI_Recv(nvTrainData->getDevData(), proTrainDataLen, \
			MPI_FLOAT, 0, rank, MPI_COMM_WORLD, &status);
	MPI_Recv(nvTrainLabel->getDevData(), proTrainLabelLen, \
			MPI_FLOAT, 0, rank, MPI_COMM_WORLD, &status);
	MPI_Recv(nvValidData->getDevData(), proValidDataLen, \
			MPI_FLOAT, 0, rank, MPI_COMM_WORLD, &status);
	MPI_Recv(nvValidLabel->getDevData(), proValidLabelLen, \
			MPI_FLOAT, 0, rank, MPI_COMM_WORLD, &status);


	int passMsg;	
	char* myBuffer = new char[logistic->numOut * ( 1+inSqrt ) * sizeof(float) * 50];
	int detachOut;
	int detachS;
	MPI_Buffer_attach(myBuffer, logistic->numOut * ( 1+inSqrt) * sizeof(float) * 50);
	cout << "create thread\n";

	clock_t t;
	t = clock();
	for(int epochIdx = 0; epochIdx < logistic->numEpoches; epochIdx++){
		int error = 0;	

		for(int batchIdx = 0; batchIdx < logistic->numMinibatches/(numProcess-1); \
				batchIdx++){
			if(batchIdx == logistic->numMinibatches - 1){ 
				if(epochIdx == logistic->numEpoches - 1)
					passMsg = THREAD_END;    
				else
					passMsg = -1;
			}
			else
				passMsg = batchIdx;
cout << rank<< ":done1\n";
			nvTrainData->slice(miniData->getDevData(), batchIdx * miniDataLen);
			nvTrainLabel->slice(miniLabel->getDevData(), batchIdx * miniLabelLen);

cout << rank<< ":done2\n";
			layer1.computeLogistic(miniData, miniLabel, true);
			layer1.computeError(miniLabel, error);

			avgOut = layer1.getAvgOut();
			outBiases = layer1.getOutBias();

cout << rank<< ":done3\n";
			if((batchIdx + 1) % logistic->nPush == 0){
				MPI_Send(&passMsg, 1, MPI_INT, 0, \
						SWAP_AVGOUT_PUSH*100 + batchIdx*(numProcess-1), \
						newComm[rank]);

				MPI_Send((rank-1)*avgOutLen/(numProcess-1) \
						+ avgOut->getDevData(), avgOutLen/(numProcess-1), \
						MPI_FLOAT, 0, SWAP_AVGOUT_PUSH * 200 \
						+ batchIdx * (numProcess - 1), \
						newComm[rank]);

				MPI_Send(&passMsg, 1, MPI_INT, 0, \
						SWAP_BIAS_PUSH*100 + batchIdx*(numProcess-1), \
						newComm[rank]);
				MPI_Send((rank-1)*outBiasLen/(numProcess-1) \
						+ outBiases->getDevData(), outBiasLen/(numProcess-1), \
						MPI_FLOAT, 0, SWAP_BIAS_PUSH*200 + batchIdx*(numProcess-1), \
						newComm[rank]);
			}

cout << rank<< ":done4\n";
			if((batchIdx + 1) % logistic->nFetch == 0){
				MPI_Send(&passMsg, 1, MPI_INT, 0, \
						SWAP_AVGOUT_FETCH*100+batchIdx*(numProcess-1), \
						newComm[rank]);
				MPI_Recv(avgOut->getDevData(), avgOutLen, MPI_FLOAT, 0, \
						SWAP_AVGOUT_FETCH*200+batchIdx*(numProcess-1), \
						newComm[rank], &status);

				MPI_Send(&passMsg, 1, MPI_INT, 0, \
						SWAP_BIAS_FETCH*100+batchIdx*(numProcess-1), \
						newComm[rank]);
				MPI_Recv(outBiases->getDevData(), outBiasLen, MPI_FLOAT, \
						0, SWAP_BIAS_FETCH*200+batchIdx*(numProcess-1), \
						newComm[rank], &status);
			}

cout << rank<< ":done5\n";
			if(batchIdx == logistic->numMinibatches - 1){
				int errorValid = 0;
				float loglihoodValid = 0;
				for(int validIdx = 0; validIdx < logistic->numValidBatches / (numProcess - 1); validIdx++){

					nvValidData->slice(miniData->getDevData(), validIdx * miniDataLen);
					nvValidLabel->slice(miniLabel->getDevData(), validIdx * miniLabelLen);

					layer1.computeLogistic(miniData, miniLabel, false);
					loglihoodValid += layer1.computeError(miniLabel, \
							errorValid);
				}
				int totalValid;
				MPI_Reduce(&errorValid, &totalValid, 1, MPI_INT, MPI_SUM, \
						1, MPI_COMM_WORLD);
				if(rank == 1){
					cout << "epochIdx: " << epochIdx << ",error: " \
						<< ((float)errorValid*(numProcess-1))/logistic->validNum << endl;
					//					<< ",likelihood: "<< loglihood<< endl;
				}
			}
		}

	}
	detachS = MPI_Buffer_detach(myBuffer, &detachOut);
	cout << (detachS == 0 ? "success\n" : "failed\n" )<< endl;

	if(rank == 1){
		t = clock() - t;
		cout << " " << ((float)t/CLOCKS_PER_SEC)/logistic->numEpoches << " seconds. \n";
	}

	delete nvTrainData;
	delete nvTrainLabel;
	delete nvValidData;
	delete nvValidLabel;
}

int main(int argc, char** argv){

	int prov;
	MPI_Init_thread(&argc,&argv,MPI_THREAD_MULTIPLE, &prov);
	if (prov < MPI_THREAD_MULTIPLE)
	{
		printf("Error: the MPI library doesn't provide the required thread level\n");
		MPI_Abort(MPI_COMM_WORLD, 0);
	}
	MPI_Comm_rank(MPI_COMM_WORLD,&rank);
	MPI_Comm_size(MPI_COMM_WORLD,&numProcess);

	//检测有几个gpu
	int numGpus;
	hipGetDeviceCount(&numGpus);
	hipSetDevice(rank%numGpus);

	pars* logistic = new pars;

	logistic->epsHidVis = 0.001;
	logistic->epsHidBias = 0.001;
	logistic->epsAvgOut = 0.13;
	logistic->epsOutBias = 0.13;
	logistic->mom = 0;
	logistic->wcHidVis = 0;
	logistic->wcAvgOut = 0;
	logistic->inSize = 28;
	logistic->inChannel = 1;
	logistic->filterSize = 5;
	logistic->numFilters = 16;
	logistic->numOut = 10;
	logistic->trainNum = 50000;
	logistic->validNum = 10000;
	logistic->minibatchSize = 1000;
	logistic->numMinibatches = logistic->trainNum / (logistic->minibatchSize \
			* (numProcess - 1));
	logistic->numValidBatches = logistic->validNum / (logistic->minibatchSize \
			* (numProcess - 1));
	logistic->numEpoches = 20; 
	logistic->nPush = 1;
	logistic->nFetch = 1;
	
cout << "start\n";
	if(rank == 0){
		for(int i = 1; i < numProcess; i++)
			MPI_Comm_split(MPI_COMM_WORLD, i, rank, &newComm[i]);
		cout << "split done1\n";
		managerNode(logistic);
	}
	else{
		MPI_Comm_split(MPI_COMM_WORLD, rank, rank, &newComm[rank]);
		cout << "split done1\n";
		workerNode(logistic);
	}

	delete logistic;
	for(int i = 1; i < numProcess; i++)
		MPI_Comm_free(&newComm[i]);
	MPI_Finalize();
	return 0;
}
