#include "hip/hip_runtime.h"
///
/// \file logistic.cu
///

#include "logistic.hpp"

using namespace std;

template <typename Dtype>
Logistic<Dtype>::Logistic<Dtype>(FullConnectParam* fcp) {
	this->_fcp = fcp;
	
}

template <typename Dtype>
Logistic<Dtype>::~Logistic<Dtype>() {

	delete this->_y;
	delete[] h_labels;
	delete[] y_CPU;
	delete[] correct_probs;
	delete d_max_pos_of_out;
	delete[] h_max_pos_of_out;
	delete _d_record;
	delete[] _h_record;
}

template <typename Dtype>
void Logistic<Dtype>::initCuda() {

	this->_y            = new Matrix<Dtype>(this->_fcp->getMinibatchSize(), \
								this->_fcp->getNumOut());
	h_labels 			= new Dtype[this->_fcp->getMinibatchSize()];
	y_CPU 				= new Dtype[this->_y->getNumEles()];
	correct_probs 		= new Dtype[this->_y->getNumRows()];
	d_max_pos_of_out 	= new Matrix<Dtype>(this->_y->getNumRows(), 1);
	h_max_pos_of_out 	= new Dtype[this->_y->getNumRows()];

	_d_record 		= new Matrix<int>(this->_y->getNumCols(), this->_y->getNumCols());
	_h_record 		= new int[this->_y->getNumCols() * this->_y->getNumCols()];
}

template <typename Dtype>
void Logistic<Dtype>::computeOutputs(Matrix<Dtype>* x){
//x->showValue("data");
	x->apply(Matrix<Dtype>::SOFTMAX, this->_y);
//this->_y->showValue("yj1");
}

template <typename Dtype>
double Logistic<Dtype>::computeError(Matrix<Dtype>* labels, int& num_error){

	/// h_labels大小是minibatch * 1
	labels->copyToHost(h_labels, labels->getNumEles());

	/// y_cpu大小是minibatch * 10
	this->_y->copyToHost(y_CPU, this->_y->getNumEles());

	/// 记录找打的最大位置上的likelihood
	/// 记录最大位置的下标
	this->_y->maxPosInRow(d_max_pos_of_out);
//d_max_pos_of_out->showValue("maxpos");
//this->_y->showValue("yj1");

	d_max_pos_of_out->copyToHost(h_max_pos_of_out, this->_y->getNumRows());

	for (int c = 0; c < this->_y->getNumRows(); c++) {
		int true_label = h_labels[c];
		int predict_label = h_max_pos_of_out[c];
		correct_probs[c] = log(y_CPU[c * this->_y->getNumCols() + true_label]);

		if(predict_label != true_label)
			num_error++;
		_h_record[predict_label * this->_y->getNumCols() + true_label]++ ;
	}
	double result = 0;
	for(int i = 0; i < labels->getNumEles(); i++){
		result -= correct_probs[i];
	}

	return result;
}

template <typename Dtype>
void Logistic<Dtype>::computeDerivsOfInput(Matrix<Dtype>* dE_dx, Matrix<Dtype>* labels){
	assert(labels->getNumRows() == dE_dx->getNumRows());

//this->_y->reValue(1.0f);
//labels->reValue(1.0f);

	const int num_thread = DIVUP(this->_fcp->getNumOut(), ADD_BLOCK_SIZE) * ADD_BLOCK_SIZE;
	compute_dE_dy<<<this->_fcp->getMinibatchSize(), num_thread>>>(this->_y->getDevData(), \
			labels->getDevData(), dE_dx->getDevData(), this->_fcp->getNumOut());


}



