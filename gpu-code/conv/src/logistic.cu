/*
 * filename: logistic.cu
 */
#include <time.h>
#include "logistic.cuh"
#include "logistic_kernel.cuh"

using namespace std;

Logistic::Logistic(Matrix* hAvgOut, Matrix* hOutBiases, pars* netWork){

	this->_numOut                = hAvgOut->getNumCols();

	this->_hAvgOut               = hAvgOut;
	this->_hOutBiases            = hOutBiases;

	//w_hk的learning rate
	this->_epsAvgOut             = netWork->epsAvgOut;
	//out bias learning rate
	this->_epsOutBias            = netWork->epsOutBias;
	//上一次更新的参数控制增长趋势
	this->_mom                   = netWork->mom;
	//hidden原值的参数
	this->_wcHidVis              = netWork->wcHidVis;
	//out原值的参数
	this->_wcAvgOut              = netWork->wcAvgOut;

	this->_minibatchSize         = netWork->minibatchSize;

	hipblasCreate(&handle);
}

Logistic::~Logistic() {
	delete _hAvgOut;
	delete _hOutBiases;

	delete _avgOut;
	delete _avgOutInc;
	delete _outBiases;
	delete _outBiasInc;

	delete  _y_j;
	delete  _dE_dy_j;
	delete _dE_db_j;
	delete _dE_dw_ij;
	hipblasDestroy(handle);
}

void Logistic::initCuda() {

	this->_avgOut            = new NVMatrix(_hAvgOut, true);
//					NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	this->_outBiases         = new NVMatrix(_hOutBiases, true);
//					NVMatrix::ALLOC_ON_UNIFIED_MEMORY);

	this->_y_j               = new NVMatrix(_minibatchSize, _numOut);

	this->_dE_dy_j           = new NVMatrix(_y_j);
	this->_dE_db_j           = new NVMatrix(_outBiases);
	this->_dE_dw_ij          = new NVMatrix(_avgOut);

	this->_avgOutInc         = new NVMatrix(_avgOut);
	this->_outBiasInc        = new NVMatrix(1, _numOut);
	this->_avgOutInc->zeros();
	this->_outBiasInc->zeros();
}

void Logistic::computeClassOutputs(NVMatrix* miniData){
	miniData->rightMult(_avgOut, 1, _y_j, handle);

	_y_j->addRowVector(_outBiases);

	_y_j->apply(NVMatrix::SOFTMAX);
}

double Logistic::computeError(const NVMatrix* const miniLabels, int& numError){

	Matrix* hlabels = new Matrix(miniLabels->getNumRows(), miniLabels->getNumCols());
	miniLabels->copyToHost(hlabels);
	Matrix* y_j_CPU = new Matrix(_y_j->getNumRows(), _y_j->getNumCols());
	_y_j->copyToHost(y_j_CPU);
	Matrix* correctProbs = new Matrix(_y_j->getNumRows(), 1);
	NVMatrix* maxPosOfOutGpu = new NVMatrix(_y_j->getNumRows(), 1);
	_y_j->maxPosInRow(maxPosOfOutGpu);
	Matrix* maxPosCpu = new Matrix(_y_j->getNumRows(), 1);
	maxPosOfOutGpu->copyToHost(maxPosCpu);
	for (int c = 0; c < _y_j->getNumRows(); c++) {
		int trueLabel = hlabels->getCell(c, 0);
		int predictLabel = maxPosCpu->getCell(c, 0);
		correctProbs->getCell(c, 0) = y_j_CPU->getCell(c, trueLabel);

		if(predictLabel != trueLabel)
			numError++;
	}
	correctProbs->apply(Matrix::LOG);
	double result = -correctProbs->sum();
	hipDeviceSynchronize();

	delete hlabels;
	delete y_j_CPU;
	delete correctProbs;
	delete maxPosOfOutGpu;
	delete maxPosCpu;
	return result;
}

void Logistic::computeDerivs(NVMatrix* miniData, NVMatrix* miniLabels){
	assert(miniLabels->getNumRows() == miniData->getNumRows());

	const int numThreads = DIVUP(_numOut, ADD_BLOCK_SIZE) * ADD_BLOCK_SIZE;
	compute_dE_dy<<<_minibatchSize, numThreads>>>(_y_j->getDevData(), \
			miniLabels->getDevData(), _dE_dy_j->getDevData(), _numOut);

	NVMatrix* data_T = new NVMatrix(miniData->getNumCols(), miniData->getNumRows());
	miniData->getTranspose(data_T);

	data_T->rightMult(_dE_dy_j, 1, _dE_dw_ij, handle);
	_dE_dy_j->sumRow(_dE_db_j);

	delete data_T;
}

void Logistic::updatePars(){
	_avgOutInc->addSum(_avgOut, _dE_dw_ij, _mom, -_wcAvgOut, \
			-_epsAvgOut / _minibatchSize);
	_avgOut->add(_avgOutInc, 1, 1);

	_outBiasInc->add(_dE_db_j, _mom, -_epsOutBias / _minibatchSize);
	_outBiases->add(_outBiasInc, 1, 1);
}



