#include "hip/hip_runtime.h"
/*
 * filename: convnet_kernel.cu
 */

#include <hip/hip_runtime.h>
#include "convnet_kernel.cuh"

//struct conv{
//	float* data;	
//};

__device__ float logistic(float x) {
	if(x < -300)
		return 0;
	else if( x > 300)
		return 1;
	else
		return 1 / (1 + __expf(-x));
}

__global__ void im2col_filt(const float* imgs, float* targets, \
		const int numKernels, const int widthNoChannel, const int width, \
		const int heightNoBatch){

	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < numKernels){
		int imgPixs = IMG_SIZE * IMG_SIZE;

		//此处的width指的是5*5*3,height指的是24*24
		const int channelIdx = (idx % width) % IMG_CHANNEL;
		//widthIdx指的是5*5的id
		const int widthIdx = (idx % width) / IMG_CHANNEL;
		const int filtRow = widthIdx / FILTER_SIZE;
		const int filtCol = widthIdx % FILTER_SIZE;
		const int imgIdx = (idx / width) / heightNoBatch;
		const int heightIdx = (idx / width) % heightNoBatch;
		const int convRow = heightIdx / CONV_FORWARD_SIZE;
		const int convCol = heightIdx % CONV_FORWARD_SIZE;
		//输入图片的位置
		imgs += imgIdx * IMG_CHANNEL * imgPixs + channelIdx * imgPixs \
				+ (convRow * CONV_STEP_SIZE + filtRow) * IMG_SIZE \
				+ (convCol * CONV_STEP_SIZE + filtCol); 
		//输出图片的位置
		targets[idx] = imgs[0];
	}
	__syncthreads();

}

__global__ void im2col_conv(const float* imgs, float* targets, \
		const int numKernels, const int widthNoBatch, const int widthNoChannel, \
		const int width, const int height){

	//行表示为minibatch*convsize*convsize*inchannel
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < numKernels){
		int imgPixs = IMG_SIZE * IMG_SIZE;
		//输入channel的值展开成行
		//此处的width指的是100*24*24,height指的是5*5
		const int imgIdx = (idx % width) / widthNoBatch;
		const int withChannelIdx = (idx % width) % widthNoBatch;
		const int channelIdx = withChannelIdx % IMG_CHANNEL;
		//widthIdx指的是conv*conv的id
		const int widthIdx = withChannelIdx / IMG_CHANNEL;
		const int convRow = widthIdx / CONV_FORWARD_SIZE;
		const int convCol = widthIdx % CONV_FORWARD_SIZE;
		const int heightIdx = (idx / width) % height;
		const int filtRow = heightIdx / FILTER_SIZE;
		const int filtCol = heightIdx % FILTER_SIZE;
		//输入图片的位置
		imgs += imgIdx * IMG_CHANNEL * imgPixs + channelIdx * imgPixs \
				+ (convRow * CONV_STEP_SIZE + filtRow) * IMG_SIZE \
				+ (convCol * CONV_STEP_SIZE + filtCol); 
		//输出图片的位置
		targets[idx] = imgs[0];
	}
	__syncthreads();

}

__global__ void reshape_y_h(const float* un_y_h, float* y_h, const int numKernels){

	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < numKernels){
		int convPixs = CONV_FORWARD_SIZE * CONV_FORWARD_SIZE;

		const int dstRow = idx / (convPixs * FILTER_CHANNEL);
		const int dstCol = idx % (convPixs * FILTER_CHANNEL);
		const int oriCol = dstCol / convPixs;
		const int oriRow = dstRow * convPixs + dstCol % convPixs;
		un_y_h += oriRow * FILTER_CHANNEL + oriCol;
		y_h[idx] = logistic(un_y_h[0]); 
	}

}

__global__ void reshape_dE_dx_h(float* un_dE_dx_h, const float* dE_dx_h, \
		const int numKernels){
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < numKernels){
		int convPixs = CONV_FORWARD_SIZE * CONV_FORWARD_SIZE;

		const int dstRow = idx / (convPixs * FILTER_CHANNEL);
		const int dstCol = idx % (convPixs * FILTER_CHANNEL);
		const int oriCol = dstCol / convPixs;
		const int oriRow = dstRow * convPixs + dstCol % convPixs;
		un_dE_dx_h += oriRow * FILTER_CHANNEL + oriCol;
		un_dE_dx_h[0] = dE_dx_h[idx]; 
	}
}


__global__ void convolution_forward(const float* imgs, const float* filters, \
		const float* biases, float* targets, const int filConvtimes, \
		const int imgConvtimes) {
	int imgPixs = IMG_SIZE * IMG_SIZE;
	int filPixs = FILTER_SIZE * FILTER_SIZE;
	int convPixs = CONV_FORWARD_SIZE * CONV_FORWARD_SIZE;
	//一个block计算一张图与一个featuremap的卷积，开辟了28个线程，每个线程算28个值
	//放在shared memory里面的数据，featuremap的参数是要共享的

	//	__shared__ struct conv tmp;
	//	__device__ float* value = new float[filPixs];
	//	tmp.data = value;
	__shared__ float shImg[IMG_SIZE][IMG_SIZE];
	__shared__ float shFilter[FILTER_SIZE][FILTER_SIZE];
	__shared__ float shBias;

	const int imgIdx = blockIdx.x;
	const int filtIdx = blockIdx.y;
	const int numFilters = gridDim.y;

	//只是给shBias找个机会赋值而已
	if(threadIdx.x + threadIdx.y == 0) {
		shBias = biases[filtIdx];
	}

	//为了得到需要计算的image和filter和target的数据起始点
	imgs += imgIdx * imgPixs;
	filters += filtIdx * filPixs;
	targets += imgIdx * numFilters * convPixs + filtIdx * convPixs \
			   + threadIdx.y * CONV_FORWARD_SIZE + threadIdx.x;

	//多线程复制数据到sm里面
	for(int i = 0; i < imgConvtimes + 1; i++){
		for(int j = 0; j < imgConvtimes + 1; j++){
			int col = threadIdx.x + blockDim.x * i;
			int row = threadIdx.y + blockDim.y * j;	
			if((row < IMG_SIZE) && (col < IMG_SIZE)){
				shImg[row][col] = imgs[row * IMG_SIZE + col];
			}
		}
	}
	for(int i = 0; i < filConvtimes + 1; i++){
		for(int j = 0; j < filConvtimes + 1; j++){
			int col = threadIdx.x + blockDim.x * i;
			int row = threadIdx.y + blockDim.y * j;	
			if((row < FILTER_SIZE) && (col < FILTER_SIZE)){
				shFilter[row][col] = filters[row * FILTER_SIZE + col];
			}
		}
	}

	__syncthreads();

	float *myShImg = &shImg[0][0];
	myShImg += threadIdx.y * IMG_SIZE + threadIdx.x;
	float prod = shBias;

	for(int i = 0; i < FILTER_SIZE; i++){
		for(int j = 0; j < FILTER_SIZE; j++){
			prod += shFilter[i][j] * myShImg[i * IMG_SIZE + j];
		}
	}
	__syncthreads();

	targets[0] = logistic(prod);
	//targets[0] = prod;
}

__global__ void avg_pooling(float* convOutputs, float* targets){
	const int numFilters = gridDim.y;
	const int imgIdx = blockIdx.x;
	const int filtIdx = blockIdx.y;

	__shared__ float shFeatureMap[CONV_FORWARD_SIZE][CONV_FORWARD_SIZE];

	int convPixs = CONV_FORWARD_SIZE * CONV_FORWARD_SIZE;
	int poolPixs = POOL_FORWARD_SIZE * POOL_FORWARD_SIZE;
	convOutputs += imgIdx * numFilters * convPixs + filtIdx * convPixs; 
	targets += imgIdx * numFilters * poolPixs + filtIdx * poolPixs \
			   + threadIdx.y * POOL_FORWARD_SIZE + threadIdx.x;

	if((blockDim.x > CONV_FORWARD_SIZE) && (blockDim.y > CONV_FORWARD_SIZE) \
			&& (threadIdx.x < CONV_FORWARD_SIZE) && (threadIdx.y < CONV_FORWARD_SIZE)){
		shFeatureMap[threadIdx.y][threadIdx.x] = \
												 convOutputs[threadIdx.y * CONV_FORWARD_SIZE + threadIdx.x];
	}
	if((blockDim.x <= CONV_FORWARD_SIZE) && (blockDim.y <= CONV_FORWARD_SIZE)){
		int dist = CONV_FORWARD_SIZE - blockDim.x;

		shFeatureMap[threadIdx.y][threadIdx.x] = \
												 convOutputs[threadIdx.y * CONV_FORWARD_SIZE + threadIdx.x];
		if(threadIdx.y < dist){
			shFeatureMap[threadIdx.y + blockDim.x][threadIdx.x] = \
																  convOutputs[(threadIdx.y + blockDim.x) * CONV_FORWARD_SIZE \
																  + threadIdx.x];
		}
		if(threadIdx.x < dist){
			shFeatureMap[threadIdx.y][threadIdx.x + blockDim.x] = \
																  convOutputs[(threadIdx.y) * CONV_FORWARD_SIZE \
																  + threadIdx.x + blockDim.x];
		}
		if(threadIdx.y < dist && threadIdx.x < dist){
			shFeatureMap[threadIdx.y + blockDim.x][threadIdx.x + blockDim.x] = \
																			   convOutputs[(threadIdx.y + blockDim.x) * CONV_FORWARD_SIZE \
																			   + threadIdx.x + blockDim.x];
		}
	}
	__syncthreads();

	float *myShFM = &shFeatureMap[0][0];
	myShFM +=  threadIdx.y * CONV_FORWARD_SIZE * AVG_POOL_Y \
			   + threadIdx.x * AVG_POOL_X;

	float avg_value = 0;
	for(int i = 0; i < AVG_POOL_X; i++){
		for(int j = 0; j < AVG_POOL_Y; j++){
			avg_value += myShFM[i * CONV_FORWARD_SIZE + j];
		}
	}
	__syncthreads();
	targets[0] = avg_value / (AVG_POOL_X * AVG_POOL_Y);
}

//row-major
__global__ void compute_dE_dy_j(const float* y_j, const float* labels, \
		float* dE_dy_j, const int width) {
	const int tx = blockIdx.x;
	const int ty = blockIdx.x * width + threadIdx.x;

	const int lab = labels[tx];

	if(threadIdx.x < width)
		dE_dy_j[ty] = y_j[ty] - (lab == threadIdx.x);
	__syncthreads();
}


__global__ void compute_dE_dy_h_avg(const float* dE_dy_i, float* out){

	int convPixs = CONV_FORWARD_SIZE * CONV_FORWARD_SIZE;
	int poolPixs = POOL_FORWARD_SIZE * POOL_FORWARD_SIZE;

	const int numFilters = gridDim.y;
	const int imgIdx = blockIdx.x;
	const int filtIdx = blockIdx.y;

	if(threadIdx.x < POOL_FORWARD_SIZE && threadIdx.y < POOL_FORWARD_SIZE){
		out += imgIdx * numFilters * convPixs + filtIdx * convPixs \
			   + threadIdx.y * CONV_FORWARD_SIZE * AVG_POOL_Y \
			   + threadIdx.x * AVG_POOL_X; 
		dE_dy_i += imgIdx * numFilters * poolPixs + filtIdx * poolPixs \
				   + threadIdx.y * POOL_FORWARD_SIZE + threadIdx.x;

		for(int i = 0; i < AVG_POOL_X; i++){
			for(int j = 0; j < AVG_POOL_Y; j++){
				out[i * CONV_FORWARD_SIZE + j] 
					= dE_dy_i[0] / (AVG_POOL_X * AVG_POOL_Y); 
			}
		}
		__syncthreads();
	}
}

__global__ void convolution_backward(const float* imgs, const float* filters, \
		float* targets, int convFiltimes, int imgFiltimes) {
	//filConvtimes指的是filter的大小是卷积结果的多少倍，也就是说是线程总数的多少倍
	//通过这么多次的线程重复赋值到shared memory

	int imgPixs = IMG_SIZE * IMG_SIZE;
	int convPixs = CONV_FORWARD_SIZE * CONV_FORWARD_SIZE;
	int filPixs = FILTER_SIZE * FILTER_SIZE;
	//一个block计算一张图与一个featuremap的卷积，开辟了28个线程，每个线程算28个值
	//放在shared memory里面的数据只有输入图片
	//前向卷积生成的输出不共享，直接一张图求一个点

	__shared__ float shImg[IMG_SIZE][IMG_SIZE];
	__shared__ float shConv[CONV_FORWARD_SIZE][CONV_FORWARD_SIZE];

	const int imgIdx = blockIdx.x;
	const int filtIdx = blockIdx.y;
	const int numFilters = gridDim.y;

	//为了得到需要计算的image和filter和target的数据起始点
	imgs += imgIdx * imgPixs;
	filters += imgIdx * numFilters * convPixs + filtIdx * convPixs;
	targets += imgIdx * numFilters * filPixs + filtIdx * filPixs \
			   +threadIdx.y * FILTER_SIZE + threadIdx.x;
	//			   + (FILTER_SIZE - 1 - threadIdx.y) * FILTER_SIZE \
	//			   + FILTER_SIZE - 1 - threadIdx.x;

	//多线程复制数据到sm里面
	for(int i = 0; i < imgFiltimes + 1; i++){
		for(int j = 0; j < imgFiltimes + 1; j++){
			int col = threadIdx.x + blockDim.x * i;
			int row = threadIdx.y + blockDim.y * j;	
			if((row < IMG_SIZE) && (col < IMG_SIZE)){
				shImg[row][col] = imgs[row * IMG_SIZE + col];
			}
		}
	}
	//filp 180
	for(int i = 0; i < convFiltimes + 1; i++){
		for(int j = 0; j < convFiltimes + 1; j++){
			int col = threadIdx.x + blockDim.x * i;
			int row = threadIdx.y + blockDim.y * j;	
			if((row < CONV_FORWARD_SIZE) && (col < CONV_FORWARD_SIZE)){
				//				shConv[CONV_FORWARD_SIZE - 1 - row][CONV_FORWARD_SIZE -1 - col] \
				= filters[row * CONV_FORWARD_SIZE + col];
				shConv[row][col] \
					= filters[row * CONV_FORWARD_SIZE + col];
			}
		}
	}

	__syncthreads();

	float *myShImg = &shImg[0][0];
	myShImg += threadIdx.y * IMG_SIZE + threadIdx.x;
	float prod = 0;

	for(int i = 0; i < CONV_FORWARD_SIZE; i++){
		for(int j = 0; j < CONV_FORWARD_SIZE; j++){
			prod += shConv[i][j] * myShImg[i * IMG_SIZE + j];
		}
	}
	__syncthreads();

	targets[0] = prod;

}


__global__ void max_pooling(float* convOutputs, float* targets, int* maxPoolPos){
	const int numFilters = gridDim.y;
	const int imgIdx = blockIdx.x;
	const int filtIdx = blockIdx.y;

	__shared__ float shFeatureMap[CONV_FORWARD_SIZE][CONV_FORWARD_SIZE];

	int convPixs = CONV_FORWARD_SIZE * CONV_FORWARD_SIZE;
	int poolPixs = POOL_FORWARD_SIZE * POOL_FORWARD_SIZE;
	
	if(threadIdx.x < POOL_FORWARD_SIZE && threadIdx.y < POOL_FORWARD_SIZE){
		convOutputs += imgIdx * numFilters * convPixs + filtIdx * convPixs \
					+ threadIdx.y * CONV_FORWARD_SIZE * AVG_POOL_Y \
					+ threadIdx.x * AVG_POOL_X; 
		targets += imgIdx * numFilters * poolPixs + filtIdx * poolPixs \
					+ threadIdx.y * POOL_FORWARD_SIZE + threadIdx.x;
		maxPoolPos += imgIdx * numFilters * poolPixs + filtIdx * poolPixs \
						+ threadIdx.y * POOL_FORWARD_SIZE + threadIdx.x;

		for(int i = 0; i < AVG_POOL_Y; i++){
			for(int j = 0; j < AVG_POOL_X; j++){
				shFeatureMap[threadIdx.y * AVG_POOL_Y + i][threadIdx.x * AVG_POOL_X + j] \
					= convOutputs[i * CONV_FORWARD_SIZE + j];
			}
		}
		__syncthreads();

		float *myShFM = &shFeatureMap[0][0];
		myShFM +=  threadIdx.y * CONV_FORWARD_SIZE * AVG_POOL_Y \
				+ threadIdx.x * AVG_POOL_X;

		float max_value = -10000;
		int max_pos = 0;
		for(int i = 0; i < AVG_POOL_Y; i++){
			for(int j = 0; j < AVG_POOL_X; j++){
				if(myShFM[i * CONV_FORWARD_SIZE + j] > max_value){
					max_value = myShFM[i * CONV_FORWARD_SIZE + j];
					max_pos = i * AVG_POOL_Y + j;
				}
			}
		}
		targets[0] = max_value;
		maxPoolPos[0] = max_pos;
	}
}


__global__ void compute_dE_dy_h_max(float* dE_dy_i, float* out, int* maxPoolPos){

	int convPixs = CONV_FORWARD_SIZE * CONV_FORWARD_SIZE;
	int poolPixs = POOL_FORWARD_SIZE * POOL_FORWARD_SIZE;

	const int numFilters = gridDim.y;
	const int imgIdx = blockIdx.x;
	const int filtIdx = blockIdx.y;

	if(threadIdx.x < POOL_FORWARD_SIZE && threadIdx.y < POOL_FORWARD_SIZE){
		out += imgIdx * numFilters * convPixs + filtIdx * convPixs \
			   + threadIdx.y * CONV_FORWARD_SIZE * MAX_POOL_Y \
			   + threadIdx.x * MAX_POOL_X; 
		dE_dy_i += imgIdx * numFilters * poolPixs + filtIdx * poolPixs \
				   + threadIdx.y * POOL_FORWARD_SIZE + threadIdx.x;
		maxPoolPos += imgIdx * numFilters * poolPixs + filtIdx * poolPixs \
					  + threadIdx.y * POOL_FORWARD_SIZE + threadIdx.x;
		int pos = maxPoolPos[0];
		int row = pos / MAX_POOL_Y;
		int col = pos % MAX_POOL_X;
		out[row * CONV_FORWARD_SIZE + col] = dE_dy_i[0];
	}
}

__global__ void compute_dE_db_h(const float* dE_dx_h, float* dE_db_h) {
	extern __shared__ float result[];

	const int idx = threadIdx.x + blockDim.x * threadIdx.y; 
	const int filtIdx = blockIdx.y;
	const int imgIdx = blockIdx.x;
	const int numFilters = gridDim.y;

	if(idx == 0)
		result[0] = 0;
	//某一张24*24的起始位置，本函数是将这24*24个点全部加起来最后生成一个点
	const int filPixs = CONV_FORWARD_SIZE * CONV_FORWARD_SIZE;
	dE_dx_h += imgIdx * numFilters * filPixs + filtIdx * filPixs + threadIdx.x;

	float ele = dE_dx_h[0];

	__syncthreads();
	atomicAdd(result, ele);
	__syncthreads();

	if (idx == 0) {
		dE_db_h[imgIdx * numFilters + filtIdx] = result[0] / filPixs;
	}
} 















