///
/// \file data.cu
/// 

#include "data.hpp"

using namespace std;

template <typename Dtype>
void Data<Dtype>::copyFromHost(Dtype* data_value_in, const int data_len){
	hipError_t status = hipMemcpy(_data_value, data_value_in, \
			sizeof(Dtype) * data_len, hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		cout << stderr, "!!!! device access error (write)\n";
		exit( EXIT_FAILURE );
	}  	
}

template <typename Dtype>
void Data<Dtype>::copyFromDevice(Data* data_in){
	hipError_t status = hipMemcpy(_data_value, data_in->getDevData(), \
			sizeof(Dtype) * _amount, hipMemcpyDeviceToDevice);
	if (status != hipSuccess) {
		cout << stderr, "!!!! device access error (write)\n";
		exit( EXIT_FAILURE );

	}   
}

template <typename Dtype>
void Data<Dtype>::copyToHost(Dtype* data_value_in, const int data_len){
	hipError_t status = hipMemcpy(data_value_in, _data_value, \
			sizeof(Dtype) * data_len, hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
		cout << stderr, "!!!! device access error (write)\n";
		exit( EXIT_FAILURE );
	}  	
}

template <typename Dtype>
void Data<Dtype>::copyToDevice(Data* data_in){
	hipError_t status = hipMemcpy(_data_value, data_in->getDevData(), \
			sizeof(Dtype) * _amount, hipMemcpyDeviceToDevice);
	if (status != hipSuccess) {
		cout << stderr, "!!!! device access error (write)\n";
		exit( EXIT_FAILURE );

	}   
}

template <typename Dtype>
void Data<Dtype>::zeros(){
	hipMemset(_data_value, 0, _amount * sizeof(Dtype));
}

