/*
 * filename: data.cu
 */ 

#include "data.hpp"

template <typename Dtype>
Data<Dtype>::copyFromHost(Dtype* data_value_in, const int data_len){
	hipError_t status = hipMemcpy(_data_value, data_value_in, \
			sizeof(Dtype) * data_len, hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		cout << stderr, "!!!! device access error (write)\n";
		exit( EXIT_FAILURE );
	}  	
}

template <typename Dtype>
Data<Dtype>::copyFromDevice(Data* data_in){
	hipError_t status = hipMemcpy(_data_value, data_in->getDevData(), \
			sizeof(Dtype) * _amount, hipMemcpyDeviceToDevice);
	if (status != hipSuccess) {
		cout << stderr, "!!!! device access error (write)\n";
		exit( EXIT_FAILURE );

	}   
}

template <typename Dtype>
Data<Dtype>::copyFromHost(Dtype* data_value_in, const int data_len){
	hipError_t status = hipMemcpy(data_value_in, _data_value, \
			sizeof(Dtype) * data_len, hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
		cout << stderr, "!!!! device access error (write)\n";
		exit( EXIT_FAILURE );
	}  	
}

template <typename Dtype>
Data<Dtype>::copyFromDevice(Data* data_in){
	hipError_t status = hipMemcpy(_data_value, data_in->getDevData(), \
			sizeof(Dtype) * _amount, hipMemcpyDeviceToDevice);
	if (status != hipSuccess) {
		cout << stderr, "!!!! device access error (write)\n";
		exit( EXIT_FAILURE );

	}   
}

void Data<Dtype>::dataZeros(){
	hipMemset(_data_value, 0, _amount * sizeof(Dtype));
}

void Data<Dtype>::dataZeros(){
	hipMemset(_diff_value, 0, _amount * sizeof(Dtype));
}

