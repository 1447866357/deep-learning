#include "hip/hip_runtime.h"
/*
 * filename:main.cu
 */

#include <iostream>
#include <fstream>
#include <time.h>
#include <cmath>
#include <omp.h>
#include "mpi.h"
#include "matrix.h"
#include "nvmatrix.cuh"
#include "convnet.cuh"
#include "convnet_kernel.cuh"
#include "utils.h"
#include "logistic.cuh"

using namespace std;

#define THREAD_END 100000
enum swapInfo{SWAP_HIDVIS_PUSH, SWAP_HIDBIAS_PUSH, \
	SWAP_AVGOUT_PUSH, SWAP_OUTBIAS_PUSH,
	SWAP_HIDVIS_FETCH, SWAP_HIDBIAS_FETCH, \
		SWAP_AVGOUT_FETCH, SWAP_OUTBIAS_FETCH};

int numProcess;
int rank;

void managerNode(pars* cnn, pars* logistic){

	cout << "=========================\n" \
		<< "train: " << cnn->trainNum \
		<< "\nvalid: " << cnn->validNum \
		<< "\nfiltersize: " << cnn->filterSize \
		<< "\nnumFilters: " << cnn->numFilters \
		<< "\nepsHidVis: " << cnn->epsHidVis \
		<< "\nepsHidBias: " << cnn->epsHidBias \
		<< "\nepsAvgOut: " << logistic->epsAvgOut \
		<< "\nepsOutBias: " << logistic->epsOutBias \
		<< "\nmom: " << cnn->mom \
		<< "\nwcHidVis: " << cnn->wcHidVis \
		<< "\nwcAvgOut: " << logistic->wcAvgOut << endl;

	int inLen = cnn->inSize * cnn->inSize * cnn->inChannel;
	int hidVisLen = cnn->numFilters * cnn->filterSize \
			* cnn->filterSize;
	int hidBiasLen = cnn->numFilters * 1;
	int avgOutLen = cnn->poolResultSize * cnn->poolResultSize * cnn->numFilters * logistic->numOut;
	int outBiasLen = logistic->numOut;

	int proTrainDataLen = cnn->trainNum * inLen / (numProcess - 1);
	int proTrainLabelLen = cnn->trainNum / (numProcess - 1);
	int proValidDataLen = cnn->validNum * inLen / (numProcess - 1);
	int proValidLabelLen = cnn->validNum / (numProcess - 1);

	NVMatrix* nvTrainData = new NVMatrix(cnn->trainNum, inLen);
//					NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvValidData = new NVMatrix(cnn->validNum, inLen);
//					NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvTrainLabel = new NVMatrix(cnn->trainNum, 1);
//					NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvValidLabel = new NVMatrix(cnn->validNum, 1);
//					NVMatrix::ALLOC_ON_UNIFIED_MEMORY);

	readData(nvTrainData, "../data/input/mnist_train.bin", true);
	readData(nvValidData, "../data/input/mnist_valid.bin", true);
	readData(nvTrainLabel, "../data/input/mnist_label_train.bin", false);
	readData(nvValidLabel, "../data/input/mnist_label_valid.bin", false);

	Matrix* hHidVis = new Matrix(cnn->numFilters, cnn->filterSize * cnn->filterSize);
	Matrix* hHidBiases = new Matrix(cnn->numFilters, 1);
	Matrix* hAvgout = new Matrix(avgOutLen / logistic->numOut, logistic->numOut);
	Matrix* hOutBiases = new Matrix(1, logistic->numOut);

	NVMatrix* hidVis = new NVMatrix(cnn->numFilters, \
			cnn->filterSize * cnn->filterSize);
	NVMatrix* hidBiases = new NVMatrix(cnn->numFilters, 1);
	NVMatrix* avgOut = new NVMatrix(avgOutLen / logistic->numOut, logistic->numOut);
	NVMatrix* outBiases = new NVMatrix(1, logistic->numOut);

	initW(hHidVis->getData(), cnn->numFilters * cnn->filterSize * cnn->filterSize);
	memset(hHidBiases->getData(), 0, sizeof(float) * cnn->numFilters);
	memset(hAvgout->getData(), 0, sizeof(float) * avgOutLen);
	memset(hOutBiases->getData(), 0, sizeof(float) * logistic->numOut);

	//	readPars(hHidVis, "hHidVis_t1.bin");
	//	readPars(hHidBiases, "hHidBiases_t1.bin");
	//	readPars(hAvgout, "hAvgout_t1.bin");
	//	readPars(hOutBiases, "hOutBiases_t1.bin");

	MPI_Bcast(hHidVis->getData(), hidVisLen, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(hHidBiases->getData(), hidBiasLen, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(hAvgout->getData(), avgOutLen, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(hOutBiases->getData(), outBiasLen, MPI_FLOAT, 0, MPI_COMM_WORLD);
/*
float* send = new float[proTrainDataLen];
for(int i =0; i < proTrainDataLen; i++){
	send[i] = 1;
}*/
	for(int i = 1; i < numProcess; i++){
		MPI_Send(nvTrainData->getDevData()+(i-1)*proTrainDataLen, proTrainDataLen, \
				MPI_FLOAT, i, i, MPI_COMM_WORLD);
		MPI_Send(nvTrainLabel->getDevData()+(i-1)*proTrainLabelLen, \
				proTrainLabelLen, MPI_FLOAT, i, i, MPI_COMM_WORLD);
		MPI_Send(nvValidData->getDevData()+(i-1)*proValidDataLen, proValidDataLen, \
				MPI_FLOAT, i, i, MPI_COMM_WORLD);
		MPI_Send(nvValidLabel->getDevData()+(i-1)*proValidLabelLen, \
				proValidLabelLen, MPI_FLOAT, i, i, MPI_COMM_WORLD);

	}

	//pro进程，每个进程进行的数据交换次数，0123是push，4567是fetch
	//4个数据地址，8个线程来分别实现两种操作
	const int transOPTimesInPro = 8;
	const int numDataType = 4;
	float* myData[numDataType] = {hidVis->getDevData(), hidBiases->getDevData(), \
		avgOut->getDevData(), outBiases->getDevData()};
	int myLen[numDataType] = {hidVisLen, hidBiasLen, avgOutLen, outBiasLen};

	#pragma omp parallel num_threads(transOPTimesInPro * (numProcess - 1)) 
	{

		MPI_Status status;
		int myState = 0;

		int tid = omp_get_thread_num();
		int pid = tid / transOPTimesInPro + 1;
		int swapId = tid % transOPTimesInPro;
		int dataAddr = tid % numDataType;

//		cout << "tid" << tid<< endl;

		while(myState != THREAD_END){
			MPI_Recv(&myState, 1, MPI_INT, pid, \
					swapId*10000, MPI_COMM_WORLD, &status);

			if(swapId < numDataType){
				MPI_Recv(myData[dataAddr], myLen[dataAddr], MPI_FLOAT, pid, \
						swapId+ myState, MPI_COMM_WORLD, &status);
			}else{
				MPI_Send(myData[dataAddr], myLen[dataAddr], MPI_FLOAT, pid, \
						swapId + myState, MPI_COMM_WORLD);
			}   
		}
	}

	delete nvTrainData;
	delete nvTrainLabel;
	delete nvValidData;
	delete nvValidLabel;
	delete hHidVis;
	delete hHidBiases;
	delete hAvgout;
	delete hOutBiases;
	delete hidVis;
	delete hidBiases;
	delete avgOut;
	delete outBiases;
}


void workerNode(pars* cnn, pars* logistic){
	int inLen = cnn->inSize * cnn->inSize * cnn->inChannel;
	int hidVisLen = cnn->numFilters * cnn->filterSize \
			* cnn->filterSize;
	int hidBiasLen = cnn->numFilters * 1;
	int avgOutLen = cnn->poolResultSize * cnn->poolResultSize * cnn->numFilters * logistic->numOut;
	int outBiasLen = logistic->numOut;
	int miniDataLen = cnn->minibatchSize * inLen;
	int miniLabelLen = cnn->minibatchSize;

	int proTrainDataLen = cnn->trainNum * inLen / (numProcess - 1);
	int proTrainLabelLen = cnn->trainNum / (numProcess - 1);
	int proValidDataLen = cnn->validNum * inLen / (numProcess - 1);
	int proValidLabelLen = cnn->validNum / (numProcess - 1);

	NVMatrix* nvTrainData = new NVMatrix(cnn->trainNum/(numProcess-1), inLen);
//				NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvValidData = new NVMatrix(cnn->validNum/(numProcess-1), inLen);
//			NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvTrainLabel = new NVMatrix(cnn->trainNum / (numProcess - 1), 1);
//				NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	NVMatrix* nvValidLabel = new NVMatrix(cnn->validNum / (numProcess - 1), 1);
//				NVMatrix::ALLOC_ON_UNIFIED_MEMORY);

	NVMatrix* miniData = new NVMatrix(nvTrainData->getDevData(), \
			cnn->minibatchSize, inLen);
	NVMatrix* miniLabel = new NVMatrix(nvTrainLabel->getDevData(), \
			cnn->minibatchSize, 1);

	Matrix* hHidVis = new Matrix(cnn->numFilters, cnn->filterSize * cnn->filterSize);
	Matrix* hHidBiases = new Matrix(cnn->numFilters, 1); 
	Matrix* hAvgout = new Matrix(avgOutLen / logistic->numOut, logistic->numOut);
	Matrix* hOutBiases = new Matrix(1, logistic->numOut);

	MPI_Bcast(hHidVis->getData(), hidVisLen, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(hHidBiases->getData(), hidBiasLen, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(hAvgout->getData(), avgOutLen, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(hOutBiases->getData(), outBiasLen, MPI_FLOAT, 0, MPI_COMM_WORLD);


	MPI_Status status;
	MPI_Recv(nvTrainData->getDevData(), proTrainDataLen, \
			MPI_FLOAT, 0, rank, MPI_COMM_WORLD, &status);
	MPI_Recv(nvTrainLabel->getDevData(), proTrainLabelLen, \
			MPI_FLOAT, 0, rank, MPI_COMM_WORLD, &status);
	MPI_Recv(nvValidData->getDevData(), proValidDataLen, \
			MPI_FLOAT, 0, rank, MPI_COMM_WORLD, &status);
	MPI_Recv(nvValidLabel->getDevData(), proValidLabelLen, \
			MPI_FLOAT, 0, rank, MPI_COMM_WORLD, &status);


	ConvNet layer1(hHidVis, hHidBiases, cnn);
	layer1.initCuda();
	Logistic layer2(hAvgout, hOutBiases, logistic);
	layer2.initCuda();

	int passMsg = 0;

	NVMatrix* y_i;
	NVMatrix* hidVis;
	NVMatrix* hidBiases;
	NVMatrix* avgOut;
	NVMatrix* outBiases;
	NVMatrix* dE_dy_j;

	clock_t t;
	t = clock();


	for(int epochIdx = 0; epochIdx < cnn->numEpoches; epochIdx++){
		int error = 0;

		for(int batchIdx = 0; batchIdx < cnn->numMinibatches; batchIdx++){

			miniData->changePtrFromStart(nvTrainData->getDevData(), \
					miniDataLen * batchIdx);
			miniLabel->changePtrFromStart(nvTrainLabel->getDevData(), \
					miniLabelLen * batchIdx);
			layer1.computeConvOutputs(miniData);
			layer1.computeMaxOutputs();
			y_i = layer1.getYI();
			layer2.computeClassOutputs(y_i);
			layer2.computeError(miniLabel, error);
			dE_dy_j = layer2.getDEDYJ();
			avgOut = layer2.getAvgOut();
			layer2.computeDerivs(y_i, miniLabel);
			layer1.computeDerivs(miniData, dE_dy_j, avgOut);
			layer1.updatePars();
			layer2.updatePars();
			avgOut = layer2.getAvgOut();
			outBiases = layer2.getOutBias();
			hidVis = layer1.getHidVis();
			hidBiases = layer1.getHidBias();
			if((batchIdx + 1) % cnn->nPush == 0){
				if(epochIdx == cnn->numEpoches - 1){
					if((batchIdx + cnn->nPush) >= cnn->numMinibatches \
							|| batchIdx == cnn->numMinibatches - 1)
						passMsg = THREAD_END;
					else
						passMsg = batchIdx;
				}
				else
					passMsg = batchIdx;
				MPI_Send(&passMsg, 1, MPI_INT, 0, SWAP_HIDVIS_PUSH*10000, \
						MPI_COMM_WORLD);
				MPI_Send(hidVis->getDevData(), hidVisLen, \
						MPI_FLOAT, 0, SWAP_HIDVIS_PUSH + passMsg, MPI_COMM_WORLD);

				MPI_Send(&passMsg, 1, MPI_INT, 0, SWAP_HIDBIAS_PUSH*10000, \
						MPI_COMM_WORLD);
				MPI_Send(hidBiases->getDevData(), hidBiasLen, \
						MPI_FLOAT, 0, SWAP_HIDBIAS_PUSH + passMsg, MPI_COMM_WORLD);

				MPI_Send(&passMsg, 1, MPI_INT, 0, SWAP_AVGOUT_PUSH*10000, \
						MPI_COMM_WORLD);
				MPI_Send(avgOut->getDevData(), avgOutLen, \
						MPI_FLOAT, 0, SWAP_AVGOUT_PUSH + passMsg, MPI_COMM_WORLD);

				MPI_Send(&passMsg, 1, MPI_INT, 0, SWAP_OUTBIAS_PUSH*10000, \
						MPI_COMM_WORLD);
				MPI_Send(outBiases->getDevData(), outBiasLen, \
						MPI_FLOAT, 0, SWAP_OUTBIAS_PUSH + passMsg, MPI_COMM_WORLD);
			}

			if((batchIdx + 1) % cnn->nFetch == 0){
				if(epochIdx == cnn->numEpoches - 1){
					if((batchIdx + cnn->nFetch) >= cnn->numMinibatches \
							|| batchIdx == cnn->numMinibatches - 1)
						passMsg = THREAD_END;
					else
						passMsg = batchIdx;
				}else
					passMsg = batchIdx;
				MPI_Send(&passMsg, 1, MPI_INT, 0, SWAP_HIDVIS_FETCH*10000, \
						MPI_COMM_WORLD);
				MPI_Recv(hidVis->getDevData(), hidVisLen, MPI_FLOAT, 0, \
						SWAP_HIDVIS_FETCH + passMsg, MPI_COMM_WORLD, &status);

				MPI_Send(&passMsg, 1, MPI_INT, 0, SWAP_HIDBIAS_FETCH*10000, \
						MPI_COMM_WORLD);
				MPI_Recv(hidBiases->getDevData(), hidBiasLen, MPI_FLOAT, \
						0, SWAP_HIDBIAS_FETCH + passMsg, \
						MPI_COMM_WORLD, &status);

				MPI_Send(&passMsg, 1, MPI_INT, 0, SWAP_AVGOUT_FETCH*10000, \
						MPI_COMM_WORLD);
				MPI_Recv(avgOut->getDevData(), avgOutLen, MPI_FLOAT, 0, \
						SWAP_AVGOUT_FETCH + passMsg, MPI_COMM_WORLD, &status);

				MPI_Send(&passMsg, 1, MPI_INT, 0, SWAP_OUTBIAS_FETCH*10000, \
						MPI_COMM_WORLD);
				MPI_Recv(outBiases->getDevData(), outBiasLen, MPI_FLOAT, \
						0, SWAP_OUTBIAS_FETCH + passMsg, \
						MPI_COMM_WORLD, &status);
			}
			if(batchIdx == cnn->numMinibatches - 1){ 
				int errorValid = 0;
				float loglihoodValid = 0;
				for(int validIdx = 0; validIdx < cnn->numValidBatches; validIdx++){

					miniData->changePtrFromStart(nvValidData->getDevData(), \
							miniDataLen * validIdx);
					miniLabel->changePtrFromStart(nvValidLabel->getDevData(), \
							miniLabelLen * validIdx);
					layer1.computeConvOutputs(miniData);
					layer1.computeMaxOutputs();
					y_i = layer1.getYI();
					layer2.computeClassOutputs(y_i);
					loglihoodValid += layer2.computeError(miniLabel, errorValid);

				}
				int totalValid = errorValid;
				if(numProcess > 2){
					if(rank == 1){
						for(int i = 2; i < numProcess; i++){
							MPI_Recv(&errorValid, 1, MPI_INT, i, i, \
									MPI_COMM_WORLD, &status);   
							totalValid += errorValid;
						}       
					}else{  
						MPI_Send(&errorValid, 1, MPI_INT, 1, rank, MPI_COMM_WORLD);
					}       
				}       
				if(rank == 1)
					cout << "epochIdx: " << epochIdx << ",error: " \
						<< (float)totalValid/cnn->validNum \
						<< ",likelihood: "<< loglihoodValid<< endl;
			}  

		}
		if(rank == 1){
			t = clock() - t;
			cout << " " << ((float)t/CLOCKS_PER_SEC) << " seconds.\n";
			t = clock();
		}

	}


	delete nvTrainData;
	delete nvTrainLabel;
	delete nvValidData;
	delete nvValidLabel;

}

int main(int argc, char** argv){

	int prov;
	MPI_Init_thread(&argc,&argv,MPI_THREAD_MULTIPLE, &prov);
	if (prov < MPI_THREAD_MULTIPLE)
	{   
		printf("Error: the MPI library doesn't provide the required thread level\n");
		MPI_Abort(MPI_COMM_WORLD, 0); 
	}   
	MPI_Comm_rank(MPI_COMM_WORLD,&rank);
	MPI_Comm_size(MPI_COMM_WORLD,&numProcess);

	if(numProcess <= 1){
		printf("Error: process number must bigger than 1\n");
		MPI_Abort(MPI_COMM_WORLD, 0); 
	}

	//检测有几个gpu
	int numGpus;
	hipGetDeviceCount(&numGpus);
	hipSetDevice(rank%numGpus);

/*
	// Ensure that RDMA ENABLED CUDA is set correctly
    int direct = getenv("MPICH_RDMA_ENABLED_CUDA")==NULL?0:atoi(getenv ("MPICH_RDMA_ENABLED_CUDA"));
    if(direct != 1){
        printf ("MPICH_RDMA_ENABLED_CUDA not enabled!\n");
        exit (EXIT_FAILURE);
    }
*/

	pars* cnn = new pars;
	pars* logistic = new pars;

	cnn->epsHidVis = 0.1;
	cnn->epsHidBias = 0.1;
	cnn->mom = 0;
	cnn->wcHidVis = 0;
	cnn->inSize = 28; 
	cnn->inChannel = 1;
	cnn->filterSize = 5;
	cnn->numFilters = 16; 
	cnn->stepSize = 1;
	cnn->convResultSize = cnn->inSize - cnn->filterSize + 1;
	cnn->poolSize = 2;
	cnn->poolResultSize = cnn->convResultSize / cnn->poolSize;
	cnn->trainNum = 50000;
	cnn->validNum = 10000;
	cnn->minibatchSize = 100;
	cnn->numMinibatches = cnn->trainNum / (cnn->minibatchSize * (numProcess - 1));
	cnn->numValidBatches = cnn->validNum / (cnn->minibatchSize * (numProcess - 1));
	cnn->numEpoches = 1; 
	cnn->nPush = 4;
	cnn->nFetch = 5;

	logistic->wcAvgOut = 0;
	logistic->epsAvgOut = 0.1;
	logistic->epsOutBias = 0.1;
	logistic->mom = 0;
	logistic->numOut = 10; 
	logistic->minibatchSize = 100;

	if(rank == 0){ 
		managerNode(cnn, logistic);
	}   
	else{
		workerNode(cnn, logistic);
	} 	

	delete cnn;
	delete logistic;
	MPI_Finalize();
	return 0;
}




















