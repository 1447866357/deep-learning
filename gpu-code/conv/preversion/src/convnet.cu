#include "hip/hip_runtime.h"
/*
 * filename: convnet.cu
 */
//#include <cutil_inline.h>
#include <time.h>

#include "convnet.cuh"
#include "convnet_kernel.cuh"

using namespace std;
	

ConvNet::ConvNet(Matrix* hHidVis, Matrix* hHidBiases, pars* netWork){

	this->_numFilters            = hHidVis->getNumRows();

	this->_hHidVis               = hHidVis;
	this->_hHidBiases            = hHidBiases;

	this->_epsHidVis             = netWork->epsHidVis;
	//hidden bias的learning rate
	this->_epsHidBias            = netWork->epsHidBias;
	//上一次更新的参数控制增长趋势
	this->_mom                   = netWork->mom;
	//hidden原值的参数
	this->_wcHidVis              = netWork->wcHidVis;
	//out原值的参数
	this->_minibatchSize         = netWork->minibatchSize;
	this->_inSize				 = netWork->inSize;
	this->_filterSize			 = netWork->filterSize;
	this->_stepSize              = netWork->stepSize;
	this->_convResultSize		 = _inSize - _filterSize + 1;
	this->_poolResultSize		 = this->_convResultSize / AVG_POOL_X;
	this->_inChannel			 = netWork->inChannel;
	hipblasCreate(&handle);
}
ConvNet::~ConvNet() {
		delete _hHidVis;
		delete _hHidBiases;

		delete _hidVis;
		delete _hidVisInc;
		delete _hidBiases;
		delete _hidBiasInc;

		delete _y_h; 
		delete  _y_i; 
		delete _dE_dy_i;
		delete _dE_dy_h;
		delete _dE_dx_h;
		delete _dE_dw_hk;
		delete _dE_db_h;
		hipFree(_maxPoolPos);
	hipblasDestroy(handle);
}

void ConvNet::initCuda() {
	//hipSetDevice(cutGetAvgGflopsDeviceId());
	//NVMatrix::initDeviceProps();

	//hidVis大小是16*5*5,bias是5*5
	this->_hidVis            = new NVMatrix(_hHidVis, true);
//					NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	this->_hidBiases         = new NVMatrix(_hHidBiases, true);
//					NVMatrix::ALLOC_ON_UNIFIED_MEMORY);
	this->_y_h               = new NVMatrix(_minibatchSize, \
			_numFilters * _convResultSize * _convResultSize);
	this->_y_i               = new NVMatrix(_minibatchSize, \
			_numFilters * _poolResultSize * _poolResultSize);
	this->_dE_dy_i           = new NVMatrix(_y_i);

	this->_dE_dy_h           = new NVMatrix(_y_h);
	this->_dE_dx_h           = new NVMatrix(_y_h);
	this->_dE_dw_hk          = new NVMatrix(_hidVis);
	this->_dE_db_h           = new NVMatrix(_hidBiases);

	this->_hidVisInc		 = new NVMatrix(_numFilters, _filterSize * _filterSize);
	this->_hidBiasInc		 = new NVMatrix(_numFilters, 1);

	this->_hidVisInc->zeros();
	this->_hidBiasInc->zeros();
	
	hipError_t status = hipMalloc((void**) &_maxPoolPos, \
			_minibatchSize * _numFilters * _poolResultSize * _poolResultSize * sizeof(int));
    if (status != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error\n");
        exit(EXIT_FAILURE);
	}

}

void ConvNet::computeConvOutputs(NVMatrix* miniData){
	
		
	//16*16
	dim3 blocks = dim3(_minibatchSize, _numFilters);
	//28*5，此处需要改变，低效

	dim3 threads = dim3(_convResultSize, _convResultSize);

    int filConvtimes = _filterSize / _convResultSize;
    int imgConvtimes = _inSize / _convResultSize;

//	miniData->reValue(1.0f);
//	_hidVis->reValue(1.0f);
//	_y_h->reValue(0.0f);
	convolution_forward<<<blocks, threads>>>(miniData->getDevData(), \
			_hidVis->getDevData(), _hidBiases->getDevData(), _y_h->getDevData(), \
			filConvtimes, imgConvtimes);
	hipDeviceSynchronize();
//	miniData->showValue("minidata");
//	_hidVis->showValue("hidvis");
//	_y_h->showValue("yh");
	//	cutilCheckMsg("Kernel execution failed");
}

void ConvNet::computeAvgOutputs(){
	//16*16
	dim3 blocks = dim3(_minibatchSize, _numFilters);
	dim3 threads = dim3(_poolResultSize, _poolResultSize);
	//24*24,pooling到12*12
	avg_pooling<<<blocks, threads>>>(_y_h->getDevData(), _y_i->getDevData());	
	hipDeviceSynchronize();
}

void ConvNet::computeMaxOutputs(){
	//16*16
	dim3 blocks = dim3(_minibatchSize, _numFilters);
	dim3 threads = dim3(_poolResultSize, _poolResultSize);
	//24*24,pooling到12*12
	max_pooling<<<blocks, threads>>>(_y_h->getDevData(), _y_i->getDevData(), \
			_maxPoolPos);	

	hipDeviceSynchronize();
}

void ConvNet::computeDerivs(NVMatrix* miniData, NVMatrix* dE_dy_j, NVMatrix* avgOut){
	//assert(_minibatchSize % 16 == 0);

	//dE_dy_i, 16*16*12*12
	NVMatrix* avgOut_T = new NVMatrix(avgOut->getNumCols(), avgOut->getNumRows());
	avgOut->getTranspose(avgOut_T);
	dE_dy_j->rightMult(avgOut_T, 1, _dE_dy_i, handle);
	//每次还原一个点，因为四个点只需还原一个，因此只用12*12的线程做
	dim3 blocks = dim3(_minibatchSize, _numFilters);
	dim3 threads = dim3(ceil(_poolResultSize / 16.0) * 16,  ceil(_poolResultSize / 16.0) * 16);
	//dE_dy_h, 16*16*24*24
/*
t = clock() - t;
cout << "dEdyi: " << (float)t/CLOCKS_PER_SEC << " seconds. \n";
t = clock();
*/
	_dE_dy_h->zeros();
	compute_dE_dy_h_max<<<blocks, threads>>>(_dE_dy_i->getDevData(), \
			_dE_dy_h->getDevData(), _maxPoolPos);
	hipDeviceSynchronize();

	//dE_dx_h, 16*16*24*24
	_y_h->subtractFromScalar(1, _dE_dx_h);

	_dE_dx_h->eltWiseMult(_y_h);

	_dE_dx_h->eltWiseMult(_dE_dy_h);

//clock_t t = clock();
//cout << "????2\n";
	NVMatrix* dE_dw_hk_tmp = new NVMatrix(_minibatchSize, \
			_numFilters * _filterSize *_filterSize);
//cout << "????3\n";
	blocks = dim3(_minibatchSize, _numFilters);
	threads = dim3(_filterSize, _filterSize);
	int filConvtimes = _convResultSize / _filterSize;
	int imgConvtimes = _inSize / _filterSize;
	convolution_backward<<<blocks, threads>>>(miniData->getDevData(), \
			_dE_dx_h->getDevData(), dE_dw_hk_tmp->getDevData(), \
			filConvtimes, imgConvtimes);
	hipDeviceSynchronize();

	/*
t = clock() - t;
cout << "dEdwhktmp: " << (float)t/CLOCKS_PER_SEC << " seconds. \n";
t = clock();
*/

	//按每一列作为一个线程，故两者乘积要比16*24*24大
	dE_dw_hk_tmp->sumRow(_dE_dw_hk);
	
	NVMatrix* dE_db_h_tmp = new NVMatrix(_minibatchSize, _numFilters);
	blocks = dim3(_minibatchSize, _numFilters);
	threads = dim3(_convResultSize, _convResultSize);
	compute_dE_db_h<<<blocks, threads, sizeof(float)>>>(_dE_dx_h->getDevData(), \
			dE_db_h_tmp->getDevData());
	hipDeviceSynchronize();
/*
t = clock() - t;
cout << "dEdbhtmp: " << (float)t/CLOCKS_PER_SEC << " seconds. \n";
t = clock();
*/
	dE_db_h_tmp->sumRow(_dE_db_h);

	delete avgOut_T;
	delete dE_dw_hk_tmp;
	delete dE_db_h_tmp;

}

void ConvNet::updatePars(){

	_hidVisInc->addSum(_hidVis, _dE_dw_hk, _mom, -_wcHidVis, \
			-_epsHidVis / _minibatchSize);
	_hidVis->add(_hidVisInc, 1, 1);

	_hidBiasInc->add(_dE_db_h, _mom, -_epsHidBias / _minibatchSize);
	_hidBiases->add(_hidBiasInc, 1, 1);

}































