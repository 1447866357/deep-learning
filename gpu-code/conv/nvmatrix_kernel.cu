#include "hip/hip_runtime.h"
/*
 * filename:nvmatrix_kernel.cu
 */

#include <hip/hip_runtime.h>
#include "nvmatrix_kernel.cuh"

__constant__ float dMinus = 1;

__global__ void multiRowCol(float* aData, float* bData, float scaleAB, \
		float* target, const int numInRowCol, const int times ){
	extern __shared__ float result[];
	//a的每一行与b的每一行相乘

	const unsigned int idx = threadIdx.x * blockDim.y + threadIdx.y;
	const int threadNum = blockDim.x * blockDim.y;
	const int mIdx = blockIdx.x;
	const int nIdx = blockIdx.y;
	
	aData += mIdx * numInRowCol + idx;
	bData += nIdx * numInRowCol + idx;
	target += mIdx * gridDim.y + nIdx;

	if(idx == 0){
		result[0] = 0;
	}
	
	float ele = 0;
	for(int i = 0; i < times; i++){
		ele += scaleAB * aData[i * threadNum] * bData[i * threadNum];
	}
	if((threadNum * times < numInRowCol) && (idx < numInRowCol - threadNum * times)){
		ele += scaleAB * aData[threadNum * times] \
			   * bData[threadNum * times];
	}
	__syncthreads();
	atomicAdd(result, ele);
	__syncthreads();

	if(idx == 0){
		target[0] = result[0];
	}
}


__global__ void kAddColVector(float* mat, float* vec, float* tgtMat, \
		const unsigned int width, const unsigned int height, \
		const float scaleVec) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int numThreads = blockDim.x * gridDim.x;

	for (unsigned int i = idx; i < width * height; i += numThreads) {
		tgtMat[i] = mat[i] + scaleVec * vec[i / width];
	}
}

__global__ void kAddRowVector(float* mat, float* vec, float* tgtMat, \
		const unsigned int width, const unsigned int height, \
		const float scaleVec) {
	
	const unsigned int idxY = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int idxX = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idx = idxY * width + idxX;
	const unsigned int numThreads = blockDim.x * gridDim.x * \
									blockDim.y * gridDim.y;

	//此处控制了线程数要小于行列积
	for (unsigned int i = idx; i < width * height; i += numThreads) {
		tgtMat[idx] = mat[idx] + scaleVec * vec[idx % width];
		
	}
}

__global__ void kSoftmax(float* gData, unsigned int width, \
		unsigned int height) {   

	//跟同一个block里面值比较大小取最大值，减去最大值
	const unsigned int idxY = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int idxX = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idx = idxY * width + idxX;
	//数据放入共享内存
	int shareMemLength = ((width + ADD_BLOCK_SIZE - 1) / ADD_BLOCK_SIZE) \
						 * ADD_BLOCK_SIZE;
	//计算里长度最近的2的次方
	int pow2Length = shareMemLength;
	if(pow2Length & (pow2Length - 1)){
		while(pow2Length & (pow2Length - 1)){
			pow2Length &= pow2Length - 1;
		}
	}
	__shared__ float ori[shareMemLength];

	if(idxY < width)
		ori[idxY] = gData[idx];
	else
		ori[idxY] = -10000000;
		
	__syncthreads();

	//先通过reduce来求最大值
	if(idxY >= pow2Length && idxY < width)
		ori[idxY - pow2Length] = ori[idxY - pow2Length] > ori[idxY] \
								 ? ori[idxY- pow2Length] : ori[idxY];
	__syncthreads();

	for(int activeThreads = pow2Length >> 1; activeThreads; activeThreads >> 1){
		if(idxY < activeThreads){
			ori[idxY] = ori[idxY] > ori[idxY] \
						? ori[idxY + activeThreads] : ori[idxY];
		}
		__syncthreads();
	}

/*
	gData += blockIdx.x * numCols;
	target += blockIdx.x * numCols;

	double max = gData[0];
	for (unsigned int i = 1; i < numCols; i++){
		if(max < gData[i])
			max = gData[i];
	}
	double sum = 0;
	for (unsigned int i = 0; i < numCols; i++){
		target[i] = __expf(gData[i] - max);
		sum += target[i];
	}
	for (unsigned int i = 0; i < numCols; i++){
		target[i] = target[i] / sum;
	}
*/
}

__global__ void kReciprocal(float* gData, float* target, unsigned int numElements) {

	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x)
		target[i] = 1 / gData[i];
}

__global__ void kLog(float* gData, float* target, unsigned int numElements) {   

	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x){
		double tmp = gData[i] < 1 - 10e-15 ? gData[i] : 1 - 10e-15;
		tmp = tmp > 10e-15 ? tmp : 10e-15;
		target[i] = __logf(gData[i]);
	}
}

__global__ void kDumbSumCols(float* mat, float* vec, unsigned int width, \
		unsigned int height) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < height) {
		mat += idx * width;
		float sum = 0;
		for (int j = 0; j < width; j++) {
			sum += mat[j];
		}
		vec[idx] = sum;
	}
}

__global__ void kDumbSumRows(float* mat, float* vec, unsigned int width, \
		unsigned int height) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < width) {
		mat += idx;
		float sum = 0;
		for (int j = 0; j < height; j++) {
			sum += mat[j * width];
		}
		vec[idx] = sum;
	}
}

__global__ void kSumRowInterval(float* mat, float* vec, unsigned int width, \
		unsigned int height, int interval) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < height) {
		mat += idx * width;
		float sum = 0;
		for (int j = 0; j < width; j += interval) {
			sum += mat[j];
		}
		vec[idx] = sum;
	}
}

__global__ void kDumbMaxCols(float* mat, float* vec, unsigned int width, \
		unsigned int height) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < height) {
		mat += idx * width;
		float mx = mat[0];
		for (int j = 1; j < width; j++) {
			mx = mat[j] > mx ? mat[j] : mx;
		}
		vec[idx] = mx;
	}
}

__global__ void kMultByColVector(float* mat, float* vec, float* tgtMat, \
		unsigned int width, unsigned int height) {
	//block.x表示行数，threadIdx.x表示列数
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int numThreads = blockDim.x * gridDim.x;

	tgtMat[idx] = mat[idx] * vec[blockIdx.x];


//	for (unsigned int i = idx; i < width * height; i += numThreads) {
//		tgtMat[i] = mat[i] * vec[i / width];
//	}
}

//__global__ void kSubtractFromScalar(float* gData, float scalar, float* target, \
		unsigned int numElements) {
__global__ void kSubtractFromScalar(float* gData, float* target, \
		unsigned int numElements) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x)
		target[i] = 1 - gData[i];
}

__global__ void kMult(float* matA, float* matB, float* tgtMat, \
		unsigned int width, unsigned int height) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int numThreads = blockDim.x * gridDim.x;

	for (unsigned int i = idx; i < width * height; i += numThreads) {
		tgtMat[i] = matA[i] * matB[i];
	}
}

__global__ void kAdd(float* matA, float* matB, float* tgtMat, float scaleA,  \
		float scaleB, unsigned int width, unsigned int height) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int numThreads = blockDim.x * gridDim.x;

	for (unsigned int i = idx; i < width * height; i += numThreads) {
		tgtMat[i] = scaleA * matA[i] + scaleB * matB[i];
	}
}


__global__ void kTranspose(float* srcData, float* dstData, \
		const int biggerDim, const int row, const int times){

	const unsigned int idx = threadIdx.x;
	//mIdx代表维数较小的一方，nIdx代表维数较大的一方
	const unsigned int mIdx = blockIdx.x;
	const unsigned int smallerDim = gridDim.x;

	for(int i = 0; i < times; i++){
		const unsigned int nIdx = i * blockDim.x + threadIdx.x;
		//假如行小于列，那么转置后的行大于列
		if(smallerDim == row)
			dstData[nIdx * smallerDim + mIdx] = srcData[mIdx * biggerDim + nIdx];
		else
			dstData[mIdx * biggerDim + nIdx] = srcData[nIdx * smallerDim + mIdx];

	}
	if(idx < biggerDim - blockDim.x * times){
		const unsigned int nIdx = times * blockDim.x + threadIdx.x;
		if(smallerDim == row)
			dstData[nIdx * smallerDim + mIdx] = srcData[mIdx * biggerDim + nIdx];
		else
			dstData[mIdx * biggerDim + nIdx] = srcData[nIdx * smallerDim + mIdx];
	}
}




